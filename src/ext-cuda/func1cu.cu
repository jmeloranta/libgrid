#include "hip/hip_runtime.h"
/*
 * Function #1: Backflow related function for libdft.
 *
 */

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>
#include "../hip/hip_runtime.h"
#include "../cuda-math.h"
#include "../cuda-vars.h"

#include "func1.h"

extern "C" void cuda_error_check();

__global__ void grid_func1_cuda_operate_one_product_gpu(CUREAL *dst, CUREAL *src1, CUREAL *src2, CUREAL xi, CUREAL rhobf, INT nx, INT ny, INT nz, INT nzz) {
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  CUREAL rhop;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  rhop = src2[idx];
  dst[idx] = src1[idx] * FUNCTION;
}

__global__ void grid_func1_cuda_operate_one_gpu(CUREAL *dst, CUREAL *src, CUREAL xi, CUREAL rhobf, INT nx, INT ny, INT nz, INT nzz) {
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  CUREAL rhop;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  rhop = src[idx];
  dst[idx] = FUNCTION;
}

extern "C" void grid_func1_cuda_operate_one_productW(gpu_mem_block *dst, gpu_mem_block *src1, gpu_mem_block *src2, CUREAL xi, CUREAL rhobf, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_REAL(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor, *SRC1 = src1->gpu_info->descriptor, *SRC2 = src2->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_func1_cuda_operate_one_product_gpu<<<blocks1,threads>>>((CUREAL *) DST->data[i], (CUREAL *) SRC1->data[i], (CUREAL *) SRC2->data[i], xi, rhobf, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_func1_cuda_operate_one_product_gpu<<<blocks2,threads>>>((CUREAL *) DST->data[i], (CUREAL *) SRC1->data[i], (CUREAL *) SRC2->data[i], xi, rhobf, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

extern "C" void grid_func1_cuda_operate_oneW(gpu_mem_block *dst, gpu_mem_block *src, CUREAL xi, CUREAL rhobf, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_REAL(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor, *SRC = src->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_func1_cuda_operate_one_gpu<<<blocks1,threads>>>((CUREAL *) DST->data[i], (CUREAL *) SRC->data[i], xi, rhobf, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_func1_cuda_operate_one_gpu<<<blocks2,threads>>>((CUREAL *) DST->data[i], (CUREAL *) SRC->data[i], xi, rhobf, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}
