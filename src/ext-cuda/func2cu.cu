#include "hip/hip_runtime.h"
/*
 * Function #2: Backflow related function for libdft.
 *
 */

#include <cuda/hip/hip_runtime_api.h>
#include <cuda/hip/hip_runtime.h>
#include <cuda/>
#include <hipfft/hipfft.h>
#include "../cuda-math.h"

#include "func2.h"

extern "C" void cuda_error_check();

__global__ void grid_func2_cuda_operate_one_product_gpu(CUREAL *c, CUREAL *b, CUREAL *a, CUREAL xi, CUREAL rhobf, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  CUREAL rhop;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  rhop = a[idx];
  c[idx] = b[idx] * FUNCTION;
}

__global__ void grid_func2_cuda_operate_one_gpu(CUREAL *c, CUREAL *a, CUREAL xi, CUREAL rhobf, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  CUREAL rhop;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  rhop = a[idx];
  c[idx] = FUNCTION;
}

extern "C" void grid_func2_cuda_operate_one_productW(CUREAL *gridc, CUREAL *gridb, CUREAL *grida, CUREAL xi, CUREAL rhobf, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  grid_func2_cuda_operate_one_product_gpu<<<blocks,threads>>>(gridc, gridb, grida, xi, rhobf, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

extern "C" void grid_func2_cuda_operate_oneW(CUREAL *gridc, CUREAL *grida, CUREAL xi, CUREAL rhobf, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  grid_func2_cuda_operate_one_gpu<<<blocks,threads>>>(gridc, grida, xi, rhobf, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}
