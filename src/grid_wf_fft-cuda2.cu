#include "hip/hip_runtime.h"
/*
 * CUDA device code (REAL complex; cgrid).
 *
 */

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>
#include "hip/hip_runtime.h"
#include "cuda-math.h"
#include "cuda-vars.h"

extern void *grid_gpu_mem_addr;
extern "C" void cuda_error_check();

/*
 * Kinetic energy propagation in Fourier space.
 *
 * Only periodic version implemented.
 *
 */

__global__ void grid_cuda_wf_propagate_kinetic_fft_gpu(CUCOMPLEX *b, CUREAL norm, CUREAL cx, CUREAL cy, CUREAL cz, CUREAL kx0, CUREAL ky0, CUREAL kz0, CUREAL maxk, CUCOMPLEX time_mass, INT nx, INT ny, INT nz, INT nyy, INT nx2, INT ny2, INT nz2, INT seg) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, jj = j + seg;
  CUREAL kx, ky, kz, kk;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;

  if (i <= nx2)
    kx = cx * ((CUREAL) i) - kx0;
  else
    kx = cx * ((CUREAL) (i - nx)) - kx0;

  if (jj <= ny2)
    ky = cy * ((CUREAL) jj) - ky0;
  else
    ky = cy * ((CUREAL) (jj - nyy)) - ky0;

  if (k <= nz2)
    kz = cz * ((CUREAL) k) - kz0; 
  else
    kz = cz * ((CUREAL) (k - nz)) - kz0;
      
  kk = kx * kx + ky * ky + kz * kz;
  /* psi(k,t+dt) = psi(k,t) exp( - i (hbar^2 * k^2 / 2m) dt / hbar ) */
  b[idx] = b[idx] * CUCEXP(time_mass * kk) * norm;
}

/*
 * Propagate kinetic energy in Fourier space.
 *
 * wf       = Source/destination grid for operation (gpu_mem_block *; input/output).
 * norm     = FFT norm (grid->fft_norm) (REAL; input).
 * kx0      = Momentum shift of origin along X (REAL; input).
 * ky0      = Momentum shift of origin along Y (REAL; input).
 * kz0      = Momentum shift of origin along Z (REAL; input).
 * step     = Spatial step length (REAL; input).
 * time_mass= Time step & mass (CUCOMPLEX; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 * Only periodic boundaries!
 *
 */

#include <stdio.h>

extern "C" void grid_cuda_wf_propagate_kinetic_fftW(gpu_mem_block *grid, CUREAL norm, CUREAL kx0, CUREAL ky0, CUREAL kz0, CUREAL step, CUCOMPLEX time_mass, INT nx, INT ny, INT nz) {

  INT segx = 0, segy = 0, nx2 = nx / 2, ny2 = ny / 2, nz2 = nz / 2;
  SETUP_VARIABLES_SEG(grid);
  cudaXtDesc *GRID = grid->gpu_info->descriptor;
  CUREAL cx, cy, cz;

  cx = 2.0 * M_PI / (((CUREAL) nx) * step);
  cy = 2.0 * M_PI / (((CUREAL) ny) * step);
  cz = 2.0 * M_PI / (((CUREAL) nz) * step);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(GRID->GPUs[i]);
    grid_cuda_wf_propagate_kinetic_fft_gpu<<<blocks1,threads>>>((CUCOMPLEX *) GRID->data[i], norm, cx, cy, cz, kx0, ky0, kz0, M_PI / step, 
        time_mass, nnx1, nny1, nz, ny, nx2, ny2, nz2, segy);
    segx += dsegx1;
    segy += dsegy1;
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(GRID->GPUs[i]);
    grid_cuda_wf_propagate_kinetic_fft_gpu<<<blocks2,threads>>>((CUCOMPLEX *) GRID->data[i], norm, cx, cy, cz, kx0, ky0, kz0, M_PI / step, 
        time_mass, nnx2, nny2, nz, ny, nx2, ny2, nz2, segy);
    segx += dsegx2;
    segy += dsegy2;
  }

  cuda_error_check();
}

/*
 * Kinetic energy propagation in Fourier space (Cayley).
 *
 * Only periodic version implemented.
 *
 */

__global__ void grid_cuda_wf_propagate_kinetic_cfft_gpu(CUCOMPLEX *b, CUREAL norm, CUREAL cx, CUREAL cy, CUREAL cz, CUREAL kx0, CUREAL ky0, CUREAL kz0, CUREAL maxk, CUCOMPLEX time_mass, INT nx, INT ny, INT nz, INT nyy, INT nx2, INT ny2, INT nz2, INT seg) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, jj = j + seg;
  CUREAL kx, ky, kz, kk;
  CUCOMPLEX tmp;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;

  if (i <= nx2)
    kx = cx * ((CUREAL) i) - kx0;
  else
    kx = cx * ((CUREAL) (i - nx)) - kx0;

  if (jj <= ny2)
    ky = cy * ((CUREAL) jj) - ky0;
  else
    ky = cy * ((CUREAL) (jj - nyy)) - ky0;

  if (k <= nz2)
    kz = cz * ((CUREAL) k) - kz0; 
  else
    kz = cz * ((CUREAL) (k - nz)) - kz0;
      
  kk = kx * kx + ky * ky + kz * kz;
  /* psi(k,t+dt) = psi(k,t) exp( - i (hbar^2 * k^2 / 2m) dt / hbar ) */
  /* exp ~ (1 + 0.5 * x) / (1 - 0.5 * x) */
  tmp = 0.5 * time_mass * kk;
  b[idx] = b[idx] * norm * (1.0 + tmp) / (1.0 - tmp);
}

/*
 * Propagate kinetic energy in Fourier space (CFFT).
 *
 * wf       = Source/destination grid for operation (gpu_mem_block *; input/output).
 * norm     = FFT norm (grid->fft_norm) (REAL; input).
 * kx0      = Momentum shift of origin along X (REAL; input).
 * ky0      = Momentum shift of origin along Y (REAL; input).
 * kz0      = Momentum shift of origin along Z (REAL; input).
 * step     = Spatial step length (REAL; input).
 * time_mass= Time step & mass (CUCOMPLEX; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 * Only periodic boundaries!
 *
 */

extern "C" void grid_cuda_wf_propagate_kinetic_cfftW(gpu_mem_block *grid, CUREAL norm, CUREAL kx0, CUREAL ky0, CUREAL kz0, CUREAL step, CUCOMPLEX time_mass, INT nx, INT ny, INT nz) {

  INT segx = 0, segy = 0, nx2 = nx / 2, ny2 = ny / 2, nz2 = nz / 2;
  SETUP_VARIABLES_SEG(grid);
  cudaXtDesc *GRID = grid->gpu_info->descriptor;
  CUREAL cx, cy, cz;

  cx = 2.0 * M_PI / (((CUREAL) nx) * step);
  cy = 2.0 * M_PI / (((CUREAL) ny) * step);
  cz = 2.0 * M_PI / (((CUREAL) nz) * step);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(GRID->GPUs[i]);
    grid_cuda_wf_propagate_kinetic_cfft_gpu<<<blocks1,threads>>>((CUCOMPLEX *) GRID->data[i], norm, cx, cy, cz, kx0, ky0, kz0, M_PI / step, 
        time_mass, nnx1, nny1, nz, ny, nx2, ny2, nz2, segy);
    segx += dsegx1;
    segy += dsegy1;
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(GRID->GPUs[i]);
    grid_cuda_wf_propagate_kinetic_cfft_gpu<<<blocks2,threads>>>((CUCOMPLEX *) GRID->data[i], norm, cx, cy, cz, kx0, ky0, kz0, M_PI / step, 
        time_mass, nnx2, nny2, nz, ny, nx2, ny2, nz2, segy);
    segx += dsegx2;
    segy += dsegy2;
  }

  cuda_error_check();
}
