#include "hip/hip_runtime.h"
/*
 * CUDA device code (for cuRAND).
 *
 * blockDim = # of threads
 * gridDim = # of blocks
 *
 */

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <hipfft/hipfft.h>
#ifdef SINGLE_PREC
#define CUREAL float
#define CUCOMPLEX hipfftComplex
#else
#define CUREAL double
#define CUCOMPLEX hipfftDoubleComplex
#endif
#include "cuda-math.h"
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include "cuda-vars.h"
#include "hip/hip_runtime.h"

extern cudaXtDesc *grid_gpu_rand_addr;
extern "C" void cuda_error_check();
extern "C" int cuda_ngpus();
extern "C" int *cuda_gpus();
extern "C" int cuda_unlock_block(void *);
extern "C" int cuda_remove_block(void *, char);
extern "C" gpu_mem_block *cuda_add_block(void *, size_t, hipfftHandle, char *, char);
extern "C" gpu_mem_block *cuda_find_block(void *);
extern "C" int cuda_lock_block(void *);

char grid_gpu_rand_holder;  // Place holder
void *grid_gpu_rand = NULL; // cuRAND states (host)
cudaXtDesc *grid_gpu_rand_addr = NULL; // cuRAND states (GPU)
size_t rand_prev_len = 0;

#define EXPORT

/*
 *
 * Set random number seeds for cuRAND.
 *
 * Every block has its own random number seed.
 * 
 */

__global__ void grid_cuda_random_seed_gpu(hiprandState *st, INT seed, INT nx, INT ny) {

  INT j = blockIdx.x * blockDim.x + threadIdx.x /* y */, i = blockIdx.y * blockDim.y + threadIdx.y /* x */, idx;

  if(i >= nx || j >= ny) return;

  idx = i * ny + j;

  hiprand_init(seed + idx, 0, 0, &st[idx]);
}

/*
 * Setup CURAND random number seeds. Grid points (nx, ny) will have their own RNG states.
 *
 * nx     = Max grid size along x (INT; input).
 * ny     = Max grid size along y (INT; input).
 * nz     = Max grid size along z (INT; input).
 * seed   = Base random number seed (INT; input).
 *
 */

extern "C" void grid_cuda_random_seedW(INT nx, INT ny, INT nz, INT seed) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK), 
       blocks((ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK, (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  int *gpus = (int *) cuda_gpus();  // This must be int rather than INT
  INT i;
  size_t len;

  /* Every (x,y) grid element has its own state */
  len = ((size_t) nx * ny);

  if(rand_prev_len < len) {
    if(grid_gpu_rand) {
      cuda_unlock_block(grid_gpu_rand);
      cuda_remove_block(grid_gpu_rand, 0);
    }
    rand_prev_len = len;
    grid_gpu_rand = (void *) &grid_gpu_rand_holder;
    if(!(cuda_add_block(grid_gpu_rand, len * sizeof(hiprandState), -1, (char *) "GPU RAND", 0))) {
      fprintf(stderr, "libgrid(CUDA): Failed to allocate temporary space on GPU.\n");
      abort();
    }
    grid_gpu_rand_addr = (cuda_find_block(grid_gpu_rand))->gpu_info->descriptor;
    cuda_lock_block(grid_gpu_rand);
  }

  for(i = 0; i < cuda_ngpus(); i++) {
    hipSetDevice(gpus[i]);
    grid_cuda_random_seed_gpu<<<blocks,threads>>>((hiprandState *) grid_gpu_rand_addr->data[i], seed, nx, ny);
  }
  cuda_error_check();
}

/*
 * Add uniform random numbers to real grid (uniform distribution between +- scale).
 *
 */

__global__ void rgrid_cuda_random_uniform_gpu(CUREAL *grid, hiprandState *st, CUREAL scale, INT nx, INT ny, INT nz, INT nzz) {

  INT k, j = blockIdx.x * blockDim.x + threadIdx.x, i = blockIdx.y * blockDim.y + threadIdx.y, idx, sidx;

  if(i >= nx || j >= ny) return;

  idx = (i * ny + j) * nzz;
  sidx = i * ny + j;

  for(k = 0; k < nz; k++, idx++) {
#ifdef SINGLE_PREC
    grid[idx] = grid[idx] + 2.0 * (hiprand_uniform(&st[sidx]) - 0.5) * scale;
#else
    grid[idx] = grid[idx] + 2.0 * (hiprand_uniform_double(&st[sidx]) - 0.5) * scale;
#endif
  }
}

/*
 * Add uniform random numbers between -scale and +scale to real grid.
 *
 * grid    = Destination for operation (gpu_mem_block *; output).
 * scale   = Scale for the random numbers (CUREAL; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_random_uniformW(gpu_mem_block *grid, CUREAL scale, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_REAL2(grid);
  cudaXtDesc *GRID = grid->gpu_info->descriptor;

  if(grid->gpu_info->subFormat != HIPFFT_XT_FORMAT_INPLACE) {
    fprintf(stderr, "libgird(cuda): random_uniform wrong subformat.\n");
    abort();
  }

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(GRID->GPUs[i]);
    rgrid_cuda_random_uniform_gpu<<<blocks1,threads>>>((CUREAL *) GRID->data[i], (hiprandState *) grid_gpu_rand_addr->data[i], scale, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(GRID->GPUs[i]);
    rgrid_cuda_random_uniform_gpu<<<blocks2,threads>>>((CUREAL *) GRID->data[i], (hiprandState *) grid_gpu_rand_addr->data[i], scale, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 * Add normal random numbers to real grid (normal distribution between +- scale).
 *
 */

__global__ void rgrid_cuda_random_normal_gpu(CUREAL *grid, hiprandState *st, CUREAL scale, INT nx, INT ny, INT nz, INT nzz) {

  INT k, j = blockIdx.x * blockDim.x + threadIdx.x, i = blockIdx.y * blockDim.y + threadIdx.y, idx, sidx;

  if(i >= nx || j >= ny) return;

  idx = (i * ny + j) * nzz;
  sidx = i * ny + j;

  for(k = 0; k < nz; k++, idx++) {
#ifdef SINGLE_PREC
    grid[idx] = grid[idx] + scale * hiprand_normal(&st[sidx]);
#else
    grid[idx] = grid[idx] + scale * hiprand_normal_double(&st[sidx]);
#endif
  }
}

/*
 * Add normal random numbers between -1 and 1 (scaled by "scale").
 *
 * grid    = Destination for operation (gpu_mem_block *; output).
 * scale   = Scaling factor (CUREAL; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_random_normalW(gpu_mem_block *grid, CUREAL scale, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_REAL2(grid);
  cudaXtDesc *GRID = grid->gpu_info->descriptor;
 
  if(grid->gpu_info->subFormat != HIPFFT_XT_FORMAT_INPLACE) {
    fprintf(stderr, "libgird(cuda): random_normal wrong subformat.\n");
    abort();
  }

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(GRID->GPUs[i]);
    rgrid_cuda_random_normal_gpu<<<blocks1,threads>>>((CUREAL *) GRID->data[i], (hiprandState *) grid_gpu_rand_addr->data[i], scale, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(GRID->GPUs[i]);
    rgrid_cuda_random_normal_gpu<<<blocks2,threads>>>((CUREAL *) GRID->data[i], (hiprandState *) grid_gpu_rand_addr->data[i], scale, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 * Add uniform random numbers to complex grid (uniform distribution between +- scale).
 *
 */

__global__ void cgrid_cuda_random_uniform_gpu(CUCOMPLEX *grid, hiprandState *st, CUCOMPLEX scale, INT nx, INT ny, INT nz) {

  INT k, j = blockIdx.x * blockDim.x + threadIdx.x, i = blockIdx.y * blockDim.y + threadIdx.y, idx, sidx;

  if(i >= nx || j >= ny) return;

  idx = (i * ny + j) * nz;
  sidx = i * ny + j;

  for(k = 0; k < nz; k++, idx++) {
#ifdef SINGLE_PREC
    grid[idx] = grid[idx] + CUMAKE(2.0 * (hiprand_uniform(&st[sidx]) - 0.5) * scale.x, 2.0 * (hiprand_uniform(&st[sidx]) - 0.5) * scale.y);
#else
    grid[idx] = grid[idx] + CUMAKE(2.0 * (hiprand_uniform_double(&st[sidx]) - 0.5) * scale.x, 2.0 * (hiprand_uniform_double(&st[sidx]) - 0.5) * scale.y);
#endif
  }
}

/*
 * Add uniform random numbers between -scale and +scale to real grid.
 *
 * grid    = Destination for operation (gpu_mem_block *; output).
 * scale   = Random number scle (-scale, scale) (CUCOMPLEX; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_random_uniformW(gpu_mem_block *grid, CUCOMPLEX scale, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES2(grid);
  cudaXtDesc *GRID = grid->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_random_uniform_gpu<<<blocks1,threads>>>((CUCOMPLEX *) GRID->data[i], (hiprandState *) grid_gpu_rand_addr->data[i], scale, nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_random_uniform_gpu<<<blocks2,threads>>>((CUCOMPLEX *) GRID->data[i], (hiprandState *) grid_gpu_rand_addr->data[i], scale, nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * Add uniform random numbers to complex grid (uniform distribution between +- scale).
 *
 */

__global__ void cgrid_cuda_random_uniform_sp_gpu(CUCOMPLEX *grid, hiprandState *st, CUREAL scale, INT nx, INT ny, INT nz) {

  INT k, j = blockIdx.x * blockDim.x + threadIdx.x, i = blockIdx.y * blockDim.y + threadIdx.y, idx, sidx;
  REAL r1, r2;

  if(i >= nx || j >= ny) return;

  idx = (i * ny + j) * nz;
  sidx = i * ny + j;

  for(k = 0; k < nz; k++, idx++) {
#ifdef SINGLE_PREC
    r1 = scale * hiprand_uniform(&st[sidx]);
    r2 = hiprand_uniform(&st[sidx]) * 2.0 * M_PI;
#else
    r1 = scale * hiprand_uniform_double(&st[sidx]);
    r2 = hiprand_uniform_double(&st[sidx]) * 2.0 * M_PI;
#endif
    grid[idx].x = grid[idx].x + r1 * COS(r2);
    grid[idx].y = grid[idx].y + r1 * SIN(r2);
  }
}

/*
 * Add uniform random numbers between -scale and +scale to real grid.
 *
 * grid    = Destination for operation (gpu_mem_block *; output).
 * scale   = Random number scle (-scale, scale) (CUREAL; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_random_uniform_spW(gpu_mem_block *grid, CUREAL scale, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES2(grid);
  cudaXtDesc *GRID = grid->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_random_uniform_sp_gpu<<<blocks1,threads>>>((CUCOMPLEX *) GRID->data[i], (hiprandState *) grid_gpu_rand_addr->data[i], scale, nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_random_uniform_sp_gpu<<<blocks2,threads>>>((CUCOMPLEX *) GRID->data[i], (hiprandState *) grid_gpu_rand_addr->data[i], scale, nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * Add normal random numbers to real grid (normal distribution between +- scale).
 *
 */

__global__ void cgrid_cuda_random_normal_gpu(CUCOMPLEX *grid, hiprandState *st, CUCOMPLEX scale, INT nx, INT ny, INT nz) {

  INT k, j = blockIdx.x * blockDim.x + threadIdx.x, i = blockIdx.y * blockDim.y + threadIdx.y, idx, sidx;

  if(i >= nx || j >= ny) return;

  idx = (i * ny + j) * nz;
  sidx = i * ny + j;

  for(k = 0; k < nz; k++, idx++) {
#ifdef SINGLE_PREC
    grid[idx] = grid[idx] + CUMAKE(scale.x * hiprand_normal(&st[sidx]), scale.y * hiprand_normal(&st[sidx]));
#else
    grid[idx] = grid[idx] + CUMAKE(scale.x * hiprand_normal_double(&st[sidx]), scale.y * hiprand_normal_double(&st[sidx]));
#endif
  }
}

/*
 * Add normal random numbers between -scale and +scale to real grid.
 *
 * grid    = Destination for operation (gpu_mem_block *; output).
 * scale   = Scale for the random numbers (CUCOMPLEX; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_random_normalW(gpu_mem_block *grid, CUCOMPLEX scale, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES2(grid);
  cudaXtDesc *GRID = grid->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_random_normal_gpu<<<blocks1,threads>>>((CUCOMPLEX *) GRID->data[i], (hiprandState *) grid_gpu_rand_addr->data[i], scale, nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_random_normal_gpu<<<blocks2,threads>>>((CUCOMPLEX *) GRID->data[i], (hiprandState *) grid_gpu_rand_addr->data[i], scale, nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * Add normal random numbers to real grid (normal distribution between +- scale).
 *
 */

__global__ void cgrid_cuda_random_normal_sp_gpu(CUCOMPLEX *grid, hiprandState *st, CUREAL scale, INT nx, INT ny, INT nz) {

  INT k, j = blockIdx.x * blockDim.x + threadIdx.x, i = blockIdx.y * blockDim.y + threadIdx.y, idx, sidx;
  CUREAL r1, r2;

  if(i >= nx || j >= ny) return;

  idx = (i * ny + j) * nz;
  sidx = i * ny + j;

  for(k = 0; k < nz; k++, idx++) {
#ifdef SINGLE_PREC
    r1 = scale * hiprand_normal(&st[sidx]);
    r2 = hiprand_uniform(&st[sidx]) * M_PI;
#else
    r1 = scale * hiprand_normal_double(&st[sidx]);
    r2 = hiprand_uniform_double(&st[sidx]) * M_PI;
#endif
    grid[idx].x = grid[idx].x + r1 * COS(r2);
    grid[idx].y = grid[idx].y + r1 * SIN(r2);
  }
}

/*
 * Add normal random numbers between -scale and +scale to real grid.
 *
 * grid    = Destination for operation (gpu_mem_block *; output).
 * scale   = Scale for the random numbers (CUREAL; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_random_normal_spW(gpu_mem_block *grid, CUREAL scale, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES2(grid);
  cudaXtDesc *GRID = grid->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_random_normal_sp_gpu<<<blocks1,threads>>>((CUCOMPLEX *) GRID->data[i], (hiprandState *) grid_gpu_rand_addr->data[i], scale, nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_random_normal_sp_gpu<<<blocks2,threads>>>((CUCOMPLEX *) GRID->data[i], (hiprandState *) grid_gpu_rand_addr->data[i], scale, nnx2, nny2, nz);
  }

  cuda_error_check();
}
