#include "hip/hip_runtime.h"
/*
 * CUDA device code (REAL complex = CUCOMPLEX; cgrid).
 *
 * blockDim = # of threads
 * gridDim = # of blocks
 *
 * Since the ordering of the data on multiple GPUs changes after FFT,
 * we must keep track of this. The variable space in cgrid structure (structs.h)
 * is zero if the grid is currently in real space and one if it is in reciprocal space (i.e., after FFT).
 * The real space data is distributed on GPUs according to the first index (x as in data[x][y][z])
 * where as the resiprocal space data is distributed along the second index (y as in data[x][y][z]).
 *
 * SETUP_VARIABLES and SETUP_VARIABLES_SEG are macros that follow the correct distribution over GPUs
 * based on the value of space variable. The latter also keeps track of the actual index so that
 * routines that need to access proper grid indices can do so.
 *
 */

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include "cuda-math.h"
#include "cgrid_bc-cuda.h"
#include "cuda-vars.h"
#include "hip/hip_runtime.h"

extern void *grid_gpu_mem;            /* host memory holder for the block used for reductions */
extern cudaXtDesc *grid_gpu_mem_addr; /* the corresponding GPU memory addresses */
extern "C" void cuda_error_check();

/*
 * Fourier space convolution device code.
 *
 * dst = src1 * src2 but with alternating signs for FFT.
 *
 */

__global__ void cgrid_cuda_fft_convolute_gpu(CUCOMPLEX *dst, CUCOMPLEX *src1, CUCOMPLEX *src2, CUREAL norm, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  if((i + j + k) & 1) norm *= -1.0;
  dst[idx] = src1[idx] * src2[idx] * norm;
}

/*
 * Convolution in the Fourier space.
 *
 * dst   = convolution output (gpu_mem_block *; output).
 * src1  = 1st grid to be convoluted (gpu_mem_block *; input).
 * src2  = 2nd grid to be convoluted (gpu_mem_block *; input).
 * norm  = FFT norm (REAL complex; input).
 * nx    = Grid dim x (INT; input).
 * ny    = Grid dim y (INT; input).
 * nz    = Grid dim z (INT; input).
 *
 * In Fourier space.
 *
 */

extern "C" void cgrid_cuda_fft_convoluteW(gpu_mem_block *dst, gpu_mem_block *src1, gpu_mem_block *src2, CUREAL norm, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *SRC1 = src1->gpu_info->descriptor, *SRC2 = src2->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_convolute_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) SRC1->data[i], (CUCOMPLEX *) SRC2->data[i], norm, nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_convolute_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) SRC1->data[i], (CUCOMPLEX *) SRC2->data[i], norm, nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * Grid abs power device code.
 *
 * dst = POW(|src|,x)
 *
 */

__global__ void cgrid_cuda_abs_power_gpu(CUCOMPLEX *dst, CUCOMPLEX *src, CUREAL x, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;

  dst[idx].x = POW(CUCREAL(src[idx]) * CUCREAL(src[idx]) + CUCIMAG(src[idx]) * CUCIMAG(src[idx]), x / 2.0);
  dst[idx].y = 0.0;
}

/*
 * Grid abs power.
 *
 * dst      = Destination for operation (gpu_mem_block *; input).
 * src      = Source for operation (gpu_mem_block *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 * In real space.
 *
 */

extern "C" void cgrid_cuda_abs_powerW(gpu_mem_block *dst, gpu_mem_block *src, CUREAL exponent, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *SRC = src->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_abs_power_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) SRC->data[i], exponent, nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_abs_power_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) SRC->data[i], exponent, nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * Grid power device code.
 *
 * dst = POW(src,x)
 *
 */

__global__ void cgrid_cuda_power_gpu(CUCOMPLEX *dst, CUCOMPLEX *src, CUREAL x, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;

  dst[idx] = CUCPOW(src[idx], x);
}

/*
 * Grid power.
 *
 * dst      = Destination for operation (gpu_mem_block *; output).
 * src      = Source for operation (gpu_mem_block *; input).
 * exponent = Exponent (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_powerW(gpu_mem_block *dst, gpu_mem_block *src, CUREAL exponent, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *SRC = src->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_power_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) SRC->data[i], exponent, nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_power_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) SRC->data[i], exponent, nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * Multiply grid by constant device code.
 *
 * dst = C * dst
 *
 */

__global__ void cgrid_cuda_multiply_gpu(CUCOMPLEX *dst, CUCOMPLEX c, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = dst[idx] * c;
}

/*
 * Multiply grid by a constant.
 *
 * dst      = Grid to be operated on (gpu_mem_block *; input/output).
 * c        = Multiplying constant (CUCOMPLEX; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_multiplyW(gpu_mem_block *dst, CUCOMPLEX c, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_multiply_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], c, nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_multiply_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], c, nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * Sum of two grids.
 *
 * dst = src1 + src2
 *
 */

__global__ void cgrid_cuda_sum_gpu(CUCOMPLEX *dst, CUCOMPLEX *src1, CUCOMPLEX *src2, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = src1[idx] + src2[idx];
}

/*
 * Sum of two grids.
 *
 * dst      = Destination grid (gpu_mem_block *; output).
 * src1     = Input grid 1 (gpu_mem_block *; input).
 * src2     = Input grid 2 (gpu_mem_block *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_sumW(gpu_mem_block *dst, gpu_mem_block *src1, gpu_mem_block *src2, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *SRC1 = src1->gpu_info->descriptor, *SRC2 = src2->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_sum_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) SRC1->data[i], (CUCOMPLEX *) SRC2->data[i], nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_sum_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) SRC1->data[i], (CUCOMPLEX *) SRC2->data[i], nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * Subtract of two grids.
 *
 * dst = src1 - src2
 *
 */

__global__ void cgrid_cuda_difference_gpu(CUCOMPLEX *dst, CUCOMPLEX *src1, CUCOMPLEX *src2, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = src1[idx] - src2[idx];
}

/*
 * Subtract two grids.
 *
 * dst      = Destination grid (gpu_mem_block *; output).
 * src1     = Input grid 1 (gpu_mem_block *; input).
 * src2     = Input grid 2 (gpu_mem_block *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_differenceW(gpu_mem_block *dst, gpu_mem_block *src1, gpu_mem_block *src2, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *SRC1 = src1->gpu_info->descriptor, *SRC2 = src2->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_difference_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) SRC1->data[i], (CUCOMPLEX *) SRC2->data[i], nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_difference_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) SRC1->data[i], (CUCOMPLEX *) SRC2->data[i], nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * Product of two grids.
 *
 * dst = src1 * src2.
 *
 */

__global__ void cgrid_cuda_product_gpu(CUCOMPLEX *dst, CUCOMPLEX *src1, CUCOMPLEX *src2, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = src1[idx] * src2[idx];
}

/*
 * Product of two grids.
 *
 * grida    = Destination grid (gpu_mem_block *; output).
 * gridb    = Source grid 1 (gpu_mem_block *; input).
 * gridc    = Source grid 2 (gpu_mem_block *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_productW(gpu_mem_block *dst, gpu_mem_block *src1, gpu_mem_block *src2, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *SRC1 = src1->gpu_info->descriptor, *SRC2 = src2->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_product_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) SRC1->data[i], (CUCOMPLEX *) SRC2->data[i], nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_product_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) SRC1->data[i], (CUCOMPLEX *) SRC2->data[i], nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * Conjugate product of two grids.
 *
 * dst = src1^* X src2.
 *
 */

__global__ void cgrid_cuda_conjugate_product_gpu(CUCOMPLEX *dst, CUCOMPLEX *src1, CUCOMPLEX *src2, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = CUCONJ(src1[idx]) * src2[idx];
}

/*
 * Conjugate product of two grids.
 *
 * dst      = Destination grid (gpu_mem_block *; output).
 * src1     = Source grid 1 (gpu_mem_block *; input).
 * src2     = Source grid 2 (gpu_mem_block *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_conjugate_productW(gpu_mem_block *dst, gpu_mem_block *src1, gpu_mem_block *src2, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *SRC1 = src1->gpu_info->descriptor, *SRC2 = src2->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_conjugate_product_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) SRC1->data[i], (CUCOMPLEX *) SRC2->data[i], nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_conjugate_product_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) SRC1->data[i], (CUCOMPLEX *) SRC2->data[i], nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * Division of two grids.
 *
 * dst = src1 / src2.
 *
 * Note: Avoid division as it is slow on GPUs.
 *
 */

__global__ void cgrid_cuda_division_gpu(CUCOMPLEX *dst, CUCOMPLEX *src1, CUCOMPLEX *src2, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = src1[idx] / src2[idx];
}

/*
 * Division of two grids.
 *
 * dst      = Destination grid (gpu_mem_block *; output).
 * src1     = Source grid 1 (gpu_mem_block *; input).
 * src2     = Source grid 2 (gpu_mem_block *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_divisionW(gpu_mem_block *dst, gpu_mem_block *src1, gpu_mem_block *src2, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *SRC1 = src1->gpu_info->descriptor, *SRC2 = src2->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_division_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) SRC1->data[i], (CUCOMPLEX *) SRC2->data[i], nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_division_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) SRC1->data[i], (CUCOMPLEX *) SRC2->data[i], nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * Safe division of two grids.
 *
 * dst = src1 / (src2 + eps).
 *
 * Note: Avoid division as it is slow on GPUs.
 *
 */

__global__ void cgrid_cuda_division_eps_gpu(CUCOMPLEX *dst, CUCOMPLEX *src1, CUCOMPLEX *src2, CUREAL eps, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  
  dst[idx] = src1[idx] / (src2[idx] + eps);
}

/*
 * "Safe" division of two grids.
 *
 * dst      = Destination grid (gpu_mem_block *; output).
 * src1     = Source grid 1 (gpu_mem_block *; input).
 * src2     = Source grid 2 (gpu_mem_block *; input).
 * eps      = Epsilon (CUCOMPLEX; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_division_epsW(gpu_mem_block *dst, gpu_mem_block *src1, gpu_mem_block *src2, CUREAL eps, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *SRC1 = src1->gpu_info->descriptor, *SRC2 = src2->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_division_eps_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) SRC1->data[i], (CUCOMPLEX *) SRC2->data[i], eps, nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_division_eps_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) SRC1->data[i], (CUCOMPLEX *) SRC2->data[i], eps, nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * Add constant to grid device code.
 *
 * dst = dst + c
 *
 */

__global__ void cgrid_cuda_add_gpu(CUCOMPLEX *dst, CUCOMPLEX c, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = dst[idx] + c;
}

/*
 * Add constant to grid.
 *
 * dst      = Grid to be operated on (gpu_mem_block *; input/output).
 * c        = Constant (CUCOMPLEX).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_addW(gpu_mem_block *dst, CUCOMPLEX c, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_add_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], c, nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_add_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], c, nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * Add multiply and add device code.
 *
 * dst = cm * dst + ca
 *
 */

__global__ void cgrid_cuda_multiply_and_add_gpu(CUCOMPLEX *dst, CUCOMPLEX cm, CUCOMPLEX ca, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = (cm * dst[idx]) + ca;
}

/*
 * Grid multiply and add.
 *
 * dst      = Grid to be operated on (gpu_mem_block *; input/output).
 * cm       = Multiplier (CUCOMPLEX; input).
 * ca       = Additive constant (CUCOMPLEX; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_multiply_and_addW(gpu_mem_block *dst, CUCOMPLEX cm, CUCOMPLEX ca, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_multiply_and_add_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], cm, ca, nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_multiply_and_add_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], cm, ca, nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * Add multiply and add device code.
 *
 * dst = cm * (dst + ca)
 *
 */

__global__ void cgrid_cuda_add_and_multiply_gpu(CUCOMPLEX *dst, CUCOMPLEX ca, CUCOMPLEX cm, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = cm * (dst[idx] + ca);
}

/*
 * Grid multiply and add.
 *
 * dst      = Grid to be operated on (gpu_mem_block *; input/output).
 * ca       = Additive constant (CUCOMPLEX; input).
 * cm       = Multiplier (CUCOMPLEX; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_add_and_multiplyW(gpu_mem_block *dst, CUCOMPLEX ca, CUCOMPLEX cm, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_add_and_multiply_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], ca, cm, nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_add_and_multiply_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], ca, cm, nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * Add scaled grid device code.
 *
 * dst = dst + d * src
 *
 */

__global__ void cgrid_cuda_add_scaled_gpu(CUCOMPLEX *dst, CUCOMPLEX d, CUCOMPLEX *src, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = dst[idx] + (d * src[idx]);
}

/*
 * Scaled add grid.
 *
 * dst      = Destination for operation (gpu_mem_block *; output).
 * d        = Scaling factor (CUCOMPLEX; input).
 * srd      = Source for operation (gpu_mem_block *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_add_scaledW(gpu_mem_block *dst, CUCOMPLEX d, gpu_mem_block *src, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *SRC = src->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_add_scaled_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], d, (CUCOMPLEX *) SRC->data[i], nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_add_scaled_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], d, (CUCOMPLEX *) SRC->data[i], nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * Add scaled product grid device code.
 *
 * dst = dst + d * src1 * src2
 *
 */

__global__ void cgrid_cuda_add_scaled_product_gpu(CUCOMPLEX *dst, CUCOMPLEX d, CUCOMPLEX *src1, CUCOMPLEX *src2, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = dst[idx] + (d * src1[idx] * src2[idx]);
}

/*
 * Add scaled product.
 *
 * dst      = Destination for operation (gpu_mem_block *; output).
 * d        = Scaling factor (CUCOMPLEX; input).
 * src1     = Source for operation (gpu_mem_block *; input).
 * src2     = Source for operation (gpu_mem_block *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_add_scaled_productW(gpu_mem_block *dst, CUCOMPLEX d, gpu_mem_block *src1, gpu_mem_block *src2, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *SRC1 = src1->gpu_info->descriptor, *SRC2 = src2->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_add_scaled_product_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], d, (CUCOMPLEX *) SRC1->data[i], (CUCOMPLEX *) SRC2->data[i], nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_add_scaled_product_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], d, (CUCOMPLEX *) SRC1->data[i], (CUCOMPLEX *) SRC2->data[i], nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * Set dst to constant: dst = c
 *
 */

__global__ void cgrid_cuda_constant_gpu(CUCOMPLEX *dst, CUCOMPLEX c, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = c;
}

/*
 * Set grid to constant.
 *
 * dst      = Destination for operation (gpu_mem_block *; output).
 * c        = Constant (CUCOMPLEX; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_constantW(gpu_mem_block *dst, CUCOMPLEX c, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_constant_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], c, nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_constant_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], c, nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * Block init (zero elements).
 *
 * blocks  = Block table (CUCOMPLEX *; output).
 * nblocks = Number of blocks in table (INT; input).
 *
 */

__global__ void cgrid_cuda_block_init(CUCOMPLEX *blocks, INT nblocks) {

  INT i;

  for(i = 0; i < nblocks; i++) blocks[i].x = blocks[i].y = 0.0;
}

/*
 * Block reduction.
 *
 * blocks  = Block list to reduce (CUCOMPLEX *; input/output). blocks[0] will contain the reduced value.
 * nblocks = Number of blocks (INT; input).
 *
 */

__global__ void cgrid_cuda_block_reduce(CUCOMPLEX *blocks, INT nblocks) {

  INT i;

  for(i = 1; i < nblocks; i++) blocks[0] = blocks[0] + blocks[i];
}

/*
 * Integrate over grid a.
 *
 */

__global__ void cgrid_cuda_integral_gpu(CUCOMPLEX *a, CUCOMPLEX *blocks, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2, t;
  extern __shared__ CUCOMPLEX els[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els[t].x = els[t].y = 0.0;
  }
  __syncthreads();

  idx = (i * ny + j) * nz + k;
  idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
  els[idx2] = els[idx2] + a[idx];
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] = blocks[idx2] + els[t]; // reduce threads
    }
  }
}

/*
 * Integrate over grid.
 *
 * grid     = Source for operation (gpu_mem_block *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 * value    = Result (CUCOMPLEX *; output).
 *
 */

extern "C" void cgrid_cuda_integralW(gpu_mem_block *grid, INT nx, INT ny, INT nz, CUCOMPLEX *value) {

  SETUP_VARIABLES(grid);
  cudaXtDesc *GRID = grid->gpu_info->descriptor;
  CUCOMPLEX tmp;
  INT s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b31 = blocks1.x * blocks1.y * blocks1.z, b32 = blocks2.x * blocks2.y * blocks2.z;
  extern int cuda_get_element(void *, int, size_t, size_t, void *);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_block_init<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b31);
    // Blocks, Threads, dynamic memory size
    cgrid_cuda_integral_gpu<<<blocks1,threads,s*sizeof(CUCOMPLEX)>>>((CUCOMPLEX *) GRID->data[i], (CUCOMPLEX *) grid_gpu_mem_addr->data[i], nnx1, nny1, nz);
    cgrid_cuda_block_reduce<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b31);  // reduce over blocks
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_block_init<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b32);
    // Blocks, Threads, dynamic memory size
    cgrid_cuda_integral_gpu<<<blocks2,threads,s*sizeof(CUCOMPLEX)>>>((CUCOMPLEX *) GRID->data[i], (CUCOMPLEX *) grid_gpu_mem_addr->data[i], nnx2, nny2, nz);
    cgrid_cuda_block_reduce<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b32);
  }

  // Reduce over GPUs
  *value = CUMAKE(0.0,0.0);
  for(i = 0; i < ngpu2; i++) {
    cuda_get_element(grid_gpu_mem, i, 0, sizeof(CUCOMPLEX), &tmp);
    value->x += tmp.x;  // + overloaded to device function - work around!
    value->y += tmp.y;
  }

  cuda_error_check();
}

/*
 * Integrate over A with limits.
 *
 * nx = number of elements for current GPU.
 * nx2 = Position in the overall grid.
 *
 */

__global__ void cgrid_cuda_integral_region_gpu(CUCOMPLEX *a, CUCOMPLEX *blocks, INT il, INT iu, INT jl, INT ju, INT kl, INT ku, INT nx, INT ny, INT nz, INT segx, INT segy) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, ii = i + segx, jj = j + segy;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2, t;
  extern __shared__ CUCOMPLEX els[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els[t].x = els[t].y = 0.0;
  }
  __syncthreads();

  if(ii >= il && ii <= iu && jj >= jl && jj <= ju && k >= kl && k <= ku) {
    idx = (i * ny + j) * nz + k;
    idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
    els[idx2] = els[idx2] + a[idx];
  }
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] = blocks[idx2] + els[t];  // reduce threads
    }
  }
}

/*
 * Integrate over grid with limits.
 *
 * grid     = Source for operation (gpu_mem_block *; input).
 * il       = Lower index for x (INT; input).
 * iu       = Upper index for x (INT; input).
 * jl       = Lower index for y (INT; input).
 * ju       = Upper index for y (INT; input).
 * kl       = Lower index for z (INT; input).
 * ku       = Upper index for z (INT; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 * value    = Result (CUCOMPLEX *; output).
 *
 */

extern "C" void cgrid_cuda_integral_regionW(gpu_mem_block *grid, INT il, INT iu, INT jl, INT ju, INT kl, INT ku, INT nx, INT ny, INT nz, CUCOMPLEX *value) {

  SETUP_VARIABLES_SEG(grid);
  cudaXtDesc *GRID = grid->gpu_info->descriptor;
  CUCOMPLEX tmp;
  INT s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b31 = blocks1.x * blocks1.y * blocks1.z, b32 = blocks2.x * blocks2.y * blocks2.z, segx = 0, segy = 0;
  extern int cuda_get_element(void *, int, size_t, size_t, void *);

  if(il < 0) il = 0;  
  if(jl < 0) jl = 0;  
  if(kl < 0) kl = 0;  
  if(iu > nx-1) iu = nx-1;
  if(ju > ny-1) ju = ny-1;
  if(ku > nz-1) ku = nz-1;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_block_init<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b31);
    // Blocks, Threads, dynamic memory size
    cgrid_cuda_integral_region_gpu<<<blocks1,threads,s*sizeof(CUCOMPLEX)>>>((CUCOMPLEX *) GRID->data[i], (CUCOMPLEX *) grid_gpu_mem_addr->data[i], il, iu, jl, ju, kl, ku, nnx1, nny1, nz, segx, segy);
    segx += dsegx1;
    segy += dsegy1;
    cgrid_cuda_block_reduce<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b31);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_block_init<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b32);
    // Blocks, Threads, dynamic memory size
    cgrid_cuda_integral_region_gpu<<<blocks2,threads,s*sizeof(CUCOMPLEX)>>>((CUCOMPLEX *) GRID->data[i], (CUCOMPLEX *) grid_gpu_mem_addr->data[i], il, iu, jl, ju, kl, ku, nnx2, nny2, nz, segx, segy);
    segx += dsegx2;
    segy += dsegy2;
    cgrid_cuda_block_reduce<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b32);
  }

  // Reduce over GPUs
  *value = CUMAKE(0.0,0.0);
  for(i = 0; i < ngpu2; i++) {
    cuda_get_element(grid_gpu_mem, i, 0, sizeof(CUCOMPLEX), &tmp);
    value->x += tmp.x;  /// + overloaded to device function - work around!
    value->y += tmp.y;
  }

  cuda_error_check();
}

/*
 * Integrate of |A|^2.
 *
 */

__global__ void cgrid_cuda_integral_of_square_gpu(CUCOMPLEX *a, CUCOMPLEX *blocks, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2, t;
  extern __shared__ CUCOMPLEX els[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els[t].x = els[t].y = 0.0;
  }
  __syncthreads();

  idx = (i * ny + j) * nz + k;
  idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
  els[idx2] = els[idx2] + (a[idx].x * a[idx].x + a[idx].y * a[idx].y);
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] = blocks[idx2] + els[t];  // reduce threads
    }
  }
}

/*
 * Integral of square (|grid|^2).
 *
 * grid     = Source for operation (udaXtDesc_t *; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 * value    = Result (CUCOMPLEX *; output).
 *
 */

extern "C" void cgrid_cuda_integral_of_squareW(gpu_mem_block *grid, INT nx, INT ny, INT nz, CUCOMPLEX *value) {

  SETUP_VARIABLES(grid);
  cudaXtDesc *GRID = grid->gpu_info->descriptor;
  CUCOMPLEX tmp;
  INT s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b31 = blocks1.x * blocks1.y * blocks1.z, b32 = blocks2.x * blocks2.y * blocks2.z;
  extern int cuda_get_element(void *, int, size_t, size_t, void *);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_block_init<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b31);
    // Blocks, Threads, dynamic memory size
    cgrid_cuda_integral_of_square_gpu<<<blocks1,threads,s*sizeof(CUCOMPLEX)>>>((CUCOMPLEX *) GRID->data[i], (CUCOMPLEX *) grid_gpu_mem_addr->data[i], nnx1, nny1, nz);
    cgrid_cuda_block_reduce<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b31);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_block_init<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b32);
    // Blocks, Threads, dynamic memory size
    cgrid_cuda_integral_of_square_gpu<<<blocks2,threads,s*sizeof(CUCOMPLEX)>>>((CUCOMPLEX *) GRID->data[i], (CUCOMPLEX *) grid_gpu_mem_addr->data[i], nnx2, nny2, nz);
    cgrid_cuda_block_reduce<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b32);
  }

  // Reduce over GPUs
  *value = CUMAKE(0.0,0.0);
  for(i = 0; i < ngpu2; i++) {
    cuda_get_element(grid_gpu_mem, i, 0, sizeof(CUCOMPLEX), &tmp);  // get on the real part
    value->x += tmp.x;
    value->y += tmp.y;
  }

  cuda_error_check();
}

/*
 * Integrate A^* X B (overlap).
 *
 */

__global__ void cgrid_cuda_integral_of_conjugate_product_gpu(CUCOMPLEX *a, CUCOMPLEX *b, CUCOMPLEX *blocks, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2, t;
  extern __shared__ CUCOMPLEX els[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els[t].x = els[t].y = 0.0;
  }
  __syncthreads();

  idx = (i * ny + j) * nz + k;
  idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
  els[idx2].x += a[idx].x * b[idx].x + a[idx].y * b[idx].y;  // A^* times B
  els[idx2].y += a[idx].x * b[idx].y - a[idx].y * b[idx].x;
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] = blocks[idx2] + els[t];  // reduce threads
    }
  }
}

/*
 * Integral of conjugate product (overlap).
 *
 * grid     = Source 1 for operation (gpu_mem_block *; input).
 * src      = Source 2 for operation (gpu_mem_block *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 * value    = Result (CUCOMPLEX *; output).
 *
 */

extern "C" void cgrid_cuda_integral_of_conjugate_productW(gpu_mem_block *grid, gpu_mem_block *src, INT nx, INT ny, INT nz, CUCOMPLEX *value) {

  SETUP_VARIABLES(grid);
  cudaXtDesc *GRID = grid->gpu_info->descriptor, *SRC = src->gpu_info->descriptor;
  CUCOMPLEX tmp;
  INT s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b31 = blocks1.x * blocks1.y * blocks1.z, b32 = blocks2.x * blocks2.y * blocks2.z;
  extern int cuda_get_element(void *, int, size_t, size_t, void *);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_block_init<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b31);
    // Blocks, Threads, dynamic memory size
    cgrid_cuda_integral_of_conjugate_product_gpu<<<blocks1,threads,s*sizeof(CUCOMPLEX)>>>((CUCOMPLEX *) GRID->data[i], (CUCOMPLEX *) SRC->data[i], (CUCOMPLEX *) grid_gpu_mem_addr->data[i], nnx1, nny1, nz);
    cgrid_cuda_block_reduce<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b31);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_block_init<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b32);
    // Blocks, Threads, dynamic memory size
    cgrid_cuda_integral_of_conjugate_product_gpu<<<blocks2,threads,s*sizeof(CUCOMPLEX)>>>((CUCOMPLEX *) GRID->data[i], (CUCOMPLEX *) SRC->data[i], (CUCOMPLEX *) grid_gpu_mem_addr->data[i], nnx2, nny2, nz);
    cgrid_cuda_block_reduce<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b32);
  }

  // Reduce over GPUs
  *value = CUMAKE(0.0,0.0);
  for(i = 0; i < ngpu2; i++) {
    cuda_get_element(grid_gpu_mem, i, 0, sizeof(CUCOMPLEX), &tmp);
    value->x += tmp.x;  /// + overloaded to device function - work around!
    value->y += tmp.y;
  }

  cuda_error_check();
}

/*
 * Integrate A * |B|^2.
 *
 */

__global__ void cgrid_cuda_grid_expectation_value_gpu(CUCOMPLEX *a, CUCOMPLEX *b, CUCOMPLEX *blocks, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, d = blockDim.x * blockDim.y * blockDim.z, idx, idx2, t;
  CUREAL tmp;
  extern __shared__ CUCOMPLEX els[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els[t].x = els[t].y = 0.0;
  }
  __syncthreads();

  idx = (i * ny + j) * nz + k;
  idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
  tmp = b[idx].x * b[idx].x + b[idx].y * b[idx].y;
  els[idx2] = els[idx2] + (a[idx] * tmp);
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] = blocks[idx2] + els[t];  // reduce threads
    }
  }
}

/*
 * Integral A * |B|^2.
 *
 * grida    = Source 1 (A) for operation (gpu_mem_block *; input).
 * gridb    = Source 2 (B) for operation (gpu_mem_block *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 * value    = Result (CUCOMPLEX *; output).
 *
 */

extern "C" void cgrid_cuda_grid_expectation_valueW(gpu_mem_block *grida, gpu_mem_block *gridb, INT nx, INT ny, INT nz, CUCOMPLEX *value) {

  SETUP_VARIABLES(grida);
  cudaXtDesc *GRIDA = grida->gpu_info->descriptor, *GRIDB = gridb->gpu_info->descriptor;
  CUCOMPLEX tmp;
  INT s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b31 = blocks1.x * blocks1.y * blocks1.z, b32 = blocks2.x * blocks2.y * blocks2.z;
  extern int cuda_get_element(void *, int, size_t, size_t, void *);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(GRIDA->GPUs[i]);
    cgrid_cuda_block_init<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b31);
    // Blocks, Threads, dynamic memory size
    cgrid_cuda_grid_expectation_value_gpu<<<blocks1,threads,s*sizeof(CUCOMPLEX)>>>((CUCOMPLEX *) GRIDA->data[i], (CUCOMPLEX *) GRIDB->data[i], (CUCOMPLEX *) grid_gpu_mem_addr->data[i], nnx1, nny1, nz);
    cgrid_cuda_block_reduce<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b31);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(GRIDA->GPUs[i]);
    cgrid_cuda_block_init<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b32);
    // Blocks, Threads, dynamic memory size
    cgrid_cuda_grid_expectation_value_gpu<<<blocks2,threads,s*sizeof(CUCOMPLEX)>>>((CUCOMPLEX *) GRIDA->data[i], (CUCOMPLEX *) GRIDB->data[i], (CUCOMPLEX *) grid_gpu_mem_addr->data[i], nnx2, nny2, nz);
    cgrid_cuda_block_reduce<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b32);
  }

  // Reduce over GPUs
  *value = CUMAKE(0.0,0.0);
  for(i = 0; i < ngpu2; i++) {
    cuda_get_element(grid_gpu_mem, i, 0, sizeof(CUCOMPLEX), &tmp);
    value->x += tmp.x;  /// + overloaded to device function - work around!
    value->y += tmp.y;
  }

  cuda_error_check();
}

/*
 * dst = FD_X(src).
 *
 */

__global__ void cgrid_cuda_fd_gradient_x_gpu(CUCOMPLEX *src, CUCOMPLEX *dst, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = inv_delta * (cgrid_cuda_bc_x_plus(src, bc, i, j, k, nx, ny, nz) - cgrid_cuda_bc_x_minus(src, bc, i, j, k, nx, ny, nz));
}

/*
 * dst = FD_X(src)
 *
 * src       = Source for operation (gpu_mem_block *; input).
 * dst       = Destination for operation (gpu_mem_block *; input).
 * inv_delta = 1 / (2 * step) (REAL; input).
 * bc        = Boundary condition: 0 = Dirichlet, 1 = Neumann, 2 = Periodic (char; input).
 * nx        = # of points along x (INT; input).
 * ny        = # of points along y (INT; input).
 * nz        = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fd_gradient_xW(gpu_mem_block *src, gpu_mem_block *dst, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  cudaXtDesc *SRC = src->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;
 
  if(DST->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(DST->GPUs[0]);
  cgrid_cuda_fd_gradient_x_gpu<<<blocks,threads>>>((CUCOMPLEX *) SRC->data[0], (CUCOMPLEX *) DST->data[0], inv_delta, bc, nx, ny, nz);

  cuda_error_check();
}

/*
 * dst = FD_Y(src).
 *
 */

__global__ void cgrid_cuda_fd_gradient_y_gpu(CUCOMPLEX *src, CUCOMPLEX *dst, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = inv_delta * (cgrid_cuda_bc_y_plus(src, bc, i, j, k, nx, ny, nz) - cgrid_cuda_bc_y_minus(src, bc, i, j, k, nx, ny, nz));
}

/*
 * B = FD_Y(A)
 *
 * src       = Source for operation (gpu_mem_block *; input).
 * dst       = Destination for operation (gpu_mem_block *; input).
 * inv_delta = 1 / (2 * step) (CUREAL; input).
 * bc        = Boundary condition: 0 = Dirichlet, 1 = Neumann, 2 = Periodic (char; input).
 * nx        = # of points along x (INT; input).
 * ny        = # of points along y (INT; input).
 * nz        = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fd_gradient_yW(gpu_mem_block *src, gpu_mem_block *dst, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  cudaXtDesc *SRC = src->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  if(DST->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(DST->GPUs[0]);
  cgrid_cuda_fd_gradient_y_gpu<<<blocks,threads>>>((CUCOMPLEX *) SRC->data[0], (CUCOMPLEX *) DST->data[0], inv_delta, bc, nx, ny, nz);

  cuda_error_check();
}

/*
 * dst = FD_Z(src).
 *
 */

__global__ void cgrid_cuda_fd_gradient_z_gpu(CUCOMPLEX *src, CUCOMPLEX *dst, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = inv_delta * (cgrid_cuda_bc_z_plus(src, bc, i, j, k, nx, ny, nz) - cgrid_cuda_bc_z_minus(src, bc, i, j, k, nx, ny, nz));
}

/*
 * B = FD_Z(A)
 *
 * src       = Source for operation (gpu_mem_block *; input).
 * dst       = Destination for operation (gpu_mem_block *; input).
 * inv_delta = 1 / (2 * step) (CUREAL; input).
 * bc        = Boundary condition: 0 = Dirichlet, 1 = Neumann, 2 = Periodic (char; input).
 * nx        = # of points along x (INT; input).
 * ny        = # of points along y (INT; input).
 * nz        = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fd_gradient_zW(gpu_mem_block *src, gpu_mem_block *dst, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  cudaXtDesc *SRC = src->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  if(DST->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(DST->GPUs[0]);
  cgrid_cuda_fd_gradient_z_gpu<<<blocks,threads>>>((CUCOMPLEX *) SRC->data[0], (CUCOMPLEX *) DST->data[0], inv_delta, bc, nx, ny, nz);

  cuda_error_check();
}

/*
 * dst = LAPLACE(src).
 *
 */

__global__ void cgrid_cuda_fd_laplace_gpu(CUCOMPLEX *src, CUCOMPLEX *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = inv_delta2 * (cgrid_cuda_bc_x_plus(src, bc, i, j, k, nx, ny, nz) + cgrid_cuda_bc_x_minus(src, bc, i, j, k, nx, ny, nz)
                         + cgrid_cuda_bc_y_plus(src, bc, i, j, k, nx, ny, nz) + cgrid_cuda_bc_y_minus(src, bc, i, j, k, nx, ny, nz)
                         + cgrid_cuda_bc_z_plus(src, bc, i, j, k, nx, ny, nz) + cgrid_cuda_bc_z_minus(src, bc, i, j, k, nx, ny, nz)
                         - 6.0 * src[idx]);
}

/*
 * B = LAPLACE(A)
 *
 * src        = Source for operation (gpu_mem_block *; input).
 * dst        = Destination for operation (gpu_mem_block *; output).
 * inv_delta2 = 1 / (2 * step) (CUREAL; input).
 * bc         = Boundary condition (char; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 * Returns laplacian in dst.
 *
 */

extern "C" void cgrid_cuda_fd_laplaceW(gpu_mem_block *src, gpu_mem_block *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  cudaXtDesc *SRC = src->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  if(DST->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(DST->GPUs[0]);
  cgrid_cuda_fd_laplace_gpu<<<blocks,threads>>>((CUCOMPLEX *) SRC->data[0], (CUCOMPLEX *) DST->data[0], inv_delta2, bc, nx, ny, nz);

  cuda_error_check();
}

/*
 * dst = LAPLACE_X(src).
 *
 */

__global__ void cgrid_cuda_fd_laplace_x_gpu(CUCOMPLEX *src, CUCOMPLEX *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = inv_delta2 * (cgrid_cuda_bc_x_plus(src, bc, i, j, k, nx, ny, nz) + cgrid_cuda_bc_x_minus(src, bc, i, j, k, nx, ny, nz) - 2.0 * src[idx]);
}

/*
 * dst = LAPLACE_X(src)
 *
 * src        = Source for operation (gpu_mem_block *; input).
 * dst        = Destination for operation (gpu_mem_block *; output).
 * inv_delta2 = 1 / (2 * step) (CUREAL; input).
 * bc         = Boundary condition (char; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fd_laplace_xW(gpu_mem_block *src, gpu_mem_block *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  cudaXtDesc *SRC = src->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  if(DST->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(DST->GPUs[0]);
  cgrid_cuda_fd_laplace_x_gpu<<<blocks,threads>>>((CUCOMPLEX *) SRC->data[0], (CUCOMPLEX *) DST->data[0], inv_delta2, bc, nx, ny, nz);

  cuda_error_check();
}

/*
 * dst = LAPLACE_Y(src).
 *
 */

__global__ void cgrid_cuda_fd_laplace_y_gpu(CUCOMPLEX *src, CUCOMPLEX *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = inv_delta2 * (cgrid_cuda_bc_y_plus(src, bc, i, j, k, nx, ny, nz) + cgrid_cuda_bc_y_minus(src, bc, i, j, k, nx, ny, nz) - 2.0 * src[idx]);
}

/*
 * B = LAPLACE_Y(A)
 *
 * src        = Source for operation (gpu_mem_block *; input).
 * dst        = Destination for operation (gpu_mem_block *; output).
 * inv_delta2 = 1 / (2 * step) (CUREAL; input).
 * bc         = Boundary condition (char; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fd_laplace_yW(gpu_mem_block *src, gpu_mem_block *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  cudaXtDesc *SRC = src->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  if(DST->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(DST->GPUs[0]);
  cgrid_cuda_fd_laplace_y_gpu<<<blocks,threads>>>((CUCOMPLEX *) SRC->data[0], (CUCOMPLEX *) DST->data[0], inv_delta2, bc, nx, ny, nz);

  cuda_error_check();
}

/*
 * dst = LAPLACE_Z(src).
 *
 */

__global__ void cgrid_cuda_fd_laplace_z_gpu(CUCOMPLEX *src, CUCOMPLEX *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = inv_delta2 * (cgrid_cuda_bc_z_plus(src, bc, i, j, k, nx, ny, nz) + cgrid_cuda_bc_z_minus(src, bc, i, j, k, nx, ny, nz) - 2.0 * src[idx]);
}

/*
 * dst = LAPLACE_Z(src)
 *
 * src        = Source for operation (gpu_mem_block *; input).
 * dst        = Destination for operation (gpu_mem_block *; output).
 * inv_delta2 = 1 / (2 * step) (CUREAL; input).
 * bc         = Boundary condition (char; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fd_laplace_zW(gpu_mem_block *src, gpu_mem_block *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  cudaXtDesc *SRC = src->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  if(DST->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(DST->GPUs[0]);
  cgrid_cuda_fd_laplace_z_gpu<<<blocks,threads>>>((CUCOMPLEX *) SRC->data[0], (CUCOMPLEX *) DST->data[0], inv_delta2, bc, nx, ny, nz);

  cuda_error_check();
}

/*
 * dst = FD_X(src)^2 + FD_Y(src)^2 + FD_Z(src)^2.
 *
 */

__global__ void cgrid_cuda_fd_gradient_dot_gradient_gpu(CUCOMPLEX *src, CUCOMPLEX *dst, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  CUCOMPLEX tmp;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;

  dst[idx] = CUMAKE(0.0, 0.0);

  tmp = inv_delta * (cgrid_cuda_bc_x_plus(src, bc, i, j, k, nx, ny, nz) - cgrid_cuda_bc_x_minus(src, bc, i, j, k, nx, ny, nz));
  dst[idx] = dst[idx] + CUCREAL(tmp) * CUCREAL(tmp) + CUCIMAG(tmp) * CUCIMAG(tmp);

  tmp = inv_delta * (cgrid_cuda_bc_y_plus(src, bc, i, j, k, nx, ny, nz) - cgrid_cuda_bc_y_minus(src, bc, i, j, k, nx, ny, nz));
  dst[idx] = dst[idx] + CUCREAL(tmp) * CUCREAL(tmp) + CUCIMAG(tmp) * CUCIMAG(tmp);

  tmp = inv_delta * (cgrid_cuda_bc_z_plus(src, bc, i, j, k, nx, ny, nz) - cgrid_cuda_bc_z_minus(src, bc, i, j, k, nx, ny, nz));
  dst[idx] = dst[idx] + CUCREAL(tmp) * CUCREAL(tmp) + CUCIMAG(tmp) * CUCIMAG(tmp);
}

/*
 * dst = FD_X(src)^2 + FD_Y(src)^2 + FD_Z(src)^2.
 *
 * src        = Source for operation (gpu_mem_block *; input).
 * dst        = Destination for operation (gpu_mem_block *; output).
 * inv_delta2 = 1 / (4 * step * step) (REAL; input).
 * bc         = Boundary condition (char; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fd_gradient_dot_gradientW(gpu_mem_block *src, gpu_mem_block *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  cudaXtDesc *SRC = src->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  if(DST->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(DST->GPUs[0]);
  cgrid_cuda_fd_gradient_dot_gradient_gpu<<<blocks,threads>>>((CUCOMPLEX *) SRC->data[0], (CUCOMPLEX *) DST->data[0], inv_delta2, bc, nx, ny, nz);

  cuda_error_check();
}

/*
 * Complex conjugate.
 *
 * dst = src*
 *
 */

__global__ void cgrid_cuda_conjugate_gpu(CUCOMPLEX *dst, CUCOMPLEX *src, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = CUCONJ(src[idx]);
}

/*
 * Grid conjugate.
 *
 * dst      = Destination for operation (gpu_mem_block *; output).
 * src      = Source for operation (gpu_mem_block *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_conjugateW(gpu_mem_block *dst, gpu_mem_block *src, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor, *SRC = src->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_conjugate_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) SRC->data[i], nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_conjugate_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) SRC->data[i], nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * FFT gradient_x
 *
 */

__global__ void cgrid_cuda_fft_gradient_x_gpu(CUCOMPLEX *b, CUREAL norm, CUREAL kx0, CUREAL step, INT nx, INT ny, INT nz, INT nx2) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  CUREAL kx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;

  if (i <= nx2)
    kx = 2.0 * M_PI * ((CUREAL) i) / (((CUREAL) nx) * step) - kx0;
  else 
    kx = 2.0 * M_PI * ((CUREAL) (i - nx)) / (((CUREAL) nx) * step) - kx0;

  b[idx] = b[idx] * kx * norm;     // multiply by I * kx * norm
}

/*
 * FFT gradient_x
 *
 * dst        = Source/destination grid for operation (gpu_mem_block *; input/output).
 * norm       = FFT norm (grid->fft_norm) (CUREAL; input).
 * kx0        = Momentum shift of origin along X (CUREAL; input).
 * step       = Spatial step length (CUREAL; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fft_gradient_xW(gpu_mem_block *dst, CUREAL norm, CUREAL kx0, CUREAL step, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor;
  INT nx2 = nx / 2;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_gradient_x_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], norm, kx0, step, nnx1, nny1, nz, nx2);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_gradient_x_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], norm, kx0, step, nnx2, nny2, nz, nx2);
  }

  cuda_error_check();
}

/*
 * FFT gradient_y
 *
 * B = B' in Fourier space.
 *
 */

__global__ void cgrid_cuda_fft_gradient_y_gpu(CUCOMPLEX *b, CUREAL norm, CUREAL ky0, CUREAL step, INT nx, INT ny, INT nz, INT ny2, INT seg, INT nyy) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, jj = j + seg;
  CUREAL ky;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;

  if (jj <= ny2)
    ky = 2.0 * M_PI * ((CUREAL) jj) / (((CUREAL) nyy) * step) - ky0;
  else 
    ky = 2.0 * M_PI * ((CUREAL) (jj - nyy)) / (((CUREAL) nyy) * step) - ky0;

  b[idx] = b[idx] * ky * norm;    // multiply by I * ky * norm
}

/*
 * FFT gradient_y
 *
 * dst        = Source/destination grid for operation (gpu_mem_block *; input/output).
 * norm       = FFT norm (grid->fft_norm) (CUREAL; input).
 * ky0        = Momentum shift of origin along Y (CUREAL; input).
 * step       = Spatial step length (CUREAL; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fft_gradient_yW(gpu_mem_block *dst, CUREAL norm, CUREAL ky0, CUREAL step, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_SEG(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor;
  INT ny2 = ny / 2, segx = 0, segy = 0;  // segx unused

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_gradient_y_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], norm, ky0, step, nnx1, nny1, nz, ny2, segy, ny);
    segx += dsegx1;
    segy += dsegy1;
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_gradient_y_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], norm, ky0, step, nnx2, nny2, nz, ny2, segy, ny);
    segx += dsegx2;
    segy += dsegy2;
  }

  cuda_error_check();
}

/*
 * FFT gradient_z
 *
 * B = B' in Fourier space.
 *
 */

__global__ void cgrid_cuda_fft_gradient_z_gpu(CUCOMPLEX *b, CUREAL norm, CUREAL kz0, CUREAL step, INT nx, INT ny, INT nz, INT nz2) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  CUREAL kz;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;

  if (k <= nz2)
    kz = 2.0 * M_PI * ((CUREAL) k) / (((CUREAL) nz) * step) - kz0;
  else 
    kz = 2.0 * M_PI * ((CUREAL) (k - nz)) / (((CUREAL) nz) * step) - kz0;

  b[idx] = b[idx] * kz * norm;   // multiply by I * kz * norm
}

/*
 * FFT gradient_z
 *
 * dst        = Source/destination grid for operation (gpu_mem_block *; input/output).
 * norm       = FFT norm (grid->fft_norm) (CUREAL; input).
 * kz0        = Momentum shift of origin along Z (CUREAL; input).
 * step       = Spatial step length (CUREAL; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fft_gradient_zW(gpu_mem_block *dst, CUREAL norm, CUREAL kz0, CUREAL step, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor;
  INT nz2 = nz / 2;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_gradient_z_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], norm, kz0, step, nnx1, nny1, nz, nz2);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_gradient_z_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], norm, kz0, step, nnx2, nny2, nz, nz2);
  }

  cuda_error_check();
}

/*
 * FFT laplace.
 *
 * B = B'' in Fourier space.
 *
 */

__global__ void cgrid_cuda_fft_laplace_gpu(CUCOMPLEX *b, CUREAL norm, CUREAL kx0, CUREAL ky0, CUREAL kz0, CUREAL step, INT nx, INT ny, INT nz, INT nx2, INT ny2, INT nz2, INT seg, INT nyy) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, jj = j + seg, idx;
  CUREAL kx, ky, kz;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  
  if (i <= nx2)
    kx = 2.0 * M_PI * ((CUREAL) i) / (((CUREAL) nx) * step) - kx0;
  else 
    kx = 2.0 * M_PI * ((CUREAL) (i - nx)) / (((CUREAL) nx) * step) - kx0;
      
  if (jj <= ny2)
    ky = 2.0 * M_PI * ((CUREAL) jj) / (((CUREAL) nyy) * step) - ky0;
  else 
    ky = 2.0 * M_PI * ((CUREAL) (jj - nyy)) / (((CUREAL) nyy) * step) - ky0;
      
  if (k <= nz2)
    kz = 2.0 * M_PI * ((CUREAL) k) / (((CUREAL) nz) * step) - kz0;
  else 
    kz = 2.0 * M_PI * ((CUREAL) (k - nz)) / (((CUREAL) nz) * step) - kz0;      

  b[idx] = b[idx] * (-(kx * kx + ky * ky + kz * kz) * norm);
}

/*
 * FFT laplace
 *
 * dst      = Source/destination grid for operation (gpu_mem_block *; input/output).
 * norm     = FFT norm (grid->fft_norm) (CUREAL; input).
 * kx0      = Momentum shift of origin along X (CUREAL; input).
 * ky0      = Momentum shift of origin along Y (CUREAL; input).
 * kz0      = Momentum shift of origin along Z (CUREAL; input).
 * step     = Spatial step length (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fft_laplaceW(gpu_mem_block *dst, CUREAL norm, CUREAL kx0, CUREAL ky0, CUREAL kz0, CUREAL step, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_SEG(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor;
  INT nx2 = nx / 2, ny2 = ny / 2, nz2 = nz / 2, segx = 0, segy = 0; // segx not used

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_laplace_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], norm, kx0, ky0, kz0, step, nnx1, nny1, nz, nx2, ny2, nz2, segy, ny);
    segx += dsegx1;
    segy += dsegy1;
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_laplace_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], norm, kx0, ky0, kz0, step, nnx2, nny2, nz, nx2, ny2, nz2, segy, ny);
    segx += dsegx2;
    segy += dsegy2;
  }

  cuda_error_check();
}

/*
 * FFT laplace expectation value.
 *
 * B = <B''> in Fourier space.
 *
 * Only periodic version implemented.
 *
 * Normalization done in cgrid-cuda.c
 *
 */

__global__ void cgrid_cuda_fft_laplace_expectation_value_gpu(CUCOMPLEX *b, CUCOMPLEX *blocks, CUREAL kx0, CUREAL ky0, CUREAL kz0, CUREAL step, INT nx, INT ny, INT nz, INT nx2, INT ny2, INT nz2, INT seg, INT nyy) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, jj = j + seg;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2, t;
  CUREAL kx, ky, kz;
  extern __shared__ CUREAL els2[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els2[t] = 0.0;
  }
  __syncthreads();

  idx = (i * ny + j) * nz + k;

  if (i <= nx2)
    kx = 2.0 * M_PI * ((REAL) i) / (((REAL) nx) * step) - kx0;
  else 
    kx = 2.0 * M_PI * ((REAL) (i - nx)) / (((REAL) nx) * step) - kx0;
      
  if (jj <= ny2)
    ky = 2.0 * M_PI * ((REAL) jj) / (((REAL) nyy) * step) - ky0;
  else 
    ky = 2.0 * M_PI * ((REAL) (jj - nyy)) / (((REAL) nyy) * step) - ky0;
      
  if (k <= nz2)
    kz = 2.0 * M_PI * ((REAL) k) / (((REAL) nz) * step) - kz0;
  else 
    kz = 2.0 * M_PI * ((REAL) (k - nz)) / (((REAL) nz) * step) - kz0;

  idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
  els2[idx2] -= (kx * kx + ky * ky + kz * kz) * (CUCREAL(b[idx]) * CUCREAL(b[idx]) + CUCIMAG(b[idx]) * CUCIMAG(b[idx]));
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2].x += els2[t];  // reduce threads
    }
  }
}

/*
 * FFT laplace expectation value
 *
 * dst      = Source/destination grid for operation (gpu_mem_block *; input/output).
 * kx0      = Momentum shift of origin along X (REAL; input).
 * ky0      = Momentum shift of origin along Y (REAL; input).
 * kz0      = Momentum shift of origin along Z (REAL; input).
 * step     = Spatial step length (REAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 * value    = Expectation value (REAL; output).
 *
 * Normalization done in cgrid-cuda.c
 *
 */

extern "C" void cgrid_cuda_fft_laplace_expectation_valueW(gpu_mem_block *dst, CUREAL kx0, CUREAL ky0, CUREAL kz0, CUREAL step, INT nx, INT ny, INT nz, CUCOMPLEX *value) {

  SETUP_VARIABLES_SEG(dst)
  cudaXtDesc *DST = dst->gpu_info->descriptor;
  CUCOMPLEX tmp;
  INT nx2 = nx / 2, ny2 = ny / 2, nz2 = nz / 2, segx = 0, segy = 0; // segx not used
  int s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b31 = blocks1.x * blocks1.y * blocks1.z, b32 = blocks2.x * blocks2.y * blocks2.z;
  extern int cuda_get_element(void *, int, size_t, size_t, void *);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_block_init<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b31);
    // Blocks, Threads, dynamic memory size
    cgrid_cuda_fft_laplace_expectation_value_gpu<<<blocks1,threads,s*sizeof(CUCOMPLEX)>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) grid_gpu_mem_addr->data[i], kx0, ky0, kz0, step, nnx1, nny1, nz, nx2, ny2, nz2, segy, ny);
    segx += dsegx1;
    segy += dsegy1;
    cgrid_cuda_block_reduce<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b31);
  }

  cuda_error_check();

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_block_init<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b32);
    // Blocks, Threads, dynamic memory size
    cgrid_cuda_fft_laplace_expectation_value_gpu<<<blocks2,threads,s*sizeof(CUCOMPLEX)>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) grid_gpu_mem_addr->data[i], kx0, ky0, kz0, step, nnx2, nny2, nz, nx2, ny2, nz2, segy, ny);
    segx += dsegx2;
    segy += dsegy2;
    cgrid_cuda_block_reduce<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b32);
  }

  // Reduce over GPUs
  *value = CUMAKE(0.0,0.0);
  for(i = 0; i < ngpu2; i++) {
    cuda_get_element(grid_gpu_mem, i, 0, sizeof(CUCOMPLEX), &tmp);
    value->x += tmp.x;  /// + overloaded to device function - work around!
    value->y += tmp.y;
  }

  cuda_error_check();
}

/*
 * Zero real part.
 *
 * A.re = 0
 *
 */

__global__ void cgrid_cuda_zero_re_gpu(CUCOMPLEX *a, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;

  a[idx].x = 0.0;
}

/*
 * Zero real part.
 *
 * grid     = Grid to be operated on (gpu_mem_block *; input/output).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_zero_reW(gpu_mem_block *grid, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(grid);
  cudaXtDesc *GRID = grid->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_zero_re_gpu<<<blocks1,threads>>>((CUCOMPLEX *) GRID->data[i], nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_zero_re_gpu<<<blocks2,threads>>>((CUCOMPLEX *) GRID->data[i], nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * Zero imaginary part.
 *
 * A.im = 0
 *
 */

__global__ void cgrid_cuda_zero_im_gpu(CUCOMPLEX *a, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;

  a[idx].y = 0.0;
}

/*
 * Zero imaginary part.
 *
 * grid     = Grid to be operated on (gpu_mem_block *; input/output).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_zero_imW(gpu_mem_block *grid, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(grid);
  cudaXtDesc *GRID = grid->gpu_info->descriptor;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_zero_im_gpu<<<blocks1,threads>>>((CUCOMPLEX *) GRID->data[i], nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_zero_im_gpu<<<blocks2,threads>>>((CUCOMPLEX *) GRID->data[i], nnx2, nny2, nz);
  }

  cuda_error_check();
}

/*
 * Zero part of complex grid.
 *
 * A = 0 in the specified range.
 *
 */

__global__ void cgrid_cuda_zero_index_gpu(CUCOMPLEX *a, INT lx, INT hx, INT ly, INT hy, INT lz, INT hz, INT nx, INT ny, INT nz, INT segx, INT segy) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, ii = i + segx, jj = j + segy, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;

  if(ii >= lx && ii < hx && jj >= ly && jj < hy && k >= lz && k < hz) a[idx] = CUMAKE(0.0, 0.0);
}

/*
 * Zero specified range of complex grid.
 *
 * grid     = Grid to be operated on (gpu_mem_block *; input/output).
 * lx       = Low x index (INT; input).
 * hx       = High x index (INT; input).
 * ly       = Low y index (INT; input).
 * hy       = High y index (INT; input).
 * lz       = Low z index (INT; input).
 * hz       = High z index (INT; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_zero_indexW(gpu_mem_block *grid, INT lx, INT hx, INT ly, INT hy, INT lz, INT hz, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_SEG(grid);
  cudaXtDesc *GRID = grid->gpu_info->descriptor;
  INT segx = 0, segy = 0;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_zero_index_gpu<<<blocks1,threads>>>((CUCOMPLEX *) GRID->data[i], lx, hx, ly, hy, lz, hz, nnx1, nny1, nz, segx, segy);
    segx += dsegx1;
    segy += dsegy1;
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_zero_index_gpu<<<blocks2,threads>>>((CUCOMPLEX *) GRID->data[i], lx, hx, ly, hy, lz, hz, nnx2, nny2, nz, segx, segy);
    segx += dsegx2;
    segy += dsegy2;
  }

  cuda_error_check();
}

/*
 * Poisson equation.
 *
 */

__global__ void cgrid_cuda_poisson_gpu(CUCOMPLEX *grid, CUREAL norm, CUREAL step2, CUREAL ilx, CUREAL ily, CUREAL ilz, INT nx, INT ny, INT nz, INT seg) {
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, jj = j + seg, idx;
  CUREAL kx, ky, kz;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  kx = COS(ilx * (CUREAL) i);
  ky = COS(ily * (CUREAL) jj);
  kz = COS(ilz * (CUREAL) k);
  if(i || jj || k)
    grid[idx] = grid[idx] * norm * step2 / (2.0 * (kx + ky + kz - 3.0));
  else
    grid[idx] = CUMAKE(0.0, 0.0);
}

/*
 * Solve Poisson.
 *
 * grid    = Grid specifying the RHS (gpu_mem_block *; input/output).
 * norm    = FFT normalization constant (CUREAL; input).
 * step2   = Spatial step ^ 2 (CUREAL; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_poissonW(gpu_mem_block *grid, CUREAL norm, CUREAL step2, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_SEG(grid);
  cudaXtDesc *GRID = grid->gpu_info->descriptor;
  
  CUREAL ilx = 2.0 * M_PI / ((CUREAL) nx), ily = 2.0 * M_PI / ((CUREAL) ny), ilz = 2.0 * M_PI / ((CUREAL) nz);
  INT segx = 0, segy = 0; // segx not used

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_poisson_gpu<<<blocks1,threads>>>((CUCOMPLEX *) GRID->data[i], norm, step2, ilx, ily, ilz, nnx1, nny1, nz, segy);
    segx += dsegx1;
    segy += dsegy1;
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(GRID->GPUs[i]);
    cgrid_cuda_poisson_gpu<<<blocks2,threads>>>((CUCOMPLEX *) GRID->data[i], norm, step2, ilx, ily, ilz, nnx2, nny2, nz, segy);
    segx += dsegx2;
    segy += dsegy2;
  }

  cuda_error_check();
}
