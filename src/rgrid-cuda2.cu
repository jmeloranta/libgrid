#include "hip/hip_runtime.h"
/*
 * CUDA device code (REAL; rgrid).
 *
 * blockDim = # of threads
 * gridDim = # of blocks
 *
 */

#include <cuda/hip/hip_runtime_api.h>
#include <cuda/hip/hip_runtime.h>
#include <cuda/>
#include <cuda/hipfft/hipfft.h>
#include "cuda-math.h"
#include "rgrid_bc-cuda.h"

extern cudaXtState *grid_gpu_mem_addr;
extern "C" void cuda_error_check();

/*
 *
 * dst = src1 * src2 but with alternating signs for FFT.
 *
 * Fourier space.
 *
 */

__global__ void rgrid_cuda_fft_convolute_gpu(CUCOMPLEX *dst, CUCOMPLEX *src1, CUCOMPLEX *src2, CUREAL norm, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  if((i + j + k) & 1) norm *= -1.0;
  dst[idx] = norm * src1[idx] * src2[idx];
}

/*
 * Convolution in the Fourier space (data in GPU). Not called directly.
 *
 * Multiplication in GPU memory: grid_gpu_mem[i] = grid_gpu_mem[i] * grid_gpu_mem[j] (with sign variation).
 * Note: this includes the sign variation needed for convolution as well as normalization!
 *
 * dst   = output (cudaXtState *; output).
 * src1  = 1st grid to be convoluted (cudaXtState *; input).
 * src2  = 2nd grid to be convoluted (cudaXtState *; input).
 * norm  = FFT norm (CUREAL; input).
 * nx    = Grid dim x (INT; input).
 * ny    = Grid dim y (INT; input).
 * nz    = Grid dim z (INT; input).
 *
 * In Fourier space.
 *
 */

extern "C" void rgrid_cuda_fft_convoluteW(cudaXtState *dst, cudaXtState *src1, cudaXtState *src2, CUREAL norm, INT nx, INT ny, INT nz) {

  INT i, ngpu2 = dst->nGPUs, ngpu1 = ny % gpu2, nny2 = ny / ngpu2, nny1 = nny2 + 1, nzz = nz / 2 + 1;
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks1((nzz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Full set of indices
              (nny1 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  dim3 blocks2((nzz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Partial set
              (nny2 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  for(i = 0; i < ngpu1; i++) { // Full sets 
    CudaSetDevice(dst->GPUs[i]);
    rgrid_cuda_fft_convolute_gpu<<<blocks1,threads>>>((CUCOMPLEX *) dst->data[i], (CUCOMPLEX *) src1->data[i], (CUCOMPLEX *) src2->data[i], norm, nx, nny1, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) { // Partial sets
    CudaSetDevice(dst->GPUs[i]);
    rgrid_cuda_fft_convolute_gpu<<<blocks2,threads>>>((CUCOMPLEX *) dst->data[i], (CUCOMPLEX *) src1->data[i], (CUCOMPLES *) src2->data[i], norm, nx, nny2, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = src^x
 *
 */

__global__ void rgrid_cuda_power_gpu(CUREAL *dst, CUREAL *src, CUREAL x, INT nx, INT ny, INT nz, INT nzz) {
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = POW(src[idx], x);
}

/*
 * Grid power.
 *
 * dst      = Destination for operation (cudaXtState *; output).
 * src      = Source for operation (cudaXtState *; input).
 * exponent = Exponent (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 * Real space.
 *
 */

extern "C" void rgrid_cuda_powerW(cudaXtState *dst, cudaXtState *src, CUREAL exponent, INT nx, INT ny, INT nz) {

  INT i, ngpu2 = dst->nGPUs, ngpu1 = nx % ngpus, nnx2 = nx / ngpu2, nnx1 = nnx2 + 1, nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks1((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Full set of indices
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx1 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  dim3 blocks2((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Partial set
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx2 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  for(i = 0; i < ngpu1; i++) { // Full sets
    CudaSetDevice(dst->GPUs[i]);
    rgrid_cuda_power_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src->data[i], exponent, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) { // Partial sets
    CudaSetDevice(dst->GPUs[i]);
    cgrid_cuda_power_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src->data[i], exponent, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = |src|^x
 *
 */

__global__ void rgrid_cuda_abs_power_gpu(CUREAL *dst, CUREAL *src, CUREAL x, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = POW(FABS(src[idx]), x);
}

/*
 * Grid abs power.
 *
 * dst      = Destination for operation (cudaXtState *; output).
 * src      = Source for operation (cudaXtState *; input).
 * exponent = Exponent (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_abs_powerW(cudaXtState *dst, cudaXtState *src, CUREAL exponent, INT nx, INT ny, INT nz) {

  INT i, ngpu2 = dst->nGPUs, ngpu1 = nx % ngpus, nnx2 = nx / ngpu2, nnx1 = nnx2 + 1, nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks1((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Full set of indices
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx1 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  dim3 blocks2((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Partial set
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx2 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  for(i = 0; i < ngpu1; i++) { // Full sets
    CudaSetDevice(dst->GPUs[i]);
    rgrid_cuda_abs_power_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src->data[i], exponent, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) { // Partial sets
    CudaSetDevice(dst->GPUs[i]);
    cgrid_cuda_abs_power_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src->data[i], exponent, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = c * dst
 *
 */

__global__ void rgrid_cuda_multiply_gpu(CUREAL *dst, CUREAL c, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;

  if(i >= nx || j >= ny || k >= nz) return;

  dst[(i * ny + j) * nzz + k] *= c;
}

/*
 * Multiply grid by a constant.
 *
 * dst      = Grid to be operated on (cudaXtState *; input/output).
 * c        = Multiplying constant (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_multiplyW(cudaXtState *grid, CUREAL c, INT nx, INT ny, INT nz) {

  INT i, ngpu2 = dst->nGPUs, ngpu1 = nx % ngpus, nnx2 = nx / ngpu2, nnx1 = nnx2 + 1, nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks1((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Full set of indices
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx1 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  dim3 blocks2((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Partial set
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx2 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  for(i = 0; i < ngpu1; i++) { // Full sets
    CudaSetDevice(dst->GPUs[i]);
    rgrid_cuda_multiply_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], c, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) { // Partial sets
    CudaSetDevice(dst->GPUs[i]);
    cgrid_cuda_multiply_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], c, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = c * dst (in Fourier space)
 *
 */

__global__ void rgrid_cuda_multiply_fft_gpu(CUCOMPLEX *dst, CUREAL c, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;

  dst[idx] = dst[idx] * c;
}

/*
 * Multiply (complex) grid by a constant (in FFT space).
 *
 * st       = Grid to be operated on (cudaXtState *; input/output).
 * c        = Multiplying constant (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_multiply_fftW(CUCOMPLEX *grid, CUREAL c, INT nx, INT ny, INT nz) {

  INT i, ngpu2 = dst->nGPUs, ngpu1 = ny % gpu2, nny2 = ny / ngpu2, nny1 = nny2 + 1, nzz = nz / 2 + 1;
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks1((nzz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Full set of indices
              (nny1 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  dim3 blocks2((nzz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Partial set
              (nny2 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  for(i = 0; i < ngpu1; i++) { // Full sets 
    CudaSetDevice(dst->GPUs[i]);
    rgrid_cuda_multiply_fft_gpu<<<blocks1,threads>>>((CUCOMPLEX *) dst->data[i], c, nx, nny1, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) { // Partial sets
    CudaSetDevice(dst->GPUs[i]);
    rgrid_cuda_multiply_fft_gpu<<<blocks2,threads>>>((CUCOMPLEX *) dst->data[i], c, nx, nny2, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = src1 + src2
 *
 */

__global__ void rgrid_cuda_sum_gpu(CUREAL *dst, CUREAL *src1, CUREAL *src2, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = src1[idx] + src2[idx];
}

/*
 * Sum of two grids.
 *
 * dst      = Destination grid (cudaXtState *; output).
 * src1     = Input grid 1 (cudaXtState *; input).
 * src2     = Input grid 2 (cudaXtState *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_sumW(cudaXtState *grida, cudaXtState *gridb, cudaXtState *gridc, INT nx, INT ny, INT nz) {

  INT i, ngpu2 = dst->nGPUs, ngpu1 = nx % ngpus, nnx2 = nx / ngpu2, nnx1 = nnx2 + 1, nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks1((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Full set of indices
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx1 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  dim3 blocks2((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Partial set
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx2 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  for(i = 0; i < ngpu1; i++) { // Full sets
    CudaSetDevice(dst->GPUs[i]);
    rgrid_cuda_sum_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) { // Partial sets
    CudaSetDevice(dst->GPUs[i]);
    cgrid_cuda_sum_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = src1 - src2
 *
 */

__global__ void rgrid_cuda_difference_gpu(CUREAL *dst, CUREAL *src1, CUREAL *src2, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = src1[idx] - src2[idx];
}

/*
 * Subtract two grids.
 *
 * dst      = Destination grid (cudaXtState *; output).
 * src1     = Input grid 1 (cudaXtState *; input).
 * src2     = Input grid 2 (cudaXtState *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_differenceW(cudaXtState *dst, cudaXtState *src1, cudaXtState *src2, INT nx, INT ny, INT nz) {

  INT i, ngpu2 = dst->nGPUs, ngpu1 = nx % ngpus, nnx2 = nx / ngpu2, nnx1 = nnx2 + 1, nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks1((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Full set of indices
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx1 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  dim3 blocks2((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Partial set
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx2 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  for(i = 0; i < ngpu1; i++) { // Full sets
    CudaSetDevice(dst->GPUs[i]);
    rgrid_cuda_difference_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) { // Partial sets
    CudaSetDevice(dst->GPUs[i]);
    cgrid_cuda_difference_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = src1 * src2.
 *
 */

__global__ void rgrid_cuda_product_gpu(CUREAL *dst, CUREAL *src1, CUREAL *src2, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = src1[idx] * src2[idx];
}

/*
 * Product of two grids.
 *
 * dst      = Destination grid (cudaXtState *; output).
 * src1     = Source grid 1 (cudaXtState *; input).
 * src2     = Source grid 2 (cudaXtState *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_productW(cudaXtState *dst, cudaXtState *src1, cudaXtState *src2, INT nx, INT ny, INT nz) {

  INT i, ngpu2 = dst->nGPUs, ngpu1 = nx % ngpus, nnx2 = nx / ngpu2, nnx1 = nnx2 + 1, nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks1((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Full set of indices
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx1 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  dim3 blocks2((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Partial set
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx2 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  for(i = 0; i < ngpu1; i++) { // Full sets
    CudaSetDevice(dst->GPUs[i]);
    rgrid_cuda_product_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) { // Partial sets
    CudaSetDevice(dst->GPUs[i]);
    cgrid_cuda_product_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = src1 / src2.
 *
 */

__global__ void rgrid_cuda_division_gpu(CUREAL *dst, CUREAL *src1, CUREAL *src2, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = src1[idx] / src2[idx];
}

/*
 * Division of two grids.
 *
 * dst      = Destination grid (cudaXtState *; output).
 * src1     = Source grid 1 (cudaXtState *; input).
 * src2     = Source grid 2 (cudaXtState *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_divisionW(cudaXtState *dst, cudaXtState *src1, cudaXtState *src2, INT nx, INT ny, INT nz) {

  INT i, ngpu2 = dst->nGPUs, ngpu1 = nx % ngpus, nnx2 = nx / ngpu2, nnx1 = nnx2 + 1, nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks1((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Full set of indices
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx1 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  dim3 blocks2((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Partial set
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx2 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  for(i = 0; i < ngpu1; i++) { // Full sets
    CudaSetDevice(dst->GPUs[i]);
    rgrid_cuda_division_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) { // Partial sets
    CudaSetDevice(dst->GPUs[i]);
    cgrid_cuda_division_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = src1 / (src2 + eps).
 *
 */

__global__ void rgrid_cuda_division_eps_gpu(CUREAL *dst, CUREAL *src1, CUREAL *src2, CUREAL eps, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = src1[idx] / (src2[idx] + eps);
}

/*
 * Division of two grids.
 *
 * dst      = Destination grid (cudaXtState *; output).
 * src1     = Source grid 1 (cudaXtState *; input).
 * src2     = Source grid 2 (cudaXtState *; input).
 * eps      = Epsilon (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_division_epsW(cudaXtState *dst, cudaXtState *src1, cudaXtState *src2, CUREAL eps, INT nx, INT ny, INT nz) {

  INT i, ngpu2 = dst->nGPUs, ngpu1 = nx % ngpus, nnx2 = nx / ngpu2, nnx1 = nnx2 + 1, nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks1((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Full set of indices
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx1 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  dim3 blocks2((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Partial set
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx2 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  for(i = 0; i < ngpu1; i++) { // Full sets
    CudaSetDevice(dst->GPUs[i]);
    rgrid_cuda_division_eps_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], eps, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) { // Partial sets
    CudaSetDevice(dst->GPUs[i]);
    cgrid_cuda_division_eps_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], eps, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = dst + c
 *
 */

__global__ void rgrid_cuda_add_gpu(CUREAL *dst, CUREAL c, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] += c;
}

/*
 * Add constant to grid.
 *
 * dst      = Grid to be operated on (cudaXtState *; input/output).
 * c        = Constant (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_addW(cudaXtState *grid, CUREAL c, INT nx, INT ny, INT nz) {

  INT i, ngpu2 = dst->nGPUs, ngpu1 = nx % ngpus, nnx2 = nx / ngpu2, nnx1 = nnx2 + 1, nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks1((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Full set of indices
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx1 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  dim3 blocks2((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Partial set
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx2 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  for(i = 0; i < ngpu1; i++) { // Full sets
    CudaSetDevice(dst->GPUs[i]);
    rgrid_cuda_add_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], c, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) { // Partial sets
    CudaSetDevice(dst->GPUs[i]);
    cgrid_cuda_add_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], c, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = cm * dst + ca
 *
 */

__global__ void rgrid_cuda_multiply_and_add_gpu(CUREAL *dst, CUREAL cm, CUREAL ca, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = dst[idx] * cm + ca;
}

/*
 * Grid multiply and add.
 *
 * dst      = Grid to be operated on (cudaXtState *; input/output).
 * cm       = Multiplier (CUREAL; input).
 * ca       = Additive constant (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_multiply_and_addW(cudaXtState *grid, CUREAL cm, REAL ca, INT nx, INT ny, INT nz) {

  INT i, ngpu2 = dst->nGPUs, ngpu1 = nx % ngpus, nnx2 = nx / ngpu2, nnx1 = nnx2 + 1, nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks1((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Full set of indices
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx1 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  dim3 blocks2((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Partial set
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx2 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  for(i = 0; i < ngpu1; i++) { // Full sets
    CudaSetDevice(dst->GPUs[i]);
    rgrid_cuda_multiply_and_add_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], cm, ca, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) { // Partial sets
    CudaSetDevice(dst->GPUs[i]);
    cgrid_cuda_multiply_and_add_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], cm, ca, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = cm * (dst + ca)
 *
 */

__global__ void rgrid_cuda_add_and_multiply_gpu(CUREAL *dst, CUREAL ca, CUREAL cm, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = (dst[idx] + ca) * cm;
}

/*
 * Grid multiply and add.
 *
 * dst      = Grid to be operated on (cudaXtState *; input/output).
 * cm       = Multiplier (CUREAL; input).
 * ca       = Additive constant (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_add_and_multiplyW(cudaXtState *dst, CUREAL ca, CUREAL cm, INT nx, INT ny, INT nz) {

  INT i, ngpu2 = dst->nGPUs, ngpu1 = nx % ngpus, nnx2 = nx / ngpu2, nnx1 = nnx2 + 1, nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks1((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Full set of indices
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx1 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  dim3 blocks2((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Partial set
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx2 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  for(i = 0; i < ngpu1; i++) { // Full sets
    CudaSetDevice(dst->GPUs[i]);
    rgrid_cuda_add_and_multiply_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], ca, cm, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) { // Partial sets
    CudaSetDevice(dst->GPUs[i]);
    cgrid_cuda_add_and_multiply_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], ca, cm, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = dst + d * src
 *
 */

__global__ void rgrid_cuda_add_scaled_gpu(CUREAL *dst, CUREAL d, CUREAL *src, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = dst[idx] + src[idx] * d;
}

/*
 * Scaled add grid.
 *
 * dst      = Destination for operation (cudaXtState *; output).
 * d        = Scaling factor (REAL; input).
 * src      = Source for operation (cudaXtState *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_add_scaledW(cudaXtState *dst, CUREAL d, cudaXtState *src, INT nx, INT ny, INT nz) {

  INT i, ngpu2 = dst->nGPUs, ngpu1 = nx % ngpus, nnx2 = nx / ngpu2, nnx1 = nnx2 + 1, nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks1((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Full set of indices
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx1 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  dim3 blocks2((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Partial set
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx2 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  for(i = 0; i < ngpu1; i++) { // Full sets
    CudaSetDevice(dst->GPUs[i]);
    rgrid_cuda_add_scaled_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], d, (CUREAL *) src->data[i], nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) { // Partial sets
    CudaSetDevice(dst->GPUs[i]);
    cgrid_cuda_add_scaled_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], d, (CUREAL *) src->data[i], nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = dst + d * src1 * src2
 *
 */

__global__ void rgrid_cuda_add_scaled_product_gpu(CUREAL *dst, CUREAL d, CUREAL *src1, CUREAL *src2, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = dst[idx] + d * src1[idx] * src2[idx];
}

/*
 * Add scaled product.
 *
 * dst      = Destination for operation (cudaXtState *; output).
 * d        = Scaling factor (REAL; input).
 * src1     = Source for operation (cudaXtState *; input).
 * src2     = Source for operation (cudaXtState *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_add_scaled_productW(cudaXtState *dst, CUREAL d, cudaXtState *src1, cudaXtState *src2, INT nx, INT ny, INT nz) {

  INT i, ngpu2 = dst->nGPUs, ngpu1 = nx % ngpus, nnx2 = nx / ngpu2, nnx1 = nnx2 + 1, nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks1((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Full set of indices
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx1 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  dim3 blocks2((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Partial set
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx2 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  for(i = 0; i < ngpu1; i++) { // Full sets
    CudaSetDevice(dst->GPUs[i]);
    rgrid_cuda_add_scaled_product_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], d, (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) { // Partial sets
    CudaSetDevice(dst->GPUs[i]);
    cgrid_cuda_add_scaled_product_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], d, (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = c
 *
 */

__global__ void rgrid_cuda_constant_gpu(CUREAL *dst, CUREAL c, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = c;
}

/*
 * Set grid to constant.
 *
 * dst      = Destination for operation (cudaXtState *; output).
 * c        = Constant (REAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_constantW(cudaXtState *dst, CUREAL c, INT nx, INT ny, INT nz) {

  INT i, ngpu2 = dst->nGPUs, ngpu1 = nx % ngpus, nnx2 = nx / ngpu2, nnx1 = nnx2 + 1, nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks1((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Full set of indices
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx1 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  dim3 blocks2((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Partial set
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx2 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  for(i = 0; i < ngpu1; i++) { // Full sets
    CudaSetDevice(dst->GPUs[i]);
    rgrid_cuda_constant_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], c, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) { // Partial sets
    CudaSetDevice(dst->GPUs[i]);
    cgrid_cuda_constant_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], c, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 * Block init (zero elements).
 *
 * blocks  = Block table (CUREAL *; output).
 * nblocks = Number of blocks in table (INT; input).
 * 
 */

__global__ void rgrid_cuda_block_init(CUREAL *blocks, INT nblocks) {

  INT i;

  for(i = 0; i < nblocks; i++) blocks[i] = 0.0;
}

/*
 * Block reduction.
 *
 * blocks  = Block list to reduce (CUREAL *; input/output). blocks[0] will contain the reduced value.
 * nblocks = Number of blocks (INT; input).
 *
 */

__global__ void rgrid_cuda_block_reduce(CUREAL *blocks, INT nblocks) {

  INT i;

  for(i = 1; i < nblocks; i++)
    blocks[0] += blocks[i];  // reduce blocks
}

/*
 * Integrate over A.
 *
 */

__global__ void rgrid_cuda_integral_gpu(CUREAL *a, CUREAL *blocks, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2, t;
  extern __shared__ CUREAL els[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els[t] = 0.0;
  }
  __syncthreads();

  idx = (i * ny + j) * nzz + k;
  idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;

  els[idx2] += a[idx];
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] += els[t];  // reduce threads
    }
  }
}

/// LEFT HERE - problems with with block reduce (1st overwritten)

/*
 * Integrate over grid.
 *
 * grid     = Source for operation (cudaXtState *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 * value    = Result (CUCOMPLEX *; output).
 *
 */

extern "C" void rgrid_cuda_integralW(cudaXtState *grid, INT nx, INT ny, INT nz, CUREAL *value) {

  CUREAL tmp;
  INT i, ngpu2 = grid->nGPUs, ngpu1 = nx % ngpus, nnx2 = nx / ngpu2, nnx1 = nnx2 + 1;
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks1((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Full set of indices
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx1 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  dim3 blocks2((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Partial set
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx2 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  INT s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b31 = blocks1.x * blocks1.y * blocks1.z, b32 = blocks2.x * blocks2.y * blocks2.z;

  for(i = 0; i < ngpu1; i++) { // Full sets
    CudaSetDevice(grid->GPUs[i]);
    cgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b31);
    cuda_error_check();
    // Blocks, Threads, dynamic memory size
    cgrid_cuda_integral_gpu<<<blocks1,threads,s*sizeof(CUREAL)>>>(CUREAL *) grid->data[i], (CUREAL *) grid_gpu_mem_addr->data[i], nnx1, ny, nz);
    cuda_error_check();
    cgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b31);
    cuda_error_check();
  }

  // Reduce over GPUs
  *value = 0.0;
  for(i = 0; i < ngpu1; i++) {
    cuda_get_element(grid_gpu_mem, grid->GPUs[i], 0, sizeof(CUREAL), &tmp);
    *value = *value + tmp;
  }

  for(i = ngpu1; i < ngpu2; i++) { // Partial sets
    CudaSetDevice(grid->GPUs[i]);
    cgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b32);
    cuda_error_check();
    // Blocks, Threads, dynamic memory size
    cgrid_cuda_integral_gpu<<<blocks1,threads,s*sizeof(CUREAL)>>>(CUREAL *) grid->data[i], (CUREAL *) grid_gpu_mem_addr->data[i], nnx2, ny, nz);
    cuda_error_check();
    cgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b32);
    cuda_error_check();
  }

  // Reduce over GPUs
  for(i = ngpu1; i < ngpu2; i++) {
    cuda_get_element(grid_gpu_mem, grid->GPUs[i], 0, sizeof(CUREAL), &tmp);
    *value = *value + tmp;
  }
}

/*
 * Integrate over A with limits.
 *
 */

__global__ void rgrid_cuda_integral_region_gpu(CUREAL *a, CUREAL *blocks, INT il, INT iu, INT jl, INT ju, INT kl, INT ku, INT nx, INT ny, INT nz, INT nzz, INT seg) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, ii = i + seg;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2, t;
  extern __shared__ CUREAL els[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els[t] = 0.0;
  }
  __syncthreads();

  if(ii >= il && ii <= iu && j >= jl && j <= ju && k >= kl && k <= ku) {
    idx = (i * ny + j) * nzz + k;
    idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
    els[idx2] += a[idx];
  }
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] += els[t];  // reduce threads
    }
  }
}

/*
 * Integrate over grid with limits.
 *
 * grid     = Source for operation (cudaXtState *; input).
 * il       = Lower index for x (INT; input).
 * iu       = Upper index for x (INT; input).
 * jl       = Lower index for y (INT; input).
 * ju       = Upper index for y (INT; input).
 * kl       = Lower index for z (INT; input).
 * ku       = Upper index for z (INT; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 * value    = Result (CUCOMPLEX *; output).
 *
 * Returns the value of integral in grid_gpu_mem[0].
 *
 */

extern "C" void rgrid_cuda_integral_regionW(cudaXtState *grid, INT il, INT iu, INT jl, INT ju, INT kl, INT ku, INT nx, INT ny, INT nz, CUREAL *value) {

  CUREAL tmp;
  INT i, ngpu2 = grid->nGPUs, ngpu1 = nx % ngpus, nnx2 = nx / ngpu2, nnx1 = nnx2 + 1, seg = 0;
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks1((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Full set of indices
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx1 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  dim3 blocks2((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,   // Partial set
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nnx2 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  INT s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b31 = blocks1.x * blocks1.y * blocks1.z, b32 = blocks2.x * blocks2.y * blocks2.z;

  if(il < 0) il = 0;  
  if(jl < 0) jl = 0;  
  if(kl < 0) kl = 0;  
  if(iu > nx-1) iu = nx-1;
  if(ju > ny-1) ju = ny-1;
  if(ku > nz-1) ku = nz-1;

  for(i = 0; i < ngpu1; i++) { // Full sets
    CudaSetDevice(grid->GPUs[i]);
    cgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b31);
    cuda_error_check();
    // Blocks, Threads, dynamic memory size
    rgrid_cuda_integral_region_gpu<<<blocks1,threads,s*sizeof(CUREAL)>>>(CUREAL *) grid->data[i], (CUREAL *) grid_gpu_mem_addr->data[i], 
                                    il, iu, jl, ju, kl, ku, nnx1, ny, nz, seg);
    seg -= nnx1;
    cuda_error_check();
    cgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b31);
    cuda_error_check();
  }


  // Reduce over GPUs
  *value = 0.0;
  for(i = 0; i < ngpu1; i++) {
    cuda_get_element(grid_gpu_mem, grid->GPUs[i], 0, sizeof(CUREAL), &tmp);
    *value = *value + tmp;
  }

  for(i = ngpu1; i < ngpu2; i++) { // Partial sets
    CudaSetDevice(grid->GPUs[i]);
    cgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b32);
    cuda_error_check();
    // Blocks, Threads, dynamic memory size
    cgrid_cuda_integral_region_gpu<<<blocks1,threads,s*sizeof(CUREAL)>>>(CUREAL *) grid->data[i], (CUREAL *) grid_gpu_mem_addr->data[i], 
                                    il, iu, jl, ju, kl, ku, nnx2, ny, nz, seg);
    seg -= nnx2;
    cuda_error_check();
    cgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b32);
    cuda_error_check();
  }

  // Reduce over GPUs
  for(i = ngpu1; i < ngpu2; i++) {
    cuda_get_element(grid_gpu_mem, grid->GPUs[i], 0, sizeof(CUREAL), &tmp);
    *value = *value + tmp;
  }
}

//// LEFT HERE

/*
 * Integrate of A^2.
 *
 */

__global__ void rgrid_cuda_integral_of_square_gpu(CUREAL *a, CUREAL *blocks, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2, t;
  extern __shared__ CUREAL els[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els[t] = 0.0;
  }
  __syncthreads();

  idx = (i * ny + j) * nzz + k;
  idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;

  els[idx2] += a[idx] * a[idx];
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] += els[t];  // reduce threads
    }
  }
}

/*
 * Integral of square.
 *
 * grid     = Source for operation (REAL *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 * Returns the value of integral in grid_gpu_mem[0].
 *
 */

extern "C" void rgrid_cuda_integral_of_squareW(CUREAL *grid, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  int s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b3 = blocks.x * blocks.y * blocks.z;

  rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr, b3);
  cuda_error_check();
  rgrid_cuda_integral_of_square_gpu<<<blocks,threads,s*sizeof(REAL)>>>(grid, (CUREAL *) grid_gpu_mem_addr, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
  rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr, b3);
  cuda_error_check();
}

/*
 * Integrate A * B.
 *
 */

__global__ void rgrid_cuda_integral_of_product_gpu(CUREAL *a, CUREAL *b, CUREAL *blocks, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2, t;
  extern __shared__ CUREAL els[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els[t] = 0.0;
  }
  __syncthreads();

  idx = (i * ny + j) * nzz + k;
  idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;

  els[idx2] += a[idx] * b[idx];
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] += els[t];  // reduce threads
    }
  }
}

/*
 * Integral of product.
 *
 * grid1    = Source 1 for operation (REAL *; input).
 * grid2    = Source 2 for operation (REAL *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 * Returns the value of integral in grid_gpu_mem[0].
 *
 */

extern "C" void rgrid_cuda_integral_of_productW(CUREAL *grid1, CUREAL *grid2, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  int s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b3 = blocks.x * blocks.y * blocks.z;

  rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr, b3);
  cuda_error_check();
  // Blocks, Threads, dynamic memory size
  rgrid_cuda_integral_of_product_gpu<<<blocks,threads,s*sizeof(REAL)>>>(grid1, grid2, (CUREAL *) grid_gpu_mem_addr, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
  rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr, b3);
  cuda_error_check();
}

/*
 * Integrate A * B^2.
 *
 */

__global__ void rgrid_cuda_grid_expectation_value_gpu(CUREAL *a, CUREAL *b, CUREAL *blocks, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2, t;
  extern __shared__ CUREAL els[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els[t] = 0.0;
  }
  __syncthreads();

  idx = (i * ny + j) * nzz + k;
  idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;

  els[idx2] += a[idx] * b[idx] * b[idx];
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] += els[t];  // reduce threads
    }
  }
}

/*
 * Integral a * b^2.
 *
 * grid1    = Source 1 (a) for operation (REAL *; input).
 * grid2    = Source 2 (b) for operation (REAL *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 * Returns the value of integral in grid_gpu_mem[0].
 *
 */

extern "C" void rgrid_cuda_grid_expectation_valueW(CUREAL *grid1, CUREAL *grid2, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  int s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b3 = blocks.x * blocks.y * blocks.z;

  rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr, b3);
  cuda_error_check();
  rgrid_cuda_grid_expectation_value_gpu<<<blocks,threads,s*sizeof(REAL)>>>(grid1, grid2, (CUREAL *) grid_gpu_mem_addr, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
  rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr, b3);
  cuda_error_check();
}

/*
 * B = FD_X(A).
 *
 */

__global__ void rgrid_cuda_fd_gradient_x_gpu(CUREAL *a, CUREAL *b, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  b[idx] = inv_delta * (rgrid_cuda_bc_x_plus(a, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_x_minus(a, bc, i, j, k, nx, ny, nz, nzz));
}

/*
 * B = FD_X(A)
 *
 * grida    = Source for operation (REAL *; input).
 * gridb    = Destination for operation (REAL *; input).
 * inv_delta= 1 / (2 * step) (REAL; input).
 * bc       = Boundary condition: 0 = Dirichlet, 1 = Neumann, 2 = Periodic (char; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_fd_gradient_xW(CUREAL *grida, CUREAL *gridb, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_fd_gradient_x_gpu<<<blocks,threads>>>(grida, gridb, inv_delta, bc, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/*
 * B = FD_Y(A).
 *
 */

__global__ void rgrid_cuda_fd_gradient_y_gpu(CUREAL *a, CUREAL *b, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  b[idx] = inv_delta * (rgrid_cuda_bc_y_plus(a, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_y_minus(a, bc, i, j, k, nx, ny, nz, nzz));
}

/*
 * B = FD_Y(A)
 *
 * grida    = Source for operation (REAL *; input).
 * gridb    = Destination for operation (REAL *; input).
 * inv_delta = 1 / (2 * step) (REAL; input).
 * bc        = Boundary condition: 0 = Dirichlet, 1 = Neumann, 2 = Periodic (char; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_fd_gradient_yW(CUREAL *grida, CUREAL *gridb, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_fd_gradient_y_gpu<<<blocks,threads>>>(grida, gridb, inv_delta, bc, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/*
 * B = FD_Z(A).
 *
 */

__global__ void rgrid_cuda_fd_gradient_z_gpu(CUREAL *a, CUREAL *b, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  b[idx] = inv_delta * (rgrid_cuda_bc_z_plus(a, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_z_minus(a, bc, i, j, k, nx, ny, nz, nzz));
}

/*
 * B = FD_Z(A)
 *
 * grida    = Source for operation (REAL *; input).
 * gridb    = Destination for operation (REAL *; input).
 * inv_delta= 1 / (2 * step) (REAL; input).
 * bc       = Boundary condition: 0 = Dirichlet, 1 = Neumann, 2 = Periodic (char; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_fd_gradient_zW(CUREAL *grida, CUREAL *gridb, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_fd_gradient_z_gpu<<<blocks,threads>>>(grida, gridb, inv_delta, bc, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/*
 * B = LAPLACE(A).
 *
 */

__global__ void rgrid_cuda_fd_laplace_gpu(CUREAL *a, CUREAL *b, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  b[idx] = inv_delta2 * (rgrid_cuda_bc_x_plus(a, bc, i, j, k, nx, ny, nz, nzz) + rgrid_cuda_bc_x_minus(a, bc, i, j, k, nx, ny, nz, nzz)
                       + rgrid_cuda_bc_y_plus(a, bc, i, j, k, nx, ny, nz, nzz) + rgrid_cuda_bc_y_minus(a, bc, i, j, k, nx, ny, nz, nzz)
                       + rgrid_cuda_bc_z_plus(a, bc, i, j, k, nx, ny, nz, nzz) + rgrid_cuda_bc_z_minus(a, bc, i, j, k, nx, ny, nz, nzz)
                       - 6.0 * a[idx]);
}

/*
 * B = LAPLACE(A)
 *
 * grid1      = Source 1 (a) for operation (REAL *; input).
 * grid2      = Source 2 (b) for operation (REAL *; input).
 * inv_delta2 = 1 / (2 * step) (REAL; input).
 * bc         = Boundary condition (char; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 * Returns the value of integral.
 *
 */

extern "C" void rgrid_cuda_fd_laplaceW(CUREAL *grida, CUREAL *gridb, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_fd_laplace_gpu<<<blocks,threads>>>(grida, gridb, inv_delta2, bc, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/*
 * B = LAPLACE_X(A).
 *
 */

__global__ void rgrid_cuda_fd_laplace_x_gpu(CUREAL *a, CUREAL *b, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  b[idx] = inv_delta2 * (rgrid_cuda_bc_x_plus(a, bc, i, j, k, nx, ny, nz, nzz) + rgrid_cuda_bc_x_minus(a, bc, i, j, k, nx, ny, nz, nzz)
                         - 2.0 * a[idx]);
}

/*
 * B = LAPLACE_X(A)
 *
 * grida      = Source for operation (REAL *; input).
 * gridb      = Destination for operation (REAL *; output).
 * inv_delta2 = 1 / (2 * step) (REAL; input).
 * bc         = Boundary condition (char; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 * Returns laplace in gridb.
 *
 */

extern "C" void rgrid_cuda_fd_laplace_xW(CUREAL *grida, CUREAL *gridb, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_fd_laplace_x_gpu<<<blocks,threads>>>(grida, gridb, inv_delta2, bc, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/*
 * B = LAPLACE_Y(A).
 *
 */

__global__ void rgrid_cuda_fd_laplace_y_gpu(CUREAL *a, CUREAL *b, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  b[idx] = inv_delta2 * (rgrid_cuda_bc_y_plus(a, bc, i, j, k, nx, ny, nz, nzz) + rgrid_cuda_bc_y_minus(a, bc, i, j, k, nx, ny, nz, nzz)
                         - 2.0 * a[idx]);
}

/*
 * B = LAPLACE_Y(A)
 *
 * grida      = Source for operation (REAL *; input).
 * gridb      = Destination for operation (REAL *; output).
 * inv_delta2 = 1 / (2 * step) (REAL; input).
 * bc         = Boundary condition (char; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 * Returns laplace in gridb.
 *
 */

extern "C" void rgrid_cuda_fd_laplace_yW(CUREAL *grida, CUREAL *gridb, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_fd_laplace_y_gpu<<<blocks,threads>>>(grida, gridb, inv_delta2, bc, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/*
 * B = LAPLACE_Z(A).
 *
 */

__global__ void rgrid_cuda_fd_laplace_z_gpu(CUREAL *a, CUREAL *b, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  b[idx] = inv_delta2 * (rgrid_cuda_bc_z_plus(a, bc, i, j, k, nx, ny, nz, nzz) + rgrid_cuda_bc_z_minus(a, bc, i, j, k, nx, ny, nz, nzz)
                         - 2.0 * a[idx]);
}

/*
 * B = LAPLACE_Z(A)
 *
 * grida      = Source for operation (REAL *; input).
 * gridb      = Destination for operation (REAL *; output).
 * inv_delta2 = 1 / (2 * step) (REAL; input).
 * bc         = Boundary condition (char; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 * Returns laplace in gridb.
 *
 */

extern "C" void rgrid_cuda_fd_laplace_zW(CUREAL *grida, CUREAL *gridb, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_fd_laplace_z_gpu<<<blocks,threads>>>(grida, gridb, inv_delta2, bc, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/*
 * B = FD_X(A)^2 + FD_Y(A)^2 + FD_Z(A)^2.
 *
 */

__global__ void rgrid_cuda_fd_gradient_dot_gradient_gpu(CUREAL *a, CUREAL *b, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT idx;
  CUREAL tmp;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  b[idx] = 0.0;

  tmp = inv_delta * (rgrid_cuda_bc_x_plus(a, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_x_minus(a, bc, i, j, k, nx, ny, nz, nzz));
  b[idx] = b[idx] + tmp * tmp;

  tmp = inv_delta * (rgrid_cuda_bc_y_plus(a, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_y_minus(a, bc, i, j, k, nx, ny, nz, nzz));
  b[idx] = b[idx] + tmp * tmp;

  tmp = inv_delta * (rgrid_cuda_bc_z_plus(a, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_z_minus(a, bc, i, j, k, nx, ny, nz, nzz));
  b[idx] = b[idx] + tmp * tmp;
}

/*
 * B = FD_X(A)^2 + FD_Y(A)^2 + FD_Z(A)^2.
 *
 * grida      = Source for operation (REAL *; input).
 * gridb      = Destination for operation (REAL *; output).
 * inv_delta2 = 1 / (4 * step * step) (REAL; input).
 * bc         = Boundary condition (char; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_fd_gradient_dot_gradientW(CUREAL *grida, CUREAL *gridb, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_fd_gradient_dot_gradient_gpu<<<blocks,threads>>>(grida, gridb, inv_delta2, bc, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/*
 * Maximum value in a grid.
 *
 */

__global__ void grid_cuda_max_gpu(CUREAL *a, CUREAL *val, INT nx, INT ny, INT nz, INT nzz) {

  /* blockIdx.x = i, threadIdx.x = j */
  INT i, j, k, idx;

  *val = a[0];
  for (i = 0; i < nx; i++)
    for (j = 0; j < ny; j++)
      for(k = 0; k < nz; k++) {
        idx = (i * ny + j) * nzz + k;
        if(a[idx] > *val) *val = a[idx];
      }
}

/*
 * Maximum value contained in a grid. (not parallel)
 *
 * grid    = Source for operation (REAL complex *; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 * Returns maximum value in grid_gpu_mem[0].
 *
 */

extern "C" void grid_cuda_maxW(CUREAL *grid, INT nx, INT ny, INT nz) {

  grid_cuda_max_gpu<<<1,1>>>(grid, (CUREAL *) grid_gpu_mem_addr, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/*
 * Minimum value in a grid.
 *
 */

__global__ void grid_cuda_min_gpu(CUREAL *a, CUREAL *val, INT nx, INT ny, INT nz, INT nzz) {

  /* blockIdx.x = i, threadIdx.x = j */
  INT i, j, k, idx;

  *val = a[0];
  for (i = 0; i < nx; i++)
    for (j = 0; j < ny; j++)
      for(k = 0; k < nz; k++) {
        idx = (i * ny + j) * nzz + k;
        if(a[idx] < *val) *val = a[idx];
      }
}

/*
 * Minimum value contained in a grid. (not parallel)
 *
 * grid    = Source for operation (REAL complex *; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 * Returns minimum value in grid_gpu_mem[0].
 *
 */

extern "C" void grid_cuda_minW(CUREAL *grid, INT nx, INT ny, INT nz) {

  grid_cuda_min_gpu<<<1,1>>>(grid, (CUREAL *) grid_gpu_mem_addr, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/*
 * |rot|
 *
 */

__global__ void rgrid_cuda_abs_rot_gpu(CUREAL *rot, CUREAL *fx, CUREAL *fy, CUREAL *fz, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz, INT nzz) { 
 
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT idx;
  CUREAL tmp;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  /* x: (d/dy) fz - (d/dz) fy */
  tmp = inv_delta * (rgrid_cuda_bc_y_plus(fz, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_y_minus(fz, bc, i, j, k, nx, ny, nz, nzz)
                     - rgrid_cuda_bc_z_plus(fy, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_z_minus(fy, bc, i, j, k, nx, ny, nz, nzz));
  rot[idx] = tmp * tmp;

  /* y: (d/dz) fx - (d/dx) fz */
  tmp = inv_delta * (rgrid_cuda_bc_z_plus(fx, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_z_minus(fx, bc, i, j, k, nx, ny, nz, nzz)
                     - rgrid_cuda_bc_x_plus(fz, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_x_minus(fz, bc, i, j, k, nx, ny, nz, nzz));
  rot[idx] = rot[idx] + tmp * tmp;

  /* z: (d/dx) fy - (d/dy) fx */
  tmp = inv_delta * (rgrid_cuda_bc_x_plus(fy, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_x_minus(fy, bc, i, j, k, nx, ny, nz, nzz)
                     - rgrid_cuda_bc_y_plus(fx, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_y_minus(fx, bc, i, j, k, nx, ny, nz, nzz));
  rot[idx] = rot[idx] + tmp * tmp;
  rot[idx] = SQRT(rot[idx]);
}

/*
 * |rot|
 *
 * rot       = Grid to be operated on (CUREAL *; input/output).
 * fx        = x component of the field (rgrid *; input).
 * fy        = y component of the field (rgrid *; input).
 * fz        = z component of the field (rgrid *; input).
 * inv_delta = 1 / (2 * step) (CUREAL; input).
 * bc        = Boundary condition (char; input).
 * nx        = # of points along x (INT; input).
 * ny        = # of points along y (INT; input).
 * nz        = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_abs_rotW(CUREAL *rot, CUREAL *fx, CUREAL *fy, CUREAL *fz, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_abs_rot_gpu<<<blocks,threads>>>(rot, fx, fy, fz, inv_delta, bc, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/*
 * A = POW(B,n) with n integer.
 *
 */

__global__ void rgrid_cuda_ipower_gpu(CUREAL *a, CUREAL *b, INT n, INT nx, INT ny, INT nz, INT nzz) {
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT idx, ii, sig;
  CUREAL value = 1.0;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  if(n == 0) {
    a[idx] = 1.0;
    return;
  }
  sig = (n < 0) ? -1:1;
  n = ABS(n);
  switch(n) {
    case 1:      
      a[idx] = b[idx];
      break;
    case 2:
      a[idx] = b[idx] * b[idx];
      break;
    case 3:
      a[idx] = b[idx] * b[idx] * b[idx];
      break;
    default:
      for(ii = 0; ii < n; ii++)
        value *= b[idx];
      a[idx] = value;
  }
  if(sig == -1) a[idx] = 1.0 / a[idx];
}

/*
 * Grid integer power.
 *
 * grida    = Destination for operation (REAL *; output).
 * gridb    = Source for operation (REAL *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_ipowerW(CUREAL *grida, CUREAL *gridb, INT exponent, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_ipower_gpu<<<blocks,threads>>>(grida, gridb, exponent, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/*
 * Grid threshold clear device code.
 *
 */

__global__ void rgrid_cuda_threshold_clear_gpu(CUREAL *dest, CUREAL *src, CUREAL ul, CUREAL ll, CUREAL uval, CUREAL lval, INT nx, INT ny, INT nz, INT nzz) {
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  if(src[idx] < ll) dest[idx] = lval;
  if(src[idx] > ul) dest[idx] = uval;
}

/*
 * Grid clear based on threshold.
 *
 * dest    = Destination for operation (REAL *; output).
 * src     = Source for operation (REAL *; input).
 * ul      = upper limit threshold for the operation (REAL; input).
 * ll      = lower limit threshold for the operation (REAL; input).
 * uval    = value to set when the upper limit was exceeded (REAL; input).
 * lval    = value to set when the lower limit was exceeded (REAL; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_threshold_clearW(CUREAL *dest, CUREAL *src, CUREAL ul, CUREAL ll, CUREAL uval, CUREAL lval, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_threshold_clear_gpu<<<blocks,threads>>>(dest, src, ul, ll, uval, lval, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/*
 * Zero part of real grid.
 *
 * A = 0 in the specified range.
 *
 */

__global__ void rgrid_cuda_zero_index_gpu(CUREAL *a, INT lx, INT hx, INT ly, INT hy, INT lz, INT hz, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;

  if(i >= lx && i < hx && j >= ly && j < hy && k >= lz && k < hz)
    a[idx] = 0.0;
}

/*
 * Zero specified range of complex grid.
 *
 * grid     = Grid to be operated on (CUREAL *; input/output).
 * lx, hx, ly, hy, lz, hz = limiting indices (INT; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_zero_indexW(CUREAL *grid, INT lx, INT hx, INT ly, INT hy, INT lz, INT hz, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_zero_index_gpu<<<blocks,threads>>>(grid, lx, hx, ly, hy, lz, hz, nx, ny, nz);
  cuda_error_check();
}

/*
 * Poisson equation.
 *
 */

__global__ void rgrid_cuda_poisson_gpu(CUCOMPLEX *grid, CUREAL norm, CUREAL step2, CUREAL ilx, CUREAL ily, CUREAL ilz, INT nx, INT ny, INT nzz) {
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT idx;
  CUREAL kx, ky, kz;

  if(i >= nx || j >= ny || k >= nzz) return;

  idx = (i * ny + j) * nzz + k;
  kx = COS(ilx * (CUREAL) i);
  ky = COS(ily * (CUREAL) j);
  kz = COS(ilz * (CUREAL) k);
  if(i || j || k)
    grid[idx] = grid[idx] * norm * step2 / (2.0 * (kx + ky + kz - 3.0));
  else
    grid[idx] = CUMAKE(0.0, 0.0);
}

/*
 * Solve Poisson.
 *
 * grid    = Grid specifying the RHS (CUREAL *; input/output).
 * norm    = FFT normalization constant (CUREAL; input).
 * step2   = Spatial step ^ 2 (CUREAL; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_poissonW(CUCOMPLEX *grid, CUREAL norm, CUREAL step2, INT nx, INT ny, INT nz) {

  CUREAL ilx = 2.0 * M_PI / ((CUREAL) nx), ily = 2.0 * M_PI / ((CUREAL) ny), ilz = 2.0 * M_PI / ((CUREAL) nz);
  INT nzz = nz / 2 + 1;
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nzz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_poisson_gpu<<<blocks,threads>>>(grid, norm, step2, ilx, ily, ilz, nx, ny, nzz);
  cuda_error_check();
}

/*
 * FFT gradient (x).
 *
 */

__global__ void rgrid_cuda_fft_gradient_x_gpu(CUCOMPLEX *gradient, REAL kx0, REAL step, REAL norm, INT nx, INT ny, INT nz, INT nx2) {
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT idx;
  REAL lx, kx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  lx = 2.0 * M_PI / (((REAL) nx) * step);
  if(i < nx2) 
    kx = ((REAL) i) * lx - kx0;
  else
    kx = -((REAL) (nx - i)) * lx - kx0;
  gradient[idx] = gradient[idx] * CUMAKE(0.0, kx * norm);
}

/*
 * Gradient of grid in Fourier space (X).
 *
 * gradient_x = Source & destination for operation (CUCOMPLEX *; input/output).
 * kx0        = Baseline momentum (grid->kx0; REAL; input).
 * step       = Step size (REAL; input).
 * norm       = FFT norm (REAL; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input). This is grid->nz2 / 2
 *
 */

extern "C" void rgrid_cuda_fft_gradient_xW(CUCOMPLEX *gradient_x, REAL kx0, REAL step, REAL norm, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_fft_gradient_x_gpu<<<blocks,threads>>>(gradient_x, kx0, step, norm, nx, ny, nz, nx / 2);
  cuda_error_check();
}

/*
 * FFT gradient (y).
 *
 */

__global__ void rgrid_cuda_fft_gradient_y_gpu(CUCOMPLEX *gradient, REAL ky0, REAL step, REAL norm, INT nx, INT ny, INT nz, INT ny2) {
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT idx;
  REAL ly, ky;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  ly = 2.0 * M_PI / (((REAL) ny) * step);
  if(j < ny2) 
    ky = ((REAL) j) * ly - ky0;
  else
    ky = -((REAL) (ny - j)) * ly - ky0;
  gradient[idx] = gradient[idx] * CUMAKE(0.0, ky * norm);
}

/*
 * Gradient of grid in Fourier space (Y).
 *
 * gradient_y = Source & destination for operation (CUCOMPLEX *; input/output).
 * ky0        = Baseline momentum (grid->ky0; REAL; input).
 * step       = Step size (REAL; input).
 * norm       = FFT norm (REAL; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input). This is grid->nz2 / 2
 *
 */

extern "C" void rgrid_cuda_fft_gradient_yW(CUCOMPLEX *gradient_y, REAL ky0, REAL step, REAL norm, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_fft_gradient_y_gpu<<<blocks,threads>>>(gradient_y, ky0, step, norm, nx, ny, nz, ny / 2);
  cuda_error_check();
}

/*
 * FFT gradient (z).
 *
 */

__global__ void rgrid_cuda_fft_gradient_z_gpu(CUCOMPLEX *gradient, REAL kz0, REAL step, REAL norm, INT nx, INT ny, INT nz, INT nz2) {
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT idx;
  REAL lz, kz;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  lz = M_PI / (((REAL) nz - 1) * step);
  if(k < nz2) 
    kz = ((REAL) k) * lz - kz0;
  else
    kz = -((REAL) (nz - k)) * lz - kz0;
  gradient[idx] = gradient[idx] * CUMAKE(0.0, kz * norm);
}

/*
 * Gradient of grid in Fourier space (Z).
 *
 * gradient_z = Source & destination for operation (CUCOMPLEX *; input/output).
 * kz0        = Baseline momentum (grid->ky0; REAL; input).
 * step       = Step size (REAL; input).
 * norm       = FFT norm (REAL; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input). This is grid->nz2 / 2
 *
 */

extern "C" void rgrid_cuda_fft_gradient_zW(CUCOMPLEX *gradient_z, REAL kz0, REAL step, REAL norm, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_fft_gradient_z_gpu<<<blocks,threads>>>(gradient_z, kz0, step, norm, nx, ny, nz, nz / 2);
  cuda_error_check();
}


/*
 * FFT laplace.
 *
 * B = B'' in Fourier space.
 *
 */

__global__ void rgrid_cuda_fft_laplace_gpu(CUCOMPLEX *b, CUREAL norm, CUREAL kx0, CUREAL ky0, CUREAL kz0, CUREAL lx, CUREAL ly, CUREAL lz, CUREAL step, INT nx, INT ny, INT nz, INT nx2, INT ny2, INT nz2) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT idx;
  CUREAL kx, ky, kz;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  
  if(i < nx2) 
    kx = ((REAL) i) * lx - kx0;
  else
    kx = -((REAL) (nx - i)) * lx - kx0;
  if(j < ny2) 
    ky = ((REAL) j) * ly - ky0;
  else
    ky = -((REAL) (ny - j)) * ly - ky0;
  if(k < nz2) 
    kz = ((REAL) k) * lz - kz0;
  else
    kz = -((REAL) (nz - k)) * lz - kz0;        

  b[idx] = b[idx] * (-(kx * kx + ky * ky + kz * kz) * norm);
}

/*
 * FFT laplace
 *
 * laplace  = Source/destination grid for operation (REAL complex *; input/output).
 * norm     = FFT norm (grid->fft_norm) (REAL; input).
 * kx0      = Momentum shift of origin along X (REAL; input).
 * ky0      = Momentum shift of origin along Y (REAL; input).
 * kz0      = Momentum shift of origin along Z (REAL; input).
 * step     = Spatial step length (REAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 * Only periodic boundaries!
 *
 */

extern "C" void rgrid_cuda_fft_laplaceW(CUCOMPLEX *laplace, CUREAL norm, CUREAL kx0, CUREAL ky0, CUREAL kz0, CUREAL step, INT nx, INT ny, INT nz) {

  INT nx2 = nx / 2, ny2 = ny / 2, nz2 = ny / 2;
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_fft_laplace_gpu<<<blocks,threads>>>(laplace, norm, kx0, ky0, kz0, 2.0 * M_PI / (((REAL) nx) * step), 2.0 * M_PI / (((REAL) ny) * step), M_PI / (((REAL) nz - 1) * step), step, nx, ny, nz, nx2, ny2, nz2);
  cuda_error_check();
}

/*
 * FFT laplace expectation value.
 *
 * B = <B''> in Fourier space.
 *
 * Only periodic version implemented.
 *
 * Normalization done in rgrid-cuda.c
 *
 */

__global__ void rgrid_cuda_fft_laplace_expectation_value_gpu(CUCOMPLEX *b, CUCOMPLEX *blocks, CUREAL kx0, CUREAL ky0, CUREAL kz0, CUREAL lx, CUREAL ly, CUREAL lz, CUREAL step, INT nx, INT ny, INT nz, INT nx2, INT ny2, INT nz2) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2, t;
  CUREAL kx, ky, kz;
  extern __shared__ CUREAL els2[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els2[t] = 0.0;
  }
  __syncthreads();

  idx = (i * ny + j) * nz + k;

  if(i < nx2) 
    kx = ((REAL) i) * lx - kx0;
  else
    kx = -((REAL) (nx - i)) * lx - kx0;
  if(j < ny2) 
    ky = ((REAL) j) * ly - ky0;
  else
    ky = -((REAL) (ny - j)) * ly - ky0;
  if(k < nz2) 
    kz = ((REAL) k) * lz - kz0;
  else
    kz = -((REAL) (nz - k)) * lz - kz0;        

  idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
  els2[idx2] -= (kx * kx + ky * ky + kz * kz) * (CUCREAL(b[idx]) * CUCREAL(b[idx]) + CUCIMAG(b[idx]) * CUCIMAG(b[idx]));
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2].x += els2[t];  // reduce threads
    }
  }
}

/*
 * FFT laplace expectation value
 *
 * laplace  = Source/destination grid for operation (REAL complex *; input/output).
 * kx0      = Momentum shift of origin along X (REAL; input).
 * ky0      = Momentum shift of origin along Y (REAL; input).
 * kz0      = Momentum shift of origin along Z (REAL; input).
 * step     = Spatial step length (REAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 * sum      = Expectation value (REAL; output).
 *
 * Only periodic boundaries!
 *
 * Normalization done in cgrid-cuda.c
 *
 */

extern "C" void rgrid_cuda_fft_laplace_expectation_valueW(CUCOMPLEX *laplace, CUREAL kx0, CUREAL ky0, CUREAL kz0, CUREAL step, INT nx, INT ny, INT nz) {

  INT nx2 = nx / 2, ny2 = ny / 2, nz2 = nz / 2;
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  int s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b3 = blocks.x * blocks.y * blocks.z;

  rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr, b3);
  cuda_error_check();
  rgrid_cuda_fft_laplace_expectation_value_gpu<<<blocks,threads,s*sizeof(CUREAL)>>>(laplace, (CUCOMPLEX *) grid_gpu_mem_addr, kx0, ky0, kz0, 2.0 * M_PI / (((REAL) nx) * step), 2.0 * M_PI / (((REAL) ny) * step), M_PI / (((REAL) nz - 1) * step), step, nx, ny, nz, nx2, ny2, nz2);
  cuda_error_check();
  rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr, b3);
  cuda_error_check();
}
