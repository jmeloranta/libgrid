#include "hip/hip_runtime.h"
/*
 * CUDA device code (REAL; rgrid).
 *
 */

#include <cuda/hip/hip_runtime_api.h>
#include <cuda/hip/hip_runtime.h>
#include <cuda/>
#include <cuda/hipfft/hipfft.h>
#include "cuda-math.h"

extern void *grid_gpu_mem_addr;
extern "C" void cuda_error_check();

/********************************************************************************************************************/

/*
 * Fourier space convolution device code. This cannot not be called directly.
 *
 * C = A * B but with alternating signs for FFT.
 *
 */

__global__ void rgrid_cuda_fft_convolute_gpu(CUCOMPLEX *c, CUCOMPLEX *a, CUCOMPLEX *b, CUREAL norm, INT nx, INT ny, INT nz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  if((i + j + k) & 1) norm *= -1.0;
  c[idx] = norm * a[idx] * b[idx];
}

/*
 * Convolution in the Fourier space (data in GPU). Not called directly.
 *
 * Multiplication in GPU memory: grid_gpu_mem[i] = grid_gpu_mem[i] * grid_gpu_mem[j] (with sign variation).
 * Note: this includes the sign variation needed for convolution as well as normalization!
 *
 * grida = 1st grid to be convoluted (CUCOMPLEX *).
 * gridb = 2nd grid to be convoluted (CUCOMPLEX *).
 * gridc = output (CUCOMPLEX *).
 * norm  = FFT norm (REAL; input).
 * nx    = Grid dim x (INT; input).
 * ny    = Grid dim y (INT; input).
 * nz    = Grid dim z (INT; input).
 *
 */

extern "C" void rgrid_cuda_fft_convoluteW(CUCOMPLEX *gridc, CUCOMPLEX *grida, CUCOMPLEX *gridb, CUREAL norm, INT nx, INT ny, INT nz) {

  INT nzz = nz / 2 + 1;
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nzz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_fft_convolute_gpu<<<blocks,threads>>>(gridc, grida, gridb, norm, nx, ny, nzz);
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Grid power device code. This cannot not be called directly.
 *
 * A = POW(B,x)
 *
 */

__global__ void rgrid_cuda_power_gpu(CUREAL *a, CUREAL *b, CUREAL x, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  a[idx] = POW(b[idx], x);
}

/*
 * Grid power.
 *
 * gridb    = Destination for operation (REAL *; output).
 * grida    = Source for operation (REAL *; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 */

extern "C" void rgrid_cuda_powerW(CUREAL *gridb, CUREAL *grida, CUREAL exponent, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_power_gpu<<<blocks,threads>>>(gridb, grida, exponent, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Grid power device code. This cannot not be called directly.
 *
 * A = POW(|B|,x)
 *
 */

__global__ void rgrid_cuda_abs_power_gpu(CUREAL *a, CUREAL *b, CUREAL x, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  a[idx] = POW(FABS(b[idx]), x);
}

/*
 * Grid abs power.
 *
 * gridb    = Destination for operation (REAL *; output).
 * grida    = Source for operation (REAL *; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 */

extern "C" void rgrid_cuda_abs_powerW(CUREAL *gridb, CUREAL *grida, CUREAL exponent, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_abs_power_gpu<<<blocks,threads>>>(gridb, grida, exponent, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Multiply grid by constant device code. This cannot not be called directly.
 *
 * A = C * A
 *
 */

__global__ void rgrid_cuda_multiply_gpu(CUREAL *a, CUREAL c, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;

  if(i >= nx || j >= ny || k >= nz) return;

  a[(i * ny + j) * nzz + k] *= c;
}

/*
 * Multiply grid by a constant.
 *
 * grid     = Grid to be operated on (CUREAL *; input/output).
 * c        = Multiplying constant (CUREAL).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 */

extern "C" void rgrid_cuda_multiplyW(CUREAL *grid, CUREAL c, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_multiply_gpu<<<blocks,threads>>>(grid, c, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Multiply (complex) grid by constant device code. This cannot not be called directly.
 *
 * A = C * A
 *
 */

__global__ void rgrid_cuda_multiply_fft_gpu(CUCOMPLEX *a, CUREAL c, INT nx, INT ny, INT nz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;

  a[idx] = a[idx] * c;
}

/*
 * Multiply (complex) grid by a constant.
 *
 * grid     = Grid to be operated on (CUCOMPLEX *; input/output).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 */

extern "C" void rgrid_cuda_multiply_fftW(CUCOMPLEX *grid, CUREAL c, INT nx, INT ny, INT nz) {

  INT nzz = nz / 2 + 1;
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nzz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_multiply_fft_gpu<<<blocks,threads>>>(grid, c, nx, ny, nzz);
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Sum of two grids.
 *
 * A = B + C
 *
 */

__global__ void rgrid_cuda_sum_gpu(CUREAL *a, CUREAL *b, CUREAL *c, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  a[idx] = b[idx] + c[idx];
}

/*
 * Sum of two grids.
 *
 * gridc    = Destination grid (CUREAL *; output).
 * grida    = Input grid 1 (CUREAL *; input).
 * gridb    = Input grid 2 (CUREAL *; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 */

extern "C" void rgrid_cuda_sumW(CUREAL *gridc, CUREAL *grida, CUREAL *gridb, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_sum_gpu<<<blocks,threads>>>(gridc, grida, gridb, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Subtract of two grids.
 *
 * A = B - C
 *
 */

__global__ void rgrid_cuda_difference_gpu(CUREAL *a, CUREAL *b, CUREAL *c, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  a[idx] = b[idx] - c[idx];
}

/*
 * Subtract two grids.
 *
 * gridc    = Destination grid (CUREAL *; output).
 * grida    = Input grid 1 (CUREAL *; input).
 * gridb    = Input grid 2 (CUREAL *; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 */

extern "C" void rgrid_cuda_differenceW(CUREAL *gridc, CUREAL *grida, CUREAL *gridb, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_difference_gpu<<<blocks,threads>>>(gridc, grida, gridb, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Product of two grids.
 *
 * A = B * C.
 *
 */

__global__ void rgrid_cuda_product_gpu(CUREAL *a, CUREAL *b, CUREAL *c, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  a[idx] = b[idx] * c[idx];
}

/*
 * Product of two grids.
 *
 * gridc    = Destination grid (CUREAL *; output).
 * grida    = Source grid 1 (CUREAL *; input).
 * gridb    = Source grid 2 (CUREAL *; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 */

extern "C" void rgrid_cuda_productW(CUREAL *gridc, CUREAL *grida, CUREAL *gridb, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_product_gpu<<<blocks,threads>>>(gridc, grida, gridb, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Division of two grids.
 *
 * A = B / C.
 *
 */

__global__ void rgrid_cuda_division_gpu(CUREAL *a, CUREAL *b, CUREAL *c, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  a[idx] = b[idx] / c[idx];
}

/*
 * Division of two grids.
 *
 * gridc    = Destination grid (CUREAL *; output).
 * grida    = Source grid 1 (CUREAL *; input).
 * gridb    = Source grid 2 (CUREAL *; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 */

extern "C" void rgrid_cuda_divisionW(CUREAL *gridc, CUREAL *grida, CUREAL *gridb, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_division_gpu<<<blocks,threads>>>(gridc, grida, gridb, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Safe division of two grids.
 *
 * A = B / (C + eps).
 *
 */

__global__ void rgrid_cuda_division_eps_gpu(CUREAL *a, CUREAL *b, CUREAL *c, CUREAL eps, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  a[idx] = b[idx] / (c[idx] + eps);
}

/*
 * Division of two grids.
 *
 * gridc    = Destination grid (CUREAL *; output).
 * grida    = Source grid 1 (CUREAL *; input).
 * gridb    = Source grid 2 (CUREAL *; input).
 * eps      = Epsilon (CUREAL).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 */

extern "C" void rgrid_cuda_division_epsW(CUREAL *gridc, CUREAL *grida, CUREAL *gridb, CUREAL eps, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_division_eps_gpu<<<blocks,threads>>>(gridc, grida, gridb, eps, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Add constant to grid device code. This cannot not be called directly.
 *
 * A = A + c
 *
 */

__global__ void rgrid_cuda_add_gpu(CUREAL *a, CUREAL c, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  a[idx] += c;
}

/*
 * Add constant to grid.
 *
 * grid     = Grid to be operated on (CUREAL *; input/output).
 * c        = Constant (CUREAL).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 */

extern "C" void rgrid_cuda_addW(CUREAL *grid, CUREAL c, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_add_gpu<<<blocks,threads>>>(grid, c, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Add multiply and add device code. This cannot not be called directly.
 *
 * A = cm * A + ca
 *
 */

__global__ void rgrid_cuda_multiply_and_add_gpu(CUREAL *a, CUREAL cm, CUREAL ca, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  a[idx] = a[idx] * cm + ca;
}

/*
 * Grid multiply and add.
 *
 * grid     = Grid to be operated on (CUREAL *; input/output).
 * cm       = Multiplier (CUREAL).
 * ca       = Additive constant (CUREAL).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 */

extern "C" void rgrid_cuda_multiply_and_addW(CUREAL *grid, CUREAL cm, REAL ca, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_multiply_and_add_gpu<<<blocks,threads>>>(grid, cm, ca, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Add multiply and add device code. This cannot not be called directly.
 *
 * A = cm * (A + ca)
 *
 */

__global__ void rgrid_cuda_add_and_multiply_gpu(CUREAL *a, CUREAL ca, CUREAL cm, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  a[idx] = (a[idx] + ca) * cm;
}

/*
 * Grid multiply and add.
 *
 * grid     = Grid to be operated on (CUREAL *; input/output).
 * cm       = Multiplier (CUREAL).
 * ca       = Additive constant (CUREAL).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 */

extern "C" void rgrid_cuda_add_and_multiplyW(CUREAL *grid, CUREAL ca, CUREAL cm, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_add_and_multiply_gpu<<<blocks,threads>>>(grid, ca, cm, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Add scaled grid device code. This cannot not be called directly.
 *
 * A = A + d * B
 *
 */

__global__ void rgrid_cuda_add_scaled_gpu(CUREAL *a, CUREAL d, CUREAL *b, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  a[idx] += b[idx] * d;
}

/*
 * Scaled add grid.
 *
 * gridc    = Destination for operation (REAL *; output).
 * d        = Scaling factor (REAL).
 * grida    = Source for operation (REAL *; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 */

extern "C" void rgrid_cuda_add_scaledW(CUREAL *gridc, CUREAL d, CUREAL *grida, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_add_scaled_gpu<<<blocks,threads>>>(gridc, d, grida, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Add scaled product grid device code. This cannot not be called directly.
 *
 * A = A + d * B * C
 *
 */

__global__ void rgrid_cuda_add_scaled_product_gpu(CUREAL *a, CUREAL d, CUREAL *b, CUREAL *c, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  a[idx] += d * b[idx] * c[idx];
}

/*
 * Add scaled product.
 *
 * gridc    = Destination for operation (REAL *; output).
 * d        = Scaling factor (REAL).
 * grida    = Source for operation (REAL *; input).
 * gridb    = Source for operation (REAL *; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 */

extern "C" void rgrid_cuda_add_scaled_productW(CUREAL *gridc, CUREAL d, CUREAL *grida, CUREAL *gridb, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_add_scaled_product_gpu<<<blocks,threads>>>(gridc, d, grida, gridb, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Set A to constant.
 *
 * A = c
 *
 */

__global__ void rgrid_cuda_constant_gpu(CUREAL *a, CUREAL c, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  a[idx] = c;
}

/*
 * Set grid to constant.
 *
 * grid     = Destination for operation (REAL *; output).
 * c        = Constant (REAL).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 */

extern "C" void rgrid_cuda_constantW(CUREAL *grid, CUREAL c, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_constant_gpu<<<blocks,threads>>>(grid, c, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/*
 * Block init.
 *
 */

__global__ void rgrid_cuda_block_init(CUREAL *blocks, INT nblocks) {

  INT i;

  for(i = 0; i < nblocks; i++) blocks[i] = 0.0;
}

/*
 * Block reduction.
 *
 */

__global__ void rgrid_cuda_block_reduce(CUREAL *blocks, INT nblocks) {

  INT i;

  for(i = 1; i < nblocks; i++)
    blocks[0] += blocks[i];  // reduce blocks
}

/********************************************************************************************************************/

/*
 * Integrate over A.
 *
 */

__global__ void rgrid_cuda_integral_gpu(CUREAL *a, CUREAL *blocks, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, t;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2;
  extern __shared__ CUREAL els[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els[t] = 0.0;
  }
  __syncthreads();

  idx = (i * ny + j) * nzz + k;
  idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;

  els[idx2] += a[idx];
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] += els[t];  // reduce threads
    }
  }
}

/*
 * Integrate over grid.
 *
 * grid     = Source for operation (REAL *; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 * Returns the value of integral.
 *
 */

extern "C" void rgrid_cuda_integralW(CUREAL *grid, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  int s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b3 = blocks.x * blocks.y * blocks.z;

  rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr, b3);
  cuda_error_check();
  // Blocks, Threads, dynamic memory size
  rgrid_cuda_integral_gpu<<<blocks,threads,s*sizeof(REAL)>>>(grid, (CUREAL *) grid_gpu_mem_addr, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
  rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr, b3);
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Integrate over A with limits.
 *
 */

__global__ void rgrid_cuda_integral_region_gpu(CUREAL *a, CUREAL *blocks, INT il, INT iu, INT jl, INT ju, INT kl, INT ku, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, t;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2;
  extern __shared__ CUREAL els[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els[t] = 0.0;
  }
  __syncthreads();

  if(i >= il && i <= iu && j >= jl && j <= ju && k >= kl && k <= ku) {
    idx = (i * ny + j) * nzz + k;
    idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
    els[idx2] += a[idx];
  }
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] += els[t];  // reduce threads
    }
  }
}

/*
 * Integrate over grid with limits.
 *
 * grid     = Source for operation (REAL *; input).
 * il       = Lower index for x (INT; input).
 * iu       = Upper index for x (INT; input).
 * jl       = Lower index for y (INT; input).
 * ju       = Upper index for y (INT; input).
 * kl       = Lower index for z (INT; input).
 * ku       = Upper index for z (INT; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 * Returns the value of integral.
 *
 */

extern "C" void rgrid_cuda_integral_regionW(CUREAL *grid, INT il, INT iu, INT jl, INT ju, INT kl, INT ku, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  int s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b3 = blocks.x * blocks.y * blocks.z;

  if(il < 0) il = 0;  
  if(jl < 0) jl = 0;  
  if(kl < 0) kl = 0;  
  if(iu > nx-1) iu = nx-1;
  if(ju > ny-1) ju = ny-1;
  if(ku > nz-1) ku = nz-1;

  rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr, b3);
  cuda_error_check();
  rgrid_cuda_integral_region_gpu<<<blocks,threads,s*sizeof(REAL)>>>(grid, (CUREAL *) grid_gpu_mem_addr, il, iu, jl, ju, kl, ku, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
  rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr, b3);
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Integrate of A^2.
 *
 */

__global__ void rgrid_cuda_integral_of_square_gpu(CUREAL *a, CUREAL *blocks, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, t;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2;
  extern __shared__ CUREAL els[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els[t] = 0.0;
  }
  __syncthreads();

  idx = (i * ny + j) * nzz + k;
  idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;

  els[idx2] += a[idx] * a[idx];
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] += els[t];  // reduce threads
    }
  }
}

/*
 * Integral of square.
 *
 * grid     = Source for operation (REAL *; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 * Returns the value of integral.
 *
 */

extern "C" void rgrid_cuda_integral_of_squareW(CUREAL *grid, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  int s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b3 = blocks.x * blocks.y * blocks.z;

  rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr, b3);
  cuda_error_check();
  rgrid_cuda_integral_of_square_gpu<<<blocks,threads,s*sizeof(REAL)>>>(grid, (CUREAL *) grid_gpu_mem_addr, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
  rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr, b3);
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Integrate A * B.
 *
 */

__global__ void rgrid_cuda_integral_of_product_gpu(CUREAL *a, CUREAL *b, CUREAL *blocks, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, t;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2;
  extern __shared__ CUREAL els[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els[t] = 0.0;
  }
  __syncthreads();

  idx = (i * ny + j) * nzz + k;
  idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;

  els[idx2] += a[idx] * b[idx];
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] += els[t];  // reduce threads
    }
  }
}

/*
 * Integral of product.
 *
 * grid1    = Source 1 for operation (REAL *; input).
 * grid2    = Source 2 for operation (REAL *; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 * Returns the value of integral.
 *
 */

extern "C" void rgrid_cuda_integral_of_productW(CUREAL *grid1, CUREAL *grid2, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  int s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b3 = blocks.x * blocks.y * blocks.z;

  rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr, b3);
  cuda_error_check();
  // Blocks, Threads, dynamic memory size
  rgrid_cuda_integral_of_product_gpu<<<blocks,threads,s*sizeof(REAL)>>>(grid1, grid2, (CUREAL *) grid_gpu_mem_addr, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
  rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr, b3);
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Integrate A * B^2.
 *
 */

__global__ void rgrid_cuda_grid_expectation_value_gpu(CUREAL *a, CUREAL *b, CUREAL *blocks, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, t;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2;
  extern __shared__ CUREAL els[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els[t] = 0.0;
  }
  __syncthreads();

  idx = (i * ny + j) * nzz + k;
  idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;

  els[idx2] += a[idx] * b[idx] * b[idx];
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] += els[t];  // reduce threads
    }
  }
}

/*
 * Integral a * b^2.
 *
 * grid1    = Source 1 (a) for operation (REAL *; input).
 * grid2    = Source 2 (b) for operation (REAL *; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 * Returns the value of integral.
 *
 */

extern "C" void rgrid_cuda_grid_expectation_valueW(CUREAL *grid1, CUREAL *grid2, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  int s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b3 = blocks.x * blocks.y * blocks.z;

  rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr, b3);
  cuda_error_check();
  rgrid_cuda_grid_expectation_value_gpu<<<blocks,threads,s*sizeof(REAL)>>>(grid1, grid2, (CUREAL *) grid_gpu_mem_addr, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
  rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr, b3);
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * B = FD_X(A). Neumann BC. FIXME: This does not respect the BC of the original grid!
 *
 */

__global__ void rgrid_cuda_fd_gradient_x_gpu(CUREAL *a, CUREAL *b, CUREAL inv_delta, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  if(i == 0 || i == nx-1) b[idx] = 0.0;
  else b[idx] = inv_delta * (a[((i+1)*ny + j)*nzz + k] - a[((i-1)*ny + j)*nzz + k]);
}

/*
 * B = FD_X(A)
 *
 * grida    = Source for operation (REAL *; input).
 * gridb    = Destination for operation (REAL *; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 * Returns the value of integral.
 *
 */

extern "C" void rgrid_cuda_fd_gradient_xW(CUREAL *grida, CUREAL *gridb, CUREAL inv_delta, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_fd_gradient_x_gpu<<<blocks,threads>>>(grida, gridb, inv_delta, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * B = FD_Y(A). Neumann BC. FIXME: This does not respect the BC of the original grid!
 *
 */

__global__ void rgrid_cuda_fd_gradient_y_gpu(CUREAL *a, CUREAL *b, CUREAL inv_delta, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  if(j == 0 || j == ny-1) b[idx] = 0.0;
  else b[idx] = inv_delta * (a[(i*ny + j + 1)*nzz + k] - a[(i*ny + j - 1)*nzz + k]);
}

/*
 * B = FD_Y(A)
 *
 * grida    = Source for operation (REAL *; input).
 * gridb    = Destination for operation (REAL *; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 * Returns the value of integral.
 *
 */

extern "C" void rgrid_cuda_fd_gradient_yW(CUREAL *grida, CUREAL *gridb, CUREAL inv_delta, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_fd_gradient_y_gpu<<<blocks,threads>>>(grida, gridb, inv_delta, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * B = FD_Z(A). Neumann BC. FIXME: This does not respect the BC of the original grid!
 *
 */

__global__ void rgrid_cuda_fd_gradient_z_gpu(CUREAL *a, CUREAL *b, CUREAL inv_delta, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  if(k == 0 || k == nz-1) b[idx] = 0.0;
  else b[idx] = inv_delta * (a[(i*ny + j)*nzz + k + 1] - a[(i*ny + j)*nzz + k - 1]);
}

/*
 * B = FD_Z(A)
 *
 * grida    = Source for operation (REAL *; input).
 * gridb    = Destination for operation (REAL *; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 * Returns the value of integral.
 *
 */

extern "C" void rgrid_cuda_fd_gradient_zW(CUREAL *grida, CUREAL *gridb, CUREAL inv_delta, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_fd_gradient_z_gpu<<<blocks,threads>>>(grida, gridb, inv_delta, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * B = LAPLACE(A). Neumann BC. FIXME: This does not respect the BC of the original grid!
 *
 */

__global__ void rgrid_cuda_fd_laplace_gpu(CUREAL *a, CUREAL *b, CUREAL inv_delta2, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  CUREAL xp, xm, yp, ym, zp, zm;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  // Neumann BC
  if(i == nx-1) xp = a[j * nzz + k]; // x -> 0, i = 0
  else xp = a[((i+1) * ny + j) * nzz + k];

  if(i == 0) xm = a[((nx-1) * ny + j) * nzz + k]; // x -> (nx-1) * step, i = nx-1
  else xm = a[((i - 1) * ny + j) * nzz + k];

  if(j == ny-1) yp = a[i * ny * nzz + k]; // y -> 0, j = 0
  else yp = a[(i * ny + j + 1) * nzz + k];

  if(j == 0) ym = a[(i * ny + ny - 1) * nzz + k]; // y -> (ny-1) * step, j = ny-1
  else ym = a[(i * ny + j - 1) * nzz + k];

  if(k == nz-1) zp = a[(i * ny + j) * nzz]; // z -> 0, k = 0
  else zp = a[(i * ny + j) * nzz + k + 1];

  if(k == 0) zm = a[(i * ny + j) * nzz + nz - 1]; // z -> (nz-1) * step, k = nz-1
  else zm = a[(i * ny + j) * nzz + k - 1];

  b[idx] = inv_delta2 * (xp + xm + yp + ym + zp + zm - 6.0 * a[idx]);
}

/*
 * B = LAPLACE(A)
 *
 * grid1    = Source 1 (a) for operation (REAL *; input).
 * grid2    = Source 2 (b) for operation (REAL *; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 * Returns the value of integral.
 *
 */

extern "C" void rgrid_cuda_fd_laplaceW(CUREAL *grida, CUREAL *gridb, CUREAL inv_delta2, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_fd_laplace_gpu<<<blocks,threads>>>(grida, gridb, inv_delta2, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * B = LAPLACE_X(A). Neumann BC. FIXME: This does not respect the BC of the original grid!
 *
 */

__global__ void rgrid_cuda_fd_laplace_x_gpu(CUREAL *a, CUREAL *b, CUREAL inv_delta2, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  CUREAL xp, xm;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  // Neumann BC
  if(i == nx-1) xp = a[j * nzz + k]; // x -> 0, i = 0
  else xp = a[((i+1) * ny + j) * nzz + k];

  if(i == 0) xm = a[((nx-1) * ny + j) * nzz + k]; // x -> (nx-1) * step, i = nx-1
  else xm = a[((i - 1) * ny + j) * nzz + k];

  b[idx] = inv_delta2 * (xp + xm - 2.0 * a[idx]);
}

/*
 * B = LAPLACE_X(A)
 *
 * grid1    = Source 1 (a) for operation (REAL *; input).
 * grid2    = Source 2 (b) for operation (REAL *; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 * Returns the value of integral.
 *
 */

extern "C" void rgrid_cuda_fd_laplace_xW(CUREAL *grida, CUREAL *gridb, CUREAL inv_delta2, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_fd_laplace_x_gpu<<<blocks,threads>>>(grida, gridb, inv_delta2, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * B = LAPLACE_Y(A). Neumann BC. FIXME: This does not respect the BC of the original grid!
 *
 */

__global__ void rgrid_cuda_fd_laplace_y_gpu(CUREAL *a, CUREAL *b, CUREAL inv_delta2, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  CUREAL yp, ym;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  // Neumann BC

  if(j == ny-1) yp = a[i * ny * nzz + k]; // y -> 0, j = 0
  else yp = a[(i * ny + j + 1) * nzz + k];

  if(j == 0) ym = a[(i * ny + ny - 1) * nzz + k]; // y -> (ny-1) * step, j = ny-1
  else ym = a[(i * ny + j - 1) * nzz + k];

  b[idx] = inv_delta2 * (yp + ym - 2.0 * a[idx]);
}

/*
 * B = LAPLACE_Y(A)
 *
 * grid1    = Source 1 (a) for operation (REAL *; input).
 * grid2    = Source 2 (b) for operation (REAL *; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 * Returns the value of integral.
 *
 */

extern "C" void rgrid_cuda_fd_laplace_yW(CUREAL *grida, CUREAL *gridb, CUREAL inv_delta2, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_fd_laplace_y_gpu<<<blocks,threads>>>(grida, gridb, inv_delta2, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * B = LAPLACE_Z(A). Neumann BC. FIXME: This does not respect the BC of the original grid!
 *
 */

__global__ void rgrid_cuda_fd_laplace_z_gpu(CUREAL *a, CUREAL *b, CUREAL inv_delta2, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  CUREAL zp, zm;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  // Neumann BC
  if(k == nz-1) zp = a[(i * ny + j) * nzz]; // z -> 0, k = 0
  else zp = a[(i * ny + j) * nzz + k + 1];

  if(k == 0) zm = a[(i * ny + j) * nzz + nz - 1]; // z -> (nz-1) * step, k = nz-1
  else zm = a[(i * ny + j) * nzz + k - 1];

  b[idx] = inv_delta2 * (zp + zm - 2.0 * a[idx]);
}

/*
 * B = LAPLACE_Z(A)
 *
 * grid1    = Source 1 (a) for operation (REAL *; input).
 * grid2    = Source 2 (b) for operation (REAL *; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 * Returns the value of integral.
 *
 */

extern "C" void rgrid_cuda_fd_laplace_zW(CUREAL *grida, CUREAL *gridb, CUREAL inv_delta2, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_fd_laplace_z_gpu<<<blocks,threads>>>(grida, gridb, inv_delta2, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * B = FD_X(A)^2 + FD_Y(A)^2 + FD_Z(A)^2. Neumann BC. FIXME: This does not respect the BC of the original grid!
 *
 */

__global__ void rgrid_cuda_fd_gradient_dot_gradient_gpu(CUREAL *a, CUREAL *b, CUREAL inv_delta, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  CUREAL tmp;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  b[idx] = 0.0;
  if(i != 0 && i != nx-1) {
    tmp = inv_delta * (a[((i+1)*ny + j)*nzz + k] - a[((i-1)*ny + j)*nzz + k]);
    b[idx] += tmp * tmp;
  }
  if(j != 0 && j != ny-1) {
    tmp = inv_delta * (a[(i*ny + j + 1)*nzz + k] - a[(i*ny + j - 1)*nzz + k]);
    b[idx] += tmp * tmp;
  }
  if(k != 0 && k != nz-1) {
    tmp = inv_delta * (a[(i*ny + j)*nzz + k + 1] - a[(i*ny + j)*nzz + k - 1]);
    b[idx] += tmp * tmp;
  }
}

/*
 * B = FD_X(A)^2 + FD_Y(A)^2 + FD_Z(A)^2.
 *
 * grid1    = Source 1 (a) for operation (REAL *; input).
 * grid2    = Source 2 (b) for operation (REAL *; input).
 * inv_delta2 = 1/(4h^2) (REAL; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 * Returns the value of integral.
 *
 */

extern "C" void rgrid_cuda_fd_gradient_dot_gradientW(CUREAL *grida, CUREAL *gridb, CUREAL inv_delta2, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_fd_gradient_dot_gradient_gpu<<<blocks,threads>>>(grida, gridb, inv_delta2, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Real to complex_re.
 *
 * A.re = B(real).
 *
 */

__global__ void grid_cuda_real_to_complex_re_gpu(CUCOMPLEX *a, CUREAL *b, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, idx2;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;    // Index for complex grid
  idx2 = (i * ny + j) * nzz + k;  // Index for real grid

  a[idx] = CUMAKE(b[idx2], 0.0);
}

/*
 * Real to complex_re
 *
 * dest    = Destination for operation (REAL complex *; output).
 * source  = Source for operation (REAL *; input).
 * nx      = # of points along x (INT).
 * ny      = # of points along y (INT).
 * nz      = # of points along z (INT).
 *
 */

extern "C" void grid_cuda_real_to_complex_reW(CUCOMPLEX *gridb, CUREAL *grida, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  grid_cuda_real_to_complex_re_gpu<<<blocks,threads>>>(gridb, grida, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Real to complex_im.
 *
 * A.im = B(real).
 *
 */

__global__ void grid_cuda_real_to_complex_im_gpu(CUCOMPLEX *a, CUREAL *b, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, idx2;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  idx2 = (i * ny + j) * nzz + k;

  a[idx] = CUMAKE(0.0, b[idx2]);
}

/*
 * Real to complex_im
 *
 * dest    = Destination for operation (REAL complex *; output).
 * source  = Source for operation (REAL *; input).
 * nx      = # of points along x (INT).
 * ny      = # of points along y (INT).
 * nz      = # of points along z (INT).
 *
 */

extern "C" void grid_cuda_real_to_complex_imW(CUCOMPLEX *gridb, CUREAL *grida, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  grid_cuda_real_to_complex_im_gpu<<<blocks,threads>>>(gridb, grida, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Add real to complex_re.
 *
 * A.re = A.re + B(real).
 *
 */

__global__ void grid_cuda_add_real_to_complex_re_gpu(CUCOMPLEX *a, CUREAL *b, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, idx2;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  idx2 = (i * ny + j) * nzz + k;

  a[idx].x += b[idx2];
}

/*
 * Add real to complex.re
 *
 * dest    = Destination for operation (REAL complex *; output).
 * source  = Source for operation (REAL *; input).
 * nx      = # of points along x (INT).
 * ny      = # of points along y (INT).
 * nz      = # of points along z (INT).
 *
 */

extern "C" void grid_cuda_add_real_to_complex_reW(CUCOMPLEX *gridb, CUREAL *grida, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  grid_cuda_add_real_to_complex_re_gpu<<<blocks,threads>>>(gridb, grida, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Add real to complex_im.
 *
 * A.im = A.im + B(real).
 *
 */

__global__ void grid_cuda_add_real_to_complex_im_gpu(CUCOMPLEX *a, CUREAL *b, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, idx2;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  idx2 = (i * ny + j) * nzz + k;

  a[idx].y += b[idx2];
}

/*
 * Add real to complex_im
 *
 * dest    = Destination for operation (REAL complex *; output).
 * source  = Source for operation (REAL *; input).
 * nx      = # of points along x (INT).
 * ny      = # of points along y (INT).
 * nz      = # of points along z (INT).
 *
 */

extern "C" void grid_cuda_add_real_to_complex_imW(CUCOMPLEX *gridb, CUREAL *grida, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  grid_cuda_add_real_to_complex_im_gpu<<<blocks,threads>>>(gridb, grida, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Product A(complex) and B(real).
 *
 * A = A * B(real).
 *
 */

__global__ void grid_cuda_product_complex_with_real_gpu(CUCOMPLEX *a, CUREAL *b, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, idx2;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  idx2 = (i * ny + j) * nzz + k;

  a[idx] = a[idx] * b[idx2];
}

/*
 * Product A(complex) with B(real).
 *
 * dest    = Destination for operation (REAL complex *; output).
 * source  = Source for operation (REAL *; input).
 * nx      = # of points along x (INT).
 * ny      = # of points along y (INT).
 * nz      = # of points along z (INT).
 *
 */

extern "C" void grid_cuda_product_complex_with_realW(CUCOMPLEX *gridb, CUREAL *grida, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  grid_cuda_product_complex_with_real_gpu<<<blocks,threads>>>(gridb, grida, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Imag. part to real grid.
 *
 * A(real) = B.im;
 *
 */

__global__ void grid_cuda_complex_im_to_real_gpu(CUREAL *a, CUCOMPLEX *b, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, idx2;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  idx2 = (i * ny + j) * nzz + k;

  a[idx2] = b[idx].y;
}

/*
 * Imag. part of B to real A.
 *
 * dest    = Destination for operation (REAL complex *; output).
 * source  = Source for operation (REAL *; input).
 * nx      = # of points along x (INT).
 * ny      = # of points along y (INT).
 * nz      = # of points along z (INT).
 *
 */

extern "C" void grid_cuda_complex_im_to_realW(CUREAL *gridb, CUCOMPLEX *grida, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  grid_cuda_complex_im_to_real_gpu<<<blocks,threads>>>(gridb, grida, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Real part to real grid.
 *
 * A(real) = B.re;
 *
 */

__global__ void grid_cuda_complex_re_to_real_gpu(CUREAL *a, CUCOMPLEX *b, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, idx2;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  idx2 = (i * ny + j) * nzz + k;

  a[idx2] = b[idx].x;
}

/*
 * Real part of B to real A.
 *
 * dest    = Destination for operation (REAL complex *; output).
 * source  = Source for operation (REAL *; input).
 * nx      = # of points along x (INT).
 * ny      = # of points along y (INT).
 * nz      = # of points along z (INT).
 *
 */

extern "C" void grid_cuda_complex_re_to_realW(CUREAL *gridb, CUCOMPLEX *grida, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  grid_cuda_complex_re_to_real_gpu<<<blocks,threads>>>(gridb, grida, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Maximum value in a grid.
 *
 * No need to run in parallel. Implemented only to avoid memory copy.
 *
 */

__global__ void grid_cuda_max_gpu(CUREAL *a, CUREAL *val, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  /* blockIdx.x = i, threadIdx.x = j */
  INT i, j, k, idx;

  *val = a[0];
  for (i = 0; i < nx; i++)
    for (j = 0; j < ny; j++)
      for(k = 0; k < nz; k++) {
        idx = (i * ny + j) * nzz + k;
        if(a[idx] > *val) *val = a[idx];
      }
}

/*
 * Maximum value contained in a grid. (not parallel)
 *
 * grid    = Source for operation (REAL complex *; input).
 * nx      = # of points along x (INT).
 * ny      = # of points along y (INT).
 * nz      = # of points along z (INT).
 *
 */

extern "C" void grid_cuda_maxW(CUREAL *grid, INT nx, INT ny, INT nz) {

  grid_cuda_max_gpu<<<1,1>>>(grid, (CUREAL *) grid_gpu_mem_addr, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Minimum value in a grid.
 *
 * No need to run in parallel. Implemented only to avoid memory copy.
 *
 */

__global__ void grid_cuda_min_gpu(CUREAL *a, CUREAL *val, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */

  /* blockIdx.x = i, threadIdx.x = j */
  INT i, j, k, idx;

  *val = a[0];
  for (i = 0; i < nx; i++)
    for (j = 0; j < ny; j++)
      for(k = 0; k < nz; k++) {
        idx = (i * ny + j) * nzz + k;
        if(a[idx] < *val) *val = a[idx];
      }
}

/*
 * Minimum value contained in a grid. (not parallel)
 *
 * grid    = Source for operation (REAL complex *; input).
 * nx      = # of points along x (INT).
 * ny      = # of points along y (INT).
 * nz      = # of points along z (INT).
 *
 */

extern "C" void grid_cuda_minW(CUREAL *grid, INT nx, INT ny, INT nz) {

  grid_cuda_min_gpu<<<1,1>>>(grid, (CUREAL *) grid_gpu_mem_addr, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * |rot|
 *
 */

__global__ void rgrid_cuda_abs_rot_gpu(CUREAL *rot, CUREAL *fx, CUREAL *fy, CUREAL *fz, INT nx, INT ny, INT nz, INT nzz, CUREAL inv_delta) {  /* Exectutes at GPU */
 
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  CUREAL tmp;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  if(i == 0 || i == nx-1 || j == 0 || j == ny-1 || k == 0 || k == nz-1) rot[idx] = 0.0;
  else {
    /* x: (d/dy) fz - (d/dz) fy */
    tmp = inv_delta * ((fz[(i * ny + j + 1) * nzz + k] - fz[(i * ny + j - 1)*nzz + k]) 
                      - (fy[(i * ny + j)*nzz + k + 1] - fy[(i * ny + j)*nzz + k - 1]));
    rot[idx] = tmp * tmp;
    /* y: (d/dz) fx - (d/dx) fz */
    tmp = inv_delta * ((fx[(i * ny + j) * nzz + k + 1] - fx[(i * ny + j)*nzz + k - 1]) 
                      - (fz[((i + 1) * ny + j)*nzz + k] - fz[((i - 1) * ny + j)*nzz + k]));
    rot[idx] += tmp * tmp;
    /* z: (d/dx) fy - (d/dy) fx */
    tmp = inv_delta * ((fy[((i + 1) * ny + j) * nzz + k] - fy[((i - 1) * ny + j)*nzz + k]) 
                     - (fx[(i * ny + j + 1)*nzz + k] - fx[(i * ny + j - 1)*nzz + k]));
    rot[idx] += tmp * tmp;
    rot[idx] = SQRT(rot[idx]);
  }
}

/*
 * |rot|
 *
 * rot     = Grid to be operated on (CUREAL *; input/output).
 * fx      = x component of the field (rgrid *).
 * fy      = y component of the field (rgrid *).
 * fz      = z component of the field (rgrid *).
 * nx      = # of points along x (INT).
 * ny      = # of points along y (INT).
 * nz      = # of points along z (INT).
 *
 */

extern "C" void rgrid_cuda_abs_rotW(CUREAL *rot, CUREAL *fx, CUREAL *fy, CUREAL *fz, INT nx, INT ny, INT nz, CUREAL inv_delta) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_abs_rot_gpu<<<blocks,threads>>>(rot, fx, fy, fz, nx, ny, nz, 2 * (nz / 2 + 1), inv_delta);
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Grid integer power device code. This cannot not be called directly.
 *
 * A = POW(B,n) but with n integer.
 *
 */

__global__ void rgrid_cuda_ipower_gpu(CUREAL *a, CUREAL *b, INT n, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  INT ii, sig;
  CUREAL value = 1.0;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  if(n == 0) {
    a[idx] = 1.0;
    return;
  }
  sig = (n < 0) ? -1:1;
  n = ABS(n);
  switch(n) {
    case 1:      
      a[idx] = b[idx];
      break;
    case 2:
      a[idx] = b[idx] * b[idx];
      break;
    case 3:
      a[idx] = b[idx] * b[idx] * b[idx];
      break;
    default:
      for(ii = 0; ii < n; ii++)
        value *= b[idx];
      a[idx] = value;
  }
  if(sig == -1) a[idx] = 1.0 / a[idx];
}

/*
 * Grid integer power.
 *
 * gridb    = Destination for operation (REAL *; output).
 * grida    = Source for operation (REAL *; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 */

extern "C" void rgrid_cuda_ipowerW(CUREAL *gridb, CUREAL *grida, INT exponent, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_ipower_gpu<<<blocks,threads>>>(gridb, grida, exponent, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/

/********************************************************************************************************************/

/*
 * Grid threshold clear device code.
 *
 */

__global__ void rgrid_cuda_threshold_clear_gpu(CUREAL *dest, CUREAL *src, CUREAL ul, CUREAL ll, CUREAL uval, CUREAL lval, INT nx, INT ny, INT nz, INT nzz) {  /* Exectutes at GPU */
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  if(src[idx] < ll) dest[idx] = lval;
  if(src[idx] > ul) dest[idx] = uval;
}

/*
 * Grid clear based on threshold.
 *
 * dest    = Destination for operation (REAL *; output).
 * src     = Source for operation (REAL *; input).
 * ul      = upper limit threshold for the operation (REAL; input).
 * ll      = lower limit threshold for the operation (REAL; input).
 * uval    = value to set when the upper limit was exceeded (REAL; input).
 * lval    = value to set when the lower limit was exceeded (REAL; input).
 * nx      = # of points along x (INT).
 * ny      = # of points along y (INT).
 * nz      = # of points along z (INT).
 *
 */

extern "C" void rgrid_cuda_threshold_clearW(CUREAL *dest, CUREAL *src, CUREAL ul, CUREAL ll, CUREAL uval, CUREAL lval, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  rgrid_cuda_threshold_clear_gpu<<<blocks,threads>>>(dest, src, ul, ll, uval, lval, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/
