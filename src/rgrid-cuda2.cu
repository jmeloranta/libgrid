#include "hip/hip_runtime.h"
/*
 * CUDA device code (REAL; rgrid).
 *
 * blockDim = # of threads
 * gridDim = # of blocks
 *
 * nzz: 2 * (nz / 2 + 1) for real space
 *      (nz / 2 + 1) for reciprocal space
 *
 * x, y, z: split along x for GPUs in real space   (uses nnx1, nnx2)
 *          split along y for GPUs in reciprocal space (uses nny1, nny2)
 *
 */

#include <stdio.h>
#include <cuda/hip/hip_runtime_api.h>
#include <cuda/hip/hip_runtime.h>
#include <cuda/>
#include <cuda/hipfft/hipfft.h>
#include <cuda/hipfft/hipfftXt.h>
#include "cuda-math.h"
#include "rgrid_bc-cuda.h"
#include "cuda-vars.h"

extern void *grid_gpu_mem;
extern cudaXtDesc *grid_gpu_mem_addr;
extern "C" void cuda_error_check();

/*
 *
 * dst = src1 * src2 but with alternating signs for FFT.
 *
 * Fourier space.
 *
 */

__global__ void rgrid_cuda_fft_convolute_gpu(CUCOMPLEX *dst, CUCOMPLEX *src1, CUCOMPLEX *src2, CUREAL norm, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  if((i + j + k) & 1) norm *= -1.0;
  dst[idx] = norm * src1[idx] * src2[idx];
}

/*
 * Convolution in the Fourier space (data in GPU). Not called directly.
 *
 * Multiplication in GPU memory: grid_gpu_mem[i] = grid_gpu_mem[i] * grid_gpu_mem[j] (with sign variation).
 * Note: this includes the sign variation needed for convolution as well as normalization!
 *
 * dst   = output (cudaXtDesc *; output).
 * src1  = 1st grid to be convoluted (cudaXtDesc *; input).
 * src2  = 2nd grid to be convoluted (cudaXtDesc *; input).
 * norm  = FFT norm (CUREAL; input).
 * nx    = Grid dim x (INT; input).
 * ny    = Grid dim y (INT; input).
 * nz    = Grid dim z (INT; input).
 *
 * In Fourier space.
 *
 */

extern "C" void rgrid_cuda_fft_convoluteW(cudaXtDesc *dst, cudaXtDesc *src1, cudaXtDesc *src2, CUREAL norm, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_RECIPROCAL(dst);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_fft_convolute_gpu<<<blocks1,threads>>>((CUCOMPLEX *) dst->data[i], (CUCOMPLEX *) src1->data[i], (CUCOMPLEX *) src2->data[i], norm, nx, nny1, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_fft_convolute_gpu<<<blocks2,threads>>>((CUCOMPLEX *) dst->data[i], (CUCOMPLEX *) src1->data[i], (CUCOMPLEX *) src2->data[i], norm, nx, nny2, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = src^x
 *
 */

__global__ void rgrid_cuda_power_gpu(CUREAL *dst, CUREAL *src, CUREAL x, INT nx, INT ny, INT nz, INT nzz) {
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = POW(src[idx], x);
}

/*
 * Grid power.
 *
 * dst      = Destination for operation (cudaXtDesc *; output).
 * src      = Source for operation (cudaXtDesc *; input).
 * exponent = Exponent (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 * Real space.
 *
 */

extern "C" void rgrid_cuda_powerW(cudaXtDesc *dst, cudaXtDesc *src, CUREAL exponent, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_REAL(dst);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_power_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src->data[i], exponent, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_power_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src->data[i], exponent, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = |src|^x
 *
 */

__global__ void rgrid_cuda_abs_power_gpu(CUREAL *dst, CUREAL *src, CUREAL x, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = POW(FABS(src[idx]), x);
}

/*
 * Grid abs power.
 *
 * dst      = Destination for operation (cudaXtDesc *; output).
 * src      = Source for operation (cudaXtDesc *; input).
 * exponent = Exponent (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_abs_powerW(cudaXtDesc *dst, cudaXtDesc *src, CUREAL exponent, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_REAL(dst);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_abs_power_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src->data[i], exponent, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_abs_power_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src->data[i], exponent, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = c * dst
 *
 */

__global__ void rgrid_cuda_multiply_gpu(CUREAL *dst, CUREAL c, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;

  if(i >= nx || j >= ny || k >= nz) return;

  dst[(i * ny + j) * nzz + k] *= c;
}

/*
 * Multiply grid by a constant.
 *
 * dst      = Grid to be operated on (cudaXtDesc *; input/output).
 * c        = Multiplying constant (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_multiplyW(cudaXtDesc *grid, CUREAL c, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_REAL(grid);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_multiply_gpu<<<blocks1,threads>>>((CUREAL *) grid->data[i], c, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_multiply_gpu<<<blocks2,threads>>>((CUREAL *) grid->data[i], c, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = c * dst (in Fourier space)
 *
 */

__global__ void rgrid_cuda_multiply_fft_gpu(CUCOMPLEX *dst, CUREAL c, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;

  dst[idx] = dst[idx] * c;
}

/*
 * Multiply (complex) grid by a constant (in Fourier space).
 *
 * st       = Grid to be operated on (cudaXtDesc *; input/output).
 * c        = Multiplying constant (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_multiply_fftW(cudaXtDesc *grid, CUREAL c, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_RECIPROCAL(grid);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_multiply_fft_gpu<<<blocks1,threads>>>((CUCOMPLEX *) grid->data[i], c, nx, nny1, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_multiply_fft_gpu<<<blocks2,threads>>>((CUCOMPLEX *) grid->data[i], c, nx, nny2, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = src1 + src2
 *
 */

__global__ void rgrid_cuda_sum_gpu(CUREAL *dst, CUREAL *src1, CUREAL *src2, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = src1[idx] + src2[idx];
}

/*
 * Sum of two grids.
 *
 * dst      = Destination grid (cudaXtDesc *; output).
 * src1     = Input grid 1 (cudaXtDesc *; input).
 * src2     = Input grid 2 (cudaXtDesc *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_sumW(cudaXtDesc *dst, cudaXtDesc *src1, cudaXtDesc *src2, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_REAL(dst);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_sum_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_sum_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = src1 - src2
 *
 */

__global__ void rgrid_cuda_difference_gpu(CUREAL *dst, CUREAL *src1, CUREAL *src2, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = src1[idx] - src2[idx];
}

/*
 * Subtract two grids.
 *
 * dst      = Destination grid (cudaXtDesc *; output).
 * src1     = Input grid 1 (cudaXtDesc *; input).
 * src2     = Input grid 2 (cudaXtDesc *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_differenceW(cudaXtDesc *dst, cudaXtDesc *src1, cudaXtDesc *src2, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_REAL(dst);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_difference_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_difference_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = src1 * src2.
 *
 */

__global__ void rgrid_cuda_product_gpu(CUREAL *dst, CUREAL *src1, CUREAL *src2, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = src1[idx] * src2[idx];
}

/*
 * Product of two grids.
 *
 * dst      = Destination grid (cudaXtDesc *; output).
 * src1     = Source grid 1 (cudaXtDesc *; input).
 * src2     = Source grid 2 (cudaXtDesc *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_productW(cudaXtDesc *dst, cudaXtDesc *src1, cudaXtDesc *src2, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_REAL(dst);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_product_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_product_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = src1 / src2.
 *
 */

__global__ void rgrid_cuda_division_gpu(CUREAL *dst, CUREAL *src1, CUREAL *src2, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = src1[idx] / src2[idx];
}

/*
 * Division of two grids.
 *
 * dst      = Destination grid (cudaXtDesc *; output).
 * src1     = Source grid 1 (cudaXtDesc *; input).
 * src2     = Source grid 2 (cudaXtDesc *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_divisionW(cudaXtDesc *dst, cudaXtDesc *src1, cudaXtDesc *src2, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_REAL(dst);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_division_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_division_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = src1 / (src2 + eps).
 *
 */

__global__ void rgrid_cuda_division_eps_gpu(CUREAL *dst, CUREAL *src1, CUREAL *src2, CUREAL eps, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = src1[idx] / (src2[idx] + eps);
}

/*
 * Division of two grids.
 *
 * dst      = Destination grid (cudaXtDesc *; output).
 * src1     = Source grid 1 (cudaXtDesc *; input).
 * src2     = Source grid 2 (cudaXtDesc *; input).
 * eps      = Epsilon (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_division_epsW(cudaXtDesc *dst, cudaXtDesc *src1, cudaXtDesc *src2, CUREAL eps, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_REAL(dst);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_division_eps_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], eps, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_division_eps_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], eps, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = dst + c
 *
 */

__global__ void rgrid_cuda_add_gpu(CUREAL *dst, CUREAL c, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] += c;
}

/*
 * Add constant to grid.
 *
 * dst      = Grid to be operated on (cudaXtDesc *; input/output).
 * c        = Constant (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_addW(cudaXtDesc *grid, CUREAL c, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_REAL(grid);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_add_gpu<<<blocks1,threads>>>((CUREAL *) grid->data[i], c, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_add_gpu<<<blocks2,threads>>>((CUREAL *) grid->data[i], c, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = cm * dst + ca
 *
 */

__global__ void rgrid_cuda_multiply_and_add_gpu(CUREAL *dst, CUREAL cm, CUREAL ca, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = dst[idx] * cm + ca;
}

/*
 * Grid multiply and add.
 *
 * dst      = Grid to be operated on (cudaXtDesc *; input/output).
 * cm       = Multiplier (CUREAL; input).
 * ca       = Additive constant (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_multiply_and_addW(cudaXtDesc *grid, CUREAL cm, REAL ca, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_REAL(grid);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_multiply_and_add_gpu<<<blocks1,threads>>>((CUREAL *) grid->data[i], cm, ca, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_multiply_and_add_gpu<<<blocks2,threads>>>((CUREAL *) grid->data[i], cm, ca, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = cm * (dst + ca)
 *
 */

__global__ void rgrid_cuda_add_and_multiply_gpu(CUREAL *dst, CUREAL ca, CUREAL cm, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = (dst[idx] + ca) * cm;
}

/*
 * Grid multiply and add.
 *
 * dst      = Grid to be operated on (cudaXtDesc *; input/output).
 * cm       = Multiplier (CUREAL; input).
 * ca       = Additive constant (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_add_and_multiplyW(cudaXtDesc *dst, CUREAL ca, CUREAL cm, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_REAL(dst);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_add_and_multiply_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], ca, cm, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) { // Partial sets
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_add_and_multiply_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], ca, cm, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = dst + d * src
 *
 */

__global__ void rgrid_cuda_add_scaled_gpu(CUREAL *dst, CUREAL d, CUREAL *src, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = dst[idx] + src[idx] * d;
}

/*
 * Scaled add grid.
 *
 * dst      = Destination for operation (cudaXtDesc *; output).
 * d        = Scaling factor (REAL; input).
 * src      = Source for operation (cudaXtDesc *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_add_scaledW(cudaXtDesc *dst, CUREAL d, cudaXtDesc *src, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_REAL(dst);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_add_scaled_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], d, (CUREAL *) src->data[i], nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_add_scaled_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], d, (CUREAL *) src->data[i], nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = dst + d * src1 * src2
 *
 */

__global__ void rgrid_cuda_add_scaled_product_gpu(CUREAL *dst, CUREAL d, CUREAL *src1, CUREAL *src2, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = dst[idx] + d * src1[idx] * src2[idx];
}

/*
 * Add scaled product.
 *
 * dst      = Destination for operation (cudaXtDesc *; output).
 * d        = Scaling factor (REAL; input).
 * src1     = Source for operation (cudaXtDesc *; input).
 * src2     = Source for operation (cudaXtDesc *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_add_scaled_productW(cudaXtDesc *dst, CUREAL d, cudaXtDesc *src1, cudaXtDesc *src2, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_REAL(dst);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_add_scaled_product_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], d, (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_add_scaled_product_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], d, (CUREAL *) src1->data[i], (CUREAL *) src2->data[i], nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 *
 * dst = c
 *
 */

__global__ void rgrid_cuda_constant_gpu(CUREAL *dst, CUREAL c, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = c;
}

/*
 * Set grid to constant.
 *
 * dst      = Destination for operation (cudaXtDesc *; output).
 * c        = Constant (REAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_constantW(cudaXtDesc *dst, CUREAL c, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_REAL(dst);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_constant_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], c, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_constant_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], c, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 * Block init (zero elements).
 *
 * blocks  = Block table (CUREAL *; output).
 * nblocks = Number of blocks in table (INT; input).
 * 
 */

__global__ void rgrid_cuda_block_init(CUREAL *blocks, INT nblocks) {

  INT i;

  for(i = 0; i < nblocks; i++) blocks[i] = 0.0;
}

/*
 * Block reduction.
 *
 * blocks  = Block list to reduce (CUREAL *; input/output). blocks[0] will contain the reduced value.
 * nblocks = Number of blocks (INT; input).
 *
 */

__global__ void rgrid_cuda_block_reduce(CUREAL *blocks, INT nblocks) {

  INT i;

  for(i = 1; i < nblocks; i++)
    blocks[0] += blocks[i];  // reduce blocks
}

/*
 * Integrate over A.
 *
 */

__global__ void rgrid_cuda_integral_gpu(CUREAL *a, CUREAL *blocks, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2, t;
  extern __shared__ CUREAL els[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els[t] = 0.0;
  }
  __syncthreads();

  idx = (i * ny + j) * nzz + k;
  idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;

  els[idx2] += a[idx];
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] += els[t];  // reduce threads
    }
  }
}

/*
 * Integrate over grid.
 *
 * grid     = Source for operation (cudaXtDesc *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 * value    = Result (CUCOMPLEX *; output).
 *
 */

extern "C" void rgrid_cuda_integralW(cudaXtDesc *grid, INT nx, INT ny, INT nz, CUREAL *value) {

  SETUP_VARIABLES_REAL(grid);
  CUREAL tmp;
  INT s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b31 = blocks1.x * blocks1.y * blocks1.z, b32 = blocks2.x * blocks2.y * blocks2.z;
  extern int cuda_get_element(void *, int, size_t, size_t, void *);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b31);
    cuda_error_check();
    // Blocks, Threads, dynamic memory size
    rgrid_cuda_integral_gpu<<<blocks1,threads,s*sizeof(CUREAL)>>>((CUREAL *) grid->data[i], (CUREAL *) grid_gpu_mem_addr->data[i], nnx1, ny, nz, nzz);
    cuda_error_check();
    rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b31);
    cuda_error_check();
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b32);
    cuda_error_check();
    // Blocks, Threads, dynamic memory size
    rgrid_cuda_integral_gpu<<<blocks2,threads,s*sizeof(CUREAL)>>>((CUREAL *) grid->data[i], (CUREAL *) grid_gpu_mem_addr->data[i], nnx2, ny, nz, nzz);
    cuda_error_check();
    rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b32);
    cuda_error_check();
  }

  // Reduce over GPUs
  *value = 0.0;
  for(i = 0; i < ngpu2; i++) {
    cuda_get_element(grid_gpu_mem, grid->GPUs[i], 0, sizeof(CUREAL), &tmp);
    *value = *value + tmp;
  }
  cuda_error_check();
}

/*
 * Integrate over A with limits.
 *
 */

__global__ void rgrid_cuda_integral_region_gpu(CUREAL *a, CUREAL *blocks, INT il, INT iu, INT jl, INT ju, INT kl, INT ku, INT nx, INT ny, INT nz, INT nzz, INT seg) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, ii = i + seg;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2, t;
  extern __shared__ CUREAL els[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els[t] = 0.0;
  }
  __syncthreads();

  if(ii >= il && ii <= iu && j >= jl && j <= ju && k >= kl && k <= ku) {
    idx = (i * ny + j) * nzz + k;
    idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
    els[idx2] += a[idx];
  }
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] += els[t];  // reduce threads
    }
  }
}

/*
 * Integrate over grid with limits.
 *
 * grid     = Source for operation (cudaXtDesc *; input).
 * il       = Lower index for x (INT; input).
 * iu       = Upper index for x (INT; input).
 * jl       = Lower index for y (INT; input).
 * ju       = Upper index for y (INT; input).
 * kl       = Lower index for z (INT; input).
 * ku       = Upper index for z (INT; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 * value    = Result (CUCOMPLEX *; output).
 *
 * Returns the value of integral in grid_gpu_mem[0].
 *
 */

extern "C" void rgrid_cuda_integral_regionW(cudaXtDesc *grid, INT il, INT iu, INT jl, INT ju, INT kl, INT ku, INT nx, INT ny, INT nz, CUREAL *value) {

  SETUP_VARIABLES_REAL(grid);
  CUREAL tmp;
  INT seg = 0, s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b31 = blocks1.x * blocks1.y * blocks1.z;
  INT b32 = blocks2.x * blocks2.y * blocks2.z;
  extern int cuda_get_element(void *, int, size_t, size_t, void *);

  if(il < 0) il = 0;  
  if(jl < 0) jl = 0;  
  if(kl < 0) kl = 0;  
  if(iu > nx-1) iu = nx-1;
  if(ju > ny-1) ju = ny-1;
  if(ku > nz-1) ku = nz-1;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b31);
    cuda_error_check();
    // Blocks, Threads, dynamic memory size
    rgrid_cuda_integral_region_gpu<<<blocks1,threads,s*sizeof(CUREAL)>>>((CUREAL *) grid->data[i], (CUREAL *) grid_gpu_mem_addr->data[i], 
                                    il, iu, jl, ju, kl, ku, nnx1, ny, nz, nzz, seg);
    seg += nnx1;
    cuda_error_check();
    rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b31);
    cuda_error_check();
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b32);
    cuda_error_check();
    // Blocks, Threads, dynamic memory size
    rgrid_cuda_integral_region_gpu<<<blocks2,threads,s*sizeof(CUREAL)>>>((CUREAL *) grid->data[i], (CUREAL *) grid_gpu_mem_addr->data[i], 
                                    il, iu, jl, ju, kl, ku, nnx2, ny, nz, nzz, seg);
    seg += nnx2;
    cuda_error_check();
    rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b32);
    cuda_error_check();
  }

  // Reduce over GPUs
  *value = 0.0;
  for(i = 0; i < ngpu2; i++) {
    cuda_get_element(grid_gpu_mem, grid->GPUs[i], 0, sizeof(CUREAL), &tmp);
    *value = *value + tmp;
  }
  cuda_error_check();
}

/*
 * Integrate of A^2.
 *
 */

__global__ void rgrid_cuda_integral_of_square_gpu(CUREAL *a, CUREAL *blocks, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2, t;
  extern __shared__ CUREAL els[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els[t] = 0.0;
  }
  __syncthreads();

  idx = (i * ny + j) * nzz + k;
  idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;

  els[idx2] += a[idx] * a[idx];
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] += els[t];  // reduce threads
    }
  }
}

/*
 * Integral of square.
 *
 * grid     = Source for operation (cudaXtDesc *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 * value    = Result (CUCOMPLEX *; output).
 *
 */

extern "C" void rgrid_cuda_integral_of_squareW(cudaXtDesc *grid, INT nx, INT ny, INT nz, CUREAL *value) {

  SETUP_VARIABLES_REAL(grid);
  CUREAL tmp;
  INT s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b31 = blocks1.x * blocks1.y * blocks1.z, b32 = blocks2.x * blocks2.y * blocks2.z;
  extern int cuda_get_element(void *, int, size_t, size_t, void *);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b31);
    cuda_error_check();
    // Blocks, Threads, dynamic memory size
    rgrid_cuda_integral_of_square_gpu<<<blocks1,threads,s*sizeof(CUREAL)>>>((CUREAL *) grid->data[i], (CUREAL *) grid_gpu_mem_addr->data[i], nnx1, ny, nz, nzz);
    cuda_error_check();
    rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b31);
    cuda_error_check();
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b32);
    cuda_error_check();
    // Blocks, Threads, dynamic memory size
    rgrid_cuda_integral_of_square_gpu<<<blocks2,threads,s*sizeof(CUREAL)>>>((CUREAL *) grid->data[i], (CUREAL *) grid_gpu_mem_addr->data[i], nnx2, ny, nz, nzz);
    cuda_error_check();
    rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b32);
    cuda_error_check();
  }

  // Reduce over GPUs
  *value = 0.0;
  for(i = 0; i < ngpu2; i++) {
    cuda_get_element(grid_gpu_mem, grid->GPUs[i], 0, sizeof(CUREAL), &tmp);
    *value = *value + tmp;
  }
  cuda_error_check();
}

/*
 * Integrate A * B.
 *
 */

__global__ void rgrid_cuda_integral_of_product_gpu(CUREAL *a, CUREAL *b, CUREAL *blocks, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2, t;
  extern __shared__ CUREAL els[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els[t] = 0.0;
  }
  __syncthreads();

  idx = (i * ny + j) * nzz + k;
  idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;

  els[idx2] += a[idx] * b[idx];
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] += els[t];  // reduce threads
    }
  }
}

/*
 * Integral of product.
 *
 * grid1    = Source 1 for operation (cudaXtDesc *; input).
 * grid2    = Source 2 for operation (cudaXtDesc *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 * value    = Return value (CUREAL *; output).
 *
 */

extern "C" void rgrid_cuda_integral_of_productW(cudaXtDesc *grid1, cudaXtDesc *grid2, INT nx, INT ny, INT nz, CUREAL *value) {

  SETUP_VARIABLES_REAL(grid1);
  CUREAL tmp;
  INT s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b31 = blocks1.x * blocks1.y * blocks1.z, b32 = blocks2.x * blocks2.y * blocks2.z;
  extern int cuda_get_element(void *, int, size_t, size_t, void *);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(grid1->GPUs[i]);
    rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b31);
    cuda_error_check();
    // Blocks, Threads, dynamic memory size
    rgrid_cuda_integral_of_product_gpu<<<blocks1,threads,s*sizeof(CUREAL)>>>((CUREAL *) grid1->data[i], (CUREAL *) grid2->data[i], 
                                                                             (CUREAL *) grid_gpu_mem_addr->data[i], nnx1, ny, nz, nzz);
    cuda_error_check();
    rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b31);
    cuda_error_check();
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(grid1->GPUs[i]);
    rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b32);
    cuda_error_check();
    // Blocks, Threads, dynamic memory size
    rgrid_cuda_integral_of_product_gpu<<<blocks2,threads,s*sizeof(CUREAL)>>>((CUREAL *) grid1->data[i], (CUREAL *) grid2->data[i], 
                                                                             (CUREAL *) grid_gpu_mem_addr->data[i], nnx2, ny, nz, nzz);
    cuda_error_check();
    rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b32);
    cuda_error_check();
  }

  // Reduce over GPUs
  *value = 0.0;
  for(i = 0; i < ngpu2; i++) {
    cuda_get_element(grid_gpu_mem, grid1->GPUs[i], 0, sizeof(CUREAL), &tmp);
    *value = *value + tmp;
  }
  cuda_error_check();
}

/*
 * Integrate A * B^2.
 *
 */

__global__ void rgrid_cuda_grid_expectation_value_gpu(CUREAL *a, CUREAL *b, CUREAL *blocks, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2, t;
  extern __shared__ CUREAL els[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els[t] = 0.0;
  }
  __syncthreads();

  idx = (i * ny + j) * nzz + k;
  idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;

  els[idx2] += a[idx] * b[idx] * b[idx];
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] += els[t];  // reduce threads
    }
  }
}

/*
 * Integral a * b^2.
 *
 * grid1    = Source 1 (a) for operation (cudaXtDesc *; input).
 * grid2    = Source 2 (b) for operation (cudaXtDesc *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 * *value   = Return value (CUREAL *; output).
 *
 */

extern "C" void rgrid_cuda_grid_expectation_valueW(cudaXtDesc *grid1, cudaXtDesc *grid2, INT nx, INT ny, INT nz, CUREAL *value) {

  SETUP_VARIABLES_REAL(grid1);
  CUREAL tmp;
  INT s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b31 = blocks1.x * blocks1.y * blocks1.z, b32 = blocks2.x * blocks2.y * blocks2.z;
  extern int cuda_get_element(void *, int, size_t, size_t, void *);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(grid1->GPUs[i]);
    rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b31);
    cuda_error_check();
    // Blocks, Threads, dynamic memory size
    rgrid_cuda_grid_expectation_value_gpu<<<blocks1,threads,s*sizeof(CUREAL)>>>((CUREAL *) grid1->data[i], (CUREAL *) grid2->data[i], 
                                                                                (CUREAL *) grid_gpu_mem_addr->data[i], nnx1, ny, nz, nzz);
    cuda_error_check();
    rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b31);
    cuda_error_check();
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(grid1->GPUs[i]);
    rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b32);
    cuda_error_check();
    // Blocks, Threads, dynamic memory size
    rgrid_cuda_grid_expectation_value_gpu<<<blocks2,threads,s*sizeof(CUREAL)>>>((CUREAL *) grid1->data[i], (CUREAL *) grid2->data[i], 
                                                                                (CUREAL *) grid_gpu_mem_addr->data[i], nnx2, ny, nz, nzz);
    cuda_error_check();
    rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b32);
    cuda_error_check();
  }

  // Reduce over GPUs
  *value = 0.0;
  for(i = 0; i < ngpu2; i++) {
    cuda_get_element(grid_gpu_mem, grid1->GPUs[i], 0, sizeof(CUREAL), &tmp);
    *value = *value + tmp;
  }
  cuda_error_check();
}

/*
 * B = FD_X(A).
 *
 */

__global__ void rgrid_cuda_fd_gradient_x_gpu(CUREAL *src, CUREAL *dst, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = inv_delta * (rgrid_cuda_bc_x_plus(src, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_x_minus(src, bc, i, j, k, nx, ny, nz, nzz));
}

/*
 * B = FD_X(A)
 *
 * src      = Source for operation (cudaXtDesc *; input).
 * dst      = Destination for operation (cudaXtDesc *; input).
 * inv_delta= 1 / (2 * step) (REAL; input).
 * bc       = Boundary condition: 0 = Dirichlet, 1 = Neumann, 2 = Periodic (char; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_fd_gradient_xW(cudaXtDesc *src, cudaXtDesc *dst, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz) {

  INT nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  if(dst->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(dst->GPUs[0]);
  rgrid_cuda_fd_gradient_x_gpu<<<blocks,threads>>>((CUREAL *) src->data[0], (CUREAL *) dst->data[0], inv_delta, bc, nx, ny, nz, nzz);
  cuda_error_check();
}

/*
 * B = FD_Y(A).
 *
 */

__global__ void rgrid_cuda_fd_gradient_y_gpu(CUREAL *src, CUREAL *dst, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = inv_delta * (rgrid_cuda_bc_y_plus(src, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_y_minus(src, bc, i, j, k, nx, ny, nz, nzz));
}

/*
 * B = FD_Y(A)
 *
 * src      = Source for operation (cudaXtDesc *; input).
 * dst      = Destination for operation (cudaXtDesc *; input).
 * inv_delta= 1 / (2 * step) (REAL; input).
 * bc       = Boundary condition: 0 = Dirichlet, 1 = Neumann, 2 = Periodic (char; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_fd_gradient_yW(cudaXtDesc *src, cudaXtDesc *dst, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz) {

  INT nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  if(dst->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(dst->GPUs[0]);
  rgrid_cuda_fd_gradient_y_gpu<<<blocks,threads>>>((CUREAL *) src->data[0], (CUREAL *) dst->data[0], inv_delta, bc, nx, ny, nz, nzz);
  cuda_error_check();
}

/*
 * B = FD_Z(A).
 *
 */

__global__ void rgrid_cuda_fd_gradient_z_gpu(CUREAL *src, CUREAL *dst, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = inv_delta * (rgrid_cuda_bc_z_plus(src, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_z_minus(src, bc, i, j, k, nx, ny, nz, nzz));
}

/*
 * B = FD_Z(A)
 *
 * src      = Source for operation (cudaXtDesc *; input).
 * dst      = Destination for operation (cudaXtDesc *; input).
 * inv_delta= 1 / (2 * step) (REAL; input).
 * bc       = Boundary condition: 0 = Dirichlet, 1 = Neumann, 2 = Periodic (char; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_fd_gradient_zW(cudaXtDesc *src, cudaXtDesc *dst, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz) {

  INT nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  if(dst->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(dst->GPUs[0]);
  rgrid_cuda_fd_gradient_z_gpu<<<blocks,threads>>>((CUREAL *) src->data[0], (CUREAL *) dst->data[0], inv_delta, bc, nx, ny, nz, nzz);
  cuda_error_check();
}

/*
 * B = LAPLACE(A).
 *
 */

__global__ void rgrid_cuda_fd_laplace_gpu(CUREAL *src, CUREAL *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = inv_delta2 * (rgrid_cuda_bc_x_plus(src, bc, i, j, k, nx, ny, nz, nzz) + rgrid_cuda_bc_x_minus(src, bc, i, j, k, nx, ny, nz, nzz)
                       + rgrid_cuda_bc_y_plus(src, bc, i, j, k, nx, ny, nz, nzz) + rgrid_cuda_bc_y_minus(src, bc, i, j, k, nx, ny, nz, nzz)
                       + rgrid_cuda_bc_z_plus(src, bc, i, j, k, nx, ny, nz, nzz) + rgrid_cuda_bc_z_minus(src, bc, i, j, k, nx, ny, nz, nzz)
                       - 6.0 * src[idx]);
}

/*
 * B = LAPLACE(A)
 *
 * src       = Source for operation (cudaXtDesc *; input).
 * dst       = Destination for operation (cudaXtDesc *; input).
 * inv_delta2= 1 / (2 * step) (REAL; input).
 * bc        = Boundary condition (char; input).
 * nx        = # of points along x (INT; input).
 * ny        = # of points along y (INT; input).
 * nz        = # of points along z (INT; input).
 *
 * Returns the value of integral.
 *
 */

extern "C" void rgrid_cuda_fd_laplaceW(cudaXtDesc *src, cudaXtDesc *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  INT nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  if(dst->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(dst->GPUs[0]);
  rgrid_cuda_fd_laplace_gpu<<<blocks,threads>>>((CUREAL *) src->data[0], (CUREAL *) dst->data[0], inv_delta2, bc, nx, ny, nz, nzz);
  cuda_error_check();
}

/*
 * B = LAPLACE_X(A).
 *
 */

__global__ void rgrid_cuda_fd_laplace_x_gpu(CUREAL *src, CUREAL *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = inv_delta2 * (rgrid_cuda_bc_x_plus(src, bc, i, j, k, nx, ny, nz, nzz) + rgrid_cuda_bc_x_minus(src, bc, i, j, k, nx, ny, nz, nzz)
                         - 2.0 * src[idx]);
}

/*
 * B = LAPLACE_X(A)
 *
 * src        = Source for operation (cudaXtDesc *; input).
 * dst        = Destination for operation (cudaXtDesc *; output).
 * inv_delta2 = 1 / (2 * step) (REAL; input).
 * bc         = Boundary condition (char; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 * Returns laplace in gridb.
 *
 */

extern "C" void rgrid_cuda_fd_laplace_xW(cudaXtDesc *src, cudaXtDesc *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  INT nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  if(dst->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(dst->GPUs[0]);
  rgrid_cuda_fd_laplace_x_gpu<<<blocks,threads>>>((CUREAL *) src->data[0], (CUREAL *) dst->data[0], inv_delta2, bc, nx, ny, nz, nzz);
  cuda_error_check();
}

/*
 * B = LAPLACE_Y(A).
 *
 */

__global__ void rgrid_cuda_fd_laplace_y_gpu(CUREAL *src, CUREAL *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = inv_delta2 * (rgrid_cuda_bc_y_plus(src, bc, i, j, k, nx, ny, nz, nzz) + rgrid_cuda_bc_y_minus(src, bc, i, j, k, nx, ny, nz, nzz)
                         - 2.0 * src[idx]);
}

/*
 * B = LAPLACE_Y(A)
 *
 * src        = Source for operation (cudaXtDesc *; input).
 * dst        = Destination for operation (cudaXtDesc *; output).
 * inv_delta2 = 1 / (2 * step) (CUREAL; input).
 * bc         = Boundary condition (char; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 * Returns laplace in gridb.
 *
 */

extern "C" void rgrid_cuda_fd_laplace_yW(cudaXtDesc *src, cudaXtDesc *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  INT nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  if(dst->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(dst->GPUs[0]);
  rgrid_cuda_fd_laplace_y_gpu<<<blocks,threads>>>((CUREAL *) src->data[0], (CUREAL *) dst->data[0], inv_delta2, bc, nx, ny, nz, nzz);
  cuda_error_check();
}

/*
 * B = LAPLACE_Z(A).
 *
 */

__global__ void rgrid_cuda_fd_laplace_z_gpu(CUREAL *src, CUREAL *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = inv_delta2 * (rgrid_cuda_bc_z_plus(src, bc, i, j, k, nx, ny, nz, nzz) + rgrid_cuda_bc_z_minus(src, bc, i, j, k, nx, ny, nz, nzz)
                         - 2.0 * src[idx]);
}

/*
 * B = LAPLACE_Z(A)
 *
 * src        = Source for operation (cudaXtDesc *; input).
 * dst        = Destination for operation (cudaXtDesc *; output).
 * inv_delta2 = 1 / (2 * step) (CUREAL; input).
 * bc         = Boundary condition (char; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 * Returns laplace in gridb.
 *
 */

extern "C" void rgrid_cuda_fd_laplace_zW(cudaXtDesc *src, cudaXtDesc *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  INT nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  if(dst->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(dst->GPUs[0]);
  rgrid_cuda_fd_laplace_z_gpu<<<blocks,threads>>>((CUREAL *) src->data[0], (CUREAL *) dst->data[0], inv_delta2, bc, nx, ny, nz, nzz);
  cuda_error_check();
}

/*
 * B = FD_X(A)^2 + FD_Y(A)^2 + FD_Z(A)^2.
 *
 */

__global__ void rgrid_cuda_fd_gradient_dot_gradient_gpu(CUREAL *src, CUREAL *dst, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  CUREAL tmp;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  dst[idx] = 0.0;

  tmp = inv_delta * (rgrid_cuda_bc_x_plus(src, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_x_minus(src, bc, i, j, k, nx, ny, nz, nzz));
  dst[idx] = dst[idx] + tmp * tmp;

  tmp = inv_delta * (rgrid_cuda_bc_y_plus(src, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_y_minus(src, bc, i, j, k, nx, ny, nz, nzz));
  dst[idx] = dst[idx] + tmp * tmp;

  tmp = inv_delta * (rgrid_cuda_bc_z_plus(src, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_z_minus(src, bc, i, j, k, nx, ny, nz, nzz));
  dst[idx] = dst[idx] + tmp * tmp;
}

/*
 * B = FD_X(A)^2 + FD_Y(A)^2 + FD_Z(A)^2.
 *
 * src        = Source for operation (cudaXtDesc *; input).
 * dst        = Destination for operation (cudaXtDesc *; output).
 * inv_delta2 = 1 / (4 * step * step) (CUREAL; input).
 * bc         = Boundary condition (char; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_fd_gradient_dot_gradientW(cudaXtDesc *src, cudaXtDesc *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  INT nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  if(dst->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(dst->GPUs[0]);
  rgrid_cuda_fd_gradient_dot_gradient_gpu<<<blocks,threads>>>((CUREAL *) src->data[0], (CUREAL *) dst->data[0], inv_delta2, bc, nx, ny, nz, nzz);
  cuda_error_check();
}

/*
 * Maximum value in a grid.
 *
 */

__global__ void rgrid_cuda_max_gpu(CUREAL *a, CUREAL *val, INT nx, INT ny, INT nz, INT nzz) {

  /* blockIdx.x = i, threadIdx.x = j */
  INT i, j, k, idx;

  *val = a[0];
  for (i = 0; i < nx; i++)
    for (j = 0; j < ny; j++)
      for(k = 0; k < nz; k++) {
        idx = (i * ny + j) * nzz + k;
        if(a[idx] > *val) *val = a[idx];
      }
}

/*
 * Maximum value contained in a grid. (not parallel)
 *
 * grid    = Source for operation (cudaXtDesc complex *; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 * value   = Return value (CUREAL *; output).
 *
 */

extern "C" void grid_cuda_maxW(cudaXtDesc *grid, INT nx, INT ny, INT nz, CUREAL *value) {

  INT i, ngpu2 = grid->nGPUs, ngpu1 = nx % ngpu2, nnx2 = nx / ngpu2, nnx1 = nnx2 + 1, nzz = 2 * (nz / 2 + 1);  CUREAL tmp;
  extern int cuda_get_element(void *, int, size_t, size_t, void *);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_max_gpu<<<1,1>>>((CUREAL *) grid->data[i], (CUREAL *) grid_gpu_mem_addr->data[i], nnx1, ny, nz, nzz);
  }
  cuda_error_check();

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_max_gpu<<<1,1>>>((CUREAL *) grid->data[i], (CUREAL *) grid_gpu_mem_addr->data[i], nnx2, ny, nz, nzz);
  }
  cuda_error_check();

  // Reduce over GPUs
  for(i = 0; i < ngpu2; i++) {
    cuda_get_element(grid_gpu_mem, grid->GPUs[i], 0, sizeof(CUREAL), &tmp);
    if(i == 0) *value = tmp;
    else if(tmp > *value) *value = tmp;    
  }
  cuda_error_check();
}

/*
 * Minimum value in a grid.
 *
 */

__global__ void rgrid_cuda_min_gpu(CUREAL *a, CUREAL *val, INT nx, INT ny, INT nz, INT nzz) {

  /* blockIdx.x = i, threadIdx.x = j */
  INT i, j, k, idx;

  *val = a[0];
  for (i = 0; i < nx; i++)
    for (j = 0; j < ny; j++)
      for(k = 0; k < nz; k++) {
        idx = (i * ny + j) * nzz + k;
        if(a[idx] < *val) *val = a[idx];
      }
}

/*
 * Minimum value contained in a grid. (not parallel)
 *
 * grid    = Source for operation (cudaXtDesc *; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 * value   = Return value (CUREAL *; output).
 *
 */

extern "C" void grid_cuda_minW(cudaXtDesc *grid, INT nx, INT ny, INT nz, CUREAL *value) {

  INT i, ngpu2 = grid->nGPUs, ngpu1 = nx % ngpu2, nnx2 = nx / ngpu2, nnx1 = nnx2 + 1, nzz = 2 * (nz / 2 + 1);
  CUREAL tmp;
  extern int cuda_get_element(void *, int, size_t, size_t, void *);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_min_gpu<<<1,1>>>((CUREAL *) grid->data[i], (CUREAL *) grid_gpu_mem_addr->data[i], nnx1, ny, nz, nzz);
  }
  cuda_error_check();

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_min_gpu<<<1,1>>>((CUREAL *) grid->data[i], (CUREAL *) grid_gpu_mem_addr->data[i], nnx2, ny, nz, nzz);
  }
  cuda_error_check();

  // Reduce over GPUs
  for(i = 0; i < ngpu2; i++) {
    cuda_get_element(grid_gpu_mem, grid->GPUs[i], 0, sizeof(CUREAL), &tmp);
    if(i == 0) *value = tmp;
    else if(tmp > *value) *value = tmp;    
  }
  cuda_error_check();
}

/*
 * |rot|
 *
 */

__global__ void rgrid_cuda_abs_rot_gpu(CUREAL *rot, CUREAL *fx, CUREAL *fy, CUREAL *fz, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz, INT nzz) { 
 
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  CUREAL tmp;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  /* x: (d/dy) fz - (d/dz) fy */
  tmp = inv_delta * (rgrid_cuda_bc_y_plus(fz, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_y_minus(fz, bc, i, j, k, nx, ny, nz, nzz)
                     - rgrid_cuda_bc_z_plus(fy, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_z_minus(fy, bc, i, j, k, nx, ny, nz, nzz));
  rot[idx] = tmp * tmp;

  /* y: (d/dz) fx - (d/dx) fz */
  tmp = inv_delta * (rgrid_cuda_bc_z_plus(fx, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_z_minus(fx, bc, i, j, k, nx, ny, nz, nzz)
                     - rgrid_cuda_bc_x_plus(fz, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_x_minus(fz, bc, i, j, k, nx, ny, nz, nzz));
  rot[idx] = rot[idx] + tmp * tmp;

  /* z: (d/dx) fy - (d/dy) fx */
  tmp = inv_delta * (rgrid_cuda_bc_x_plus(fy, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_x_minus(fy, bc, i, j, k, nx, ny, nz, nzz)
                     - rgrid_cuda_bc_y_plus(fx, bc, i, j, k, nx, ny, nz, nzz) - rgrid_cuda_bc_y_minus(fx, bc, i, j, k, nx, ny, nz, nzz));
  rot[idx] = rot[idx] + tmp * tmp;
  rot[idx] = SQRT(rot[idx]);
}

/*
 * |rot|
 *
 * rot       = Grid to be operated on (cudaXtDesc *; input/output).
 * fx        = x component of the field (cudaXtDesc *; input).
 * fy        = y component of the field (cudaXtDesc *; input).
 * fz        = z component of the field (cudaXtDesc *; input).
 * inv_delta = 1 / (2 * step) (CUREAL; input).
 * bc        = Boundary condition (char; input).
 * nx        = # of points along x (INT; input).
 * ny        = # of points along y (INT; input).
 * nz        = # of points along z (INT; input).
 *
 * TODO: For this it probably makes sense to force transferring the blocks to host memory and do the operation there.
 *
 */

extern "C" void rgrid_cuda_abs_rotW(cudaXtDesc *rot, cudaXtDesc *fx, cudaXtDesc *fy, cudaXtDesc *fz, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz) {

  INT nzz = 2 * (nz / 2 + 1);
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  if(rot->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(rot->GPUs[0]);
  rgrid_cuda_abs_rot_gpu<<<blocks,threads>>>((CUREAL *) rot->data[0], (CUREAL *) fx->data[0], (CUREAL *) fy->data[0], (CUREAL *) fz->data[0], 
                                             inv_delta, bc, nx, ny, nz, nzz);
  cuda_error_check();
}

/*
 * dst = POW(src, n) with n integer.
 *
 */

__global__ void rgrid_cuda_ipower_gpu(CUREAL *dst, CUREAL *src, INT n, INT nx, INT ny, INT nz, INT nzz) {
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, ii, sig;
  CUREAL value = 1.0;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  if(n == 0) {
    dst[idx] = 1.0;
    return;
  }
  sig = (n < 0) ? -1:1;
  n = ABS(n);
  switch(n) {
    case 1:      
      dst[idx] = src[idx];
      break;
    case 2:
      dst[idx] = src[idx] * src[idx];
      break;
    case 3:
      dst[idx] = src[idx] * src[idx] * src[idx];
      break;
    default:
      for(ii = 0; ii < n; ii++)
        value *= src[idx];
      dst[idx] = value;
  }
  if(sig == -1) dst[idx] = 1.0 / dst[idx];
}

/*
 * Grid integer power.
 *
 * dst      = Destination for operation (cudaXtDesc *; output).
 * src      = Source for operation (cudaXtDesc *; input).
 * exponent = Integer exponent (INT; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_ipowerW(cudaXtDesc *dst, cudaXtDesc *src, INT exponent, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_REAL(dst);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_ipower_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src->data[i], exponent, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_ipower_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src->data[i], exponent, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 * Grid threshold clear device code.
 *
 */

__global__ void rgrid_cuda_threshold_clear_gpu(CUREAL *dest, CUREAL *src, CUREAL ul, CUREAL ll, CUREAL uval, CUREAL lval, INT nx, INT ny, INT nz, INT nzz) {
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  if(src[idx] < ll) dest[idx] = lval;
  if(src[idx] > ul) dest[idx] = uval;
}

/*
 * Grid clear based on threshold.
 *
 * dst     = Destination for operation (cudaXtDesc *; output).
 * src     = Source for operation (cudaXtDesc *; input).
 * ul      = upper limit threshold for the operation (REAL; input).
 * ll      = lower limit threshold for the operation (REAL; input).
 * uval    = value to set when the upper limit was exceeded (REAL; input).
 * lval    = value to set when the lower limit was exceeded (REAL; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_threshold_clearW(cudaXtDesc *dst, cudaXtDesc *src, CUREAL ul, CUREAL ll, CUREAL uval, CUREAL lval, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_REAL(dst);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_threshold_clear_gpu<<<blocks1,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src->data[i], ul, ll, uval, lval, nnx1, ny, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(dst->GPUs[i]);
    rgrid_cuda_threshold_clear_gpu<<<blocks2,threads>>>((CUREAL *) dst->data[i], (CUREAL *) src->data[i], ul, ll, uval, lval, nnx2, ny, nz, nzz);
  }

  cuda_error_check();
}

/*
 * Zero part of real grid.
 *
 * A = 0 in the specified range.
 *
 */

__global__ void rgrid_cuda_zero_index_gpu(CUREAL *dst, INT lx, INT hx, INT ly, INT hy, INT lz, INT hz, INT nx, INT ny, INT nz, INT nzz, INT seg) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, ii = i + seg;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nzz + k;

  if(ii >= lx && ii < hx && j >= ly && j < hy && k >= lz && k < hz) dst[idx] = 0.0;
}

/*
 * Zero specified range of complex grid.
 *
 * grid     = Grid to be operated on (cudaXtDesc *; input/output).
 * lx       = Low x index (INT; input). 
 * hx       = Low x index (INT; input). 
 * ly       = Low y index (INT; input). 
 * hy       = Low y index (INT; input). 
 * lz       = Low z index (INT; input). 
 * hx       = Low z index (INT; input). 
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_zero_indexW(cudaXtDesc *grid, INT lx, INT hx, INT ly, INT hy, INT lz, INT hz, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_REAL(grid);
  INT seg = 0;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_zero_index_gpu<<<blocks1,threads>>>((CUREAL *) grid->data[i], lx, hx, ly, hy, lz, hz, nnx1, ny, nz, nzz, seg);
    seg += nnx1;
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_zero_index_gpu<<<blocks2,threads>>>((CUREAL *) grid->data[i], lx, hx, ly, hy, lz, hz, nnx2, ny, nz, nzz, seg);
    seg += nnx2;
  }

  cuda_error_check();
}

/*
 * Poisson equation.
 *
 */

__global__ void rgrid_cuda_poisson_gpu(CUCOMPLEX *grid, CUREAL norm, CUREAL step2, CUREAL ilx, CUREAL ily, CUREAL ilz, INT nx, INT ny, INT nzz, INT seg) {
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, jj = j + seg;
  CUREAL kx, ky, kz;

  if(i >= nx || j >= ny || k >= nzz) return;

  idx = (i * ny + j) * nzz + k;
  kx = COS(ilx * (CUREAL) i);
  ky = COS(ily * (CUREAL) jj);
  kz = COS(ilz * (CUREAL) k);
  if(i || jj || k)
    grid[idx] = grid[idx] * norm * step2 / (2.0 * (kx + ky + kz - 3.0));
  else
    grid[idx] = CUMAKE(0.0,0.0);
}

/*
 * Solve Poisson.
 *
 * grid    = Grid specifying the RHS (cudaXtDesc *; input/output).
 * norm    = FFT normalization constant (CUREAL; input).
 * step2   = Spatial step ^ 2 (CUREAL; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 * In Fourier space.
 *
 */

extern "C" void rgrid_cuda_poissonW(cudaXtDesc *grid, CUREAL norm, CUREAL step2, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_RECIPROCAL(grid);
  INT seg = 0;
  CUREAL ilx = 2.0 * M_PI / ((CUREAL) nx), ily = 2.0 * M_PI / ((CUREAL) ny), ilz = M_PI / ((CUREAL) nzz);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_poisson_gpu<<<blocks1,threads>>>((CUCOMPLEX *) grid->data[i], norm, step2, ilx, ily, ilz, nx, nny1, nzz, seg);
    seg += nny1;
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(grid->GPUs[i]);
    rgrid_cuda_poisson_gpu<<<blocks2,threads>>>((CUCOMPLEX *) grid->data[i], norm, step2, ilx, ily, ilz, nx, nny2, nzz, seg);
    seg += nny2;
  }

  cuda_error_check();
}

/*
 * FFT gradient (x).
 *
 */

__global__ void rgrid_cuda_fft_gradient_x_gpu(CUCOMPLEX *gradient, REAL kx0, REAL step, REAL norm, REAL lx, INT nx, INT ny, INT nz, INT nx2) {
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  REAL kx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  if(i < nx2) 
    kx = ((REAL) i) * lx - kx0;
  else
    kx = -((REAL) (nx - i)) * lx - kx0;
  gradient[idx] = gradient[idx] * CUMAKE(0.0, kx * norm);
}

/*
 * Gradient of grid in Fourier space (X).
 *
 * gradient_x = Source & destination for operation (cudaXtDesc *; input/output).
 * kx0        = Baseline momentum (grid->kx0; REAL; input).
 * step       = Step size (REAL; input).
 * norm       = FFT norm (REAL; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_fft_gradient_xW(cudaXtDesc *gradient_x, REAL kx0, REAL step, REAL norm, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_RECIPROCAL(gradient_x);
  REAL lx = 2.0 * M_PI / (((REAL) nx) * step);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(gradient_x->GPUs[i]);
    rgrid_cuda_fft_gradient_x_gpu<<<blocks1,threads>>>((CUCOMPLEX *) gradient_x->data[i], kx0, step, norm, lx, nx, nny1, nzz, nx / 2);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(gradient_x->GPUs[i]);
    rgrid_cuda_fft_gradient_x_gpu<<<blocks2,threads>>>((CUCOMPLEX *) gradient_x->data[i], kx0, step, norm, lx, nx, nny2, nzz, nx / 2);
  }

  cuda_error_check();
}

/*
 * FFT gradient (y).
 *
 */

__global__ void rgrid_cuda_fft_gradient_y_gpu(CUCOMPLEX *gradient, REAL ky0, REAL step, REAL norm, REAL ly, INT nx, INT ny, INT nz, INT ny2, INT seg) {
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, jj = j + seg, idx;
  REAL ky;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  if(jj < ny2) 
    ky = ((REAL) jj) * ly - ky0;
  else
    ky = -((REAL) (ny - jj)) * ly - ky0;
  gradient[idx] = gradient[idx] * CUMAKE(0.0, ky * norm);
}

/*
 * Gradient of grid in Fourier space (Y).
 *
 * gradient_y = Source & destination for operation (cudaXtDesc *; input/output).
 * ky0        = Baseline momentum (grid->ky0; REAL; input).
 * step       = Step size (REAL; input).
 * norm       = FFT norm (REAL; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_fft_gradient_yW(cudaXtDesc *gradient_y, REAL ky0, REAL step, REAL norm, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_RECIPROCAL(gradient_y);
  INT seg = 0;
  REAL ly = 2.0 * M_PI / (((REAL) ny) * step);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(gradient_y->GPUs[i]);
    rgrid_cuda_fft_gradient_y_gpu<<<blocks1,threads>>>((CUCOMPLEX *) gradient_y->data[i], ky0, step, norm, ly, nx, nny1, nzz, ny / 2, seg);
    seg += nny1;
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(gradient_y->GPUs[i]);
    rgrid_cuda_fft_gradient_y_gpu<<<blocks2,threads>>>((CUCOMPLEX *) gradient_y->data[i], ky0, step, norm, ly, nx, nny2, nzz, ny / 2, seg);
    seg += nny2;
  }

  cuda_error_check();
}

/*
 * FFT gradient (z).
 *
 */

__global__ void rgrid_cuda_fft_gradient_z_gpu(CUCOMPLEX *gradient, REAL kz0, REAL step, REAL norm, REAL lz, INT nx, INT ny, INT nz, INT nz2) {
  
  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  REAL kz;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  if(k < nz2) 
    kz = ((REAL) k) * lz - kz0;
  else
    kz = -((REAL) (nz - k)) * lz - kz0;
  gradient[idx] = gradient[idx] * CUMAKE(0.0, kz * norm);
}

/*
 * Gradient of grid in Fourier space (Z).
 *
 * gradient_z = Source & destination for operation (cudaXtDesc *; input/output).
 * kz0        = Baseline momentum (grid->ky0; REAL; input).
 * step       = Step size (REAL; input).
 * norm       = FFT norm (REAL; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 */

extern "C" void rgrid_cuda_fft_gradient_zW(cudaXtDesc *gradient_z, REAL kz0, REAL step, REAL norm, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_RECIPROCAL(gradient_z);
  REAL lz = M_PI / (((REAL) nzz - 1) * step);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(gradient_z->GPUs[i]);
    rgrid_cuda_fft_gradient_z_gpu<<<blocks1,threads>>>((CUCOMPLEX *) gradient_z->data[i], kz0, step, norm, lz, nx, nny1, nzz, nzz / 2);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(gradient_z->GPUs[i]);
    rgrid_cuda_fft_gradient_z_gpu<<<blocks2,threads>>>((CUCOMPLEX *) gradient_z->data[i], kz0, step, norm, lz, nx, nny2, nzz, nzz / 2);
  }

  cuda_error_check();
}

/*
 * FFT laplace.
 *
 * B = B'' in Fourier space.
 *
 */

__global__ void rgrid_cuda_fft_laplace_gpu(CUCOMPLEX *b, CUREAL norm, CUREAL kx0, CUREAL ky0, CUREAL kz0, CUREAL lx, CUREAL ly, CUREAL lz, CUREAL step, INT nx, INT ny, INT nz, INT nx2, INT ny2, INT nz2, INT seg, INT nyy) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, jj = j + seg;
  CUREAL kx, ky, kz;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  
  if(i < nx2) 
    kx = ((REAL) i) * lx - kx0;
  else
    kx = -((REAL) (nx - i)) * lx - kx0;
  if(jj < ny2) 
    ky = ((REAL) jj) * ly - ky0;
  else
    ky = -((REAL) (nyy - jj)) * ly - ky0;
  if(k < nz2) 
    kz = ((REAL) k) * lz - kz0;
  else
    kz = -((REAL) (nz - k)) * lz - kz0;        

  b[idx] = b[idx] * (-(kx * kx + ky * ky + kz * kz) * norm);
}

/*
 * FFT laplace
 *
 * laplace  = Source/destination grid for operation (cudaXtDesc *; input/output).
 * norm     = FFT norm (grid->fft_norm) (REAL; input).
 * kx0      = Momentum shift of origin along X (REAL; input).
 * ky0      = Momentum shift of origin along Y (REAL; input).
 * kz0      = Momentum shift of origin along Z (REAL; input).
 * step     = Spatial step length (REAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 * Only periodic boundaries!
 *
 * In Fourier space.
 *
 */

extern "C" void rgrid_cuda_fft_laplaceW(cudaXtDesc *laplace, CUREAL norm, CUREAL kx0, CUREAL ky0, CUREAL kz0, CUREAL step, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_RECIPROCAL(laplace);
  INT seg = 0;

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(laplace->GPUs[i]);
    rgrid_cuda_fft_laplace_gpu<<<blocks1,threads>>>((CUCOMPLEX *) laplace->data[i], norm, kx0, ky0, kz0, 
        2.0 * M_PI / (((REAL) nx) * step), 2.0 * M_PI / (((REAL) ny) * step), M_PI / (((REAL) nzz - 1) * step), step, nx, nny1, nzz, nx / 2, ny / 2, nzz / 2, seg, ny);
    seg += nny1;
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(laplace->GPUs[i]);
    rgrid_cuda_fft_laplace_gpu<<<blocks2,threads>>>((CUCOMPLEX *) laplace->data[i], norm, kx0, ky0, kz0,
        2.0 * M_PI / (((REAL) nx) * step), 2.0 * M_PI / (((REAL) ny) * step), M_PI / (((REAL) nzz - 1) * step), step, nx, nny2, nzz, nx / 2, ny / 2, nzz / 2, seg, ny);
    seg += nny2;
  }

  cuda_error_check();
}

/*
 * FFT laplace expectation value.
 *
 * B = <B''> in Fourier space.
 *
 * Only periodic version implemented.
 *
 * Normalization done in rgrid-cuda.c
 *
 */

__global__ void rgrid_cuda_fft_laplace_expectation_value_gpu(CUCOMPLEX *b, CUREAL *blocks, CUREAL kx0, CUREAL ky0, CUREAL kz0, CUREAL lx, CUREAL ly, CUREAL lz, CUREAL step, INT nx, INT ny, INT nz, INT nx2, INT ny2, INT nz2, INT seg, INT nyy) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, jj = j + seg;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2, t;
  CUREAL kx, ky, kz;
  extern __shared__ CUREAL els2[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els2[t] = 0.0;
  }
  __syncthreads();

  idx = (i * ny + j) * nz + k;

  if(i < nx2) 
    kx = ((REAL) i) * lx - kx0;
  else
    kx = -((REAL) (nx - i)) * lx - kx0;
  if(jj < ny2) 
    ky = ((REAL) jj) * ly - ky0;
  else
    ky = -((REAL) (nyy - jj)) * ly - ky0;
  if(k < nz2) 
    kz = ((REAL) k) * lz - kz0;
  else
    kz = -((REAL) (nz - k)) * lz - kz0;        

  idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
  els2[idx2] -= (kx * kx + ky * ky + kz * kz) * (CUCREAL(b[idx]) * CUCREAL(b[idx]) + CUCIMAG(b[idx]) * CUCIMAG(b[idx]));
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] += els2[t];  // reduce threads
    }
  }
}

/*
 * FFT laplace expectation value
 *
 * laplace  = Source/destination grid for operation (cudaXtDesc *; input/output).
 * kx0      = Momentum shift of origin along X (REAL; input).
 * ky0      = Momentum shift of origin along Y (REAL; input).
 * kz0      = Momentum shift of origin along Z (REAL; input).
 * step     = Spatial step length (REAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 * sum      = Expectation value (REAL; output).
 *
 * Only periodic boundaries!  In Fourier space.
 *
 * Normalization done in rgrid-cuda.c
 *
 */

extern "C" void rgrid_cuda_fft_laplace_expectation_valueW(cudaXtDesc *laplace, CUREAL kx0, CUREAL ky0, CUREAL kz0, CUREAL step, INT nx, INT ny, INT nz, CUREAL *value) {

  SETUP_VARIABLES_RECIPROCAL(laplace);
  CUREAL tmp;
  INT seg = 0, s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b31 = blocks1.x * blocks1.y * blocks1.z, b32 = blocks2.x * blocks2.y * blocks2.z;
  extern int cuda_get_element(void *, int, size_t, size_t, void *);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(laplace->GPUs[i]);
    rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b31);
    cuda_error_check();
    // Blocks, Threads, dynamic memory size
    rgrid_cuda_fft_laplace_expectation_value_gpu<<<blocks1,threads,s*sizeof(CUREAL)>>>((CUCOMPLEX *) laplace->data[i], (CUREAL *) grid_gpu_mem_addr->data[i], 
                               kx0, ky0, kz0, 2.0 * M_PI / (((REAL) nx) * step), 2.0 * M_PI / (((REAL) ny) * step), M_PI / (((REAL) nzz - 1) * step),
                               step, nx, nny1, nzz, nx / 2, ny / 2, nzz / 2, seg, ny);
    cuda_error_check();
    rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b31);
    cuda_error_check();
    seg += nny1;
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(laplace->GPUs[i]);
    rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b32);
    cuda_error_check();
    // Blocks, Threads, dynamic memory size
    rgrid_cuda_fft_laplace_expectation_value_gpu<<<blocks2,threads,s*sizeof(CUREAL)>>>((CUCOMPLEX *) laplace->data[i], (CUREAL *) grid_gpu_mem_addr->data[i], 
                               kx0, ky0, kz0, 2.0 * M_PI / (((REAL) nx) * step), 2.0 * M_PI / (((REAL) ny) * step), M_PI / (((REAL) nzz - 1) * step),
                               step, nx, nny2, nzz, nx / 2, ny / 2, nzz / 2, seg, ny);
    cuda_error_check();
    rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b32);
    cuda_error_check();
    seg += nny2;
  }

  // Reduce over GPUs
  *value = 0.0;
  for(i = 0; i < ngpu2; i++) {
    cuda_get_element(grid_gpu_mem, laplace->GPUs[i], 0, sizeof(CUREAL), &tmp);
    *value += tmp;
  }
  cuda_error_check();
}
