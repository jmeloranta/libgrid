#include "hip/hip_runtime.h"
/*
 * CUDA device code for wf.
 *
 */

#include <cuda/hip/hip_runtime_api.h>
#include <cuda/hip/hip_runtime.h>
#include <cuda/>
#include <hipfft/hipfft.h>
#include "hip/hip_runtime.h"
#include "cuda-math.h"
#include "defs.h"

extern void *grid_gpu_mem_addr;
extern "C" void cuda_error_check();

/********************************************************************************************************************/

/*
 * Density
 *
 */

__global__ void grid_cuda_wf_density_gpu(CUCOMPLEX *b, CUREAL *dens, INT nx, INT ny, INT nz, INT nz2) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, idx2;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  idx2 = (i * ny + j) * nz2 + k;

  dens[idx2] = CUCREAL(b[idx]) * CUCREAL(b[idx]) + CUCIMAG(b[idx]) * CUCIMAG(b[idx]);
}

/*
 * Density
 *
 * wf       = Source/destination grid for operation (REAL complex *; input).
 * dens     = Density grid (CUCOMPLEX *; output).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 */

extern "C" void grid_cuda_wf_densityW(CUCOMPLEX *grid, CUREAL *dens, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  grid_cuda_wf_density_gpu<<<blocks,threads>>>(grid, dens, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/
