#include "hip/hip_runtime.h"
/*
 * CUDA device code for wf.
 *
 */

#include <cuda/hip/hip_runtime_api.h>
#include <cuda/hip/hip_runtime.h>
#include <cuda/>
#include <hipfft/hipfft.h>
#include "hip/hip_runtime.h"
#include "cuda-math.h"
#include "defs.h"
#include "grid_wf-cuda.h"

extern void *grid_gpu_mem_addr;
extern "C" void cuda_error_check();

/********************************************************************************************************************/

/*
 * Potential energy propagation in real space (possibly with absorbing boundaries).
 *
 */

/* amp = 0 */
__global__ void grid_cuda_wf_propagate_potential_gpu1(CUCOMPLEX *b, CUCOMPLEX *pot, CUCOMPLEX c, INT nx, INT ny, INT nz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  b[idx] = b[idx] * CUCEXP(c * pot[idx]);
}

/* amp != 0 */
__global__ void grid_cuda_wf_propagate_potential_gpu2(CUCOMPLEX *b, CUCOMPLEX *pot, CUCOMPLEX c, CUREAL amp, INT lx, INT hx, INT ly, INT hy, INT lz, INT hz, INT nx, INT ny, INT nz) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  c = c * grid_cuda_wf_absorb(i, j, k, amp, lx, hx, ly, hy, lz, hz);
  b[idx] = b[idx] * CUCEXP(c * pot[idx]);
}

/*
 * Propagate potential energy in real space with absorbing boundaries.
 *
 * wf       = Source/destination grid for operation (REAL complex *; input/output).
 * pot      = Potential grid (CUCOMPLEX *; input).
 * time_step= Time step length (CUCOMPLEX; input).
 * amp      = Max amplitude for imag. part (CUREAL; input).
 * lx       = Lower bound for absorbing bc (INT; input).
 * hx       = Upper bound for absorbing bc (INT; input).
 * ly       = Lower bound for absorbing bc (INT; input).
 * hy       = Upper bound for absorbing bc (INT; input).
 * lz       = Lower bound for absorbing bc (INT; input).
 * hz       = Upper bound for absorbing bc (INT; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 * Only periodic boundaries!
 *
 */

extern "C" void grid_cuda_wf_propagate_potentialW(CUCOMPLEX *grid, CUCOMPLEX *pot, CUCOMPLEX time_step, CUREAL amp, INT lx, INT hx, INT ly, INT hy, INT lz, INT hz, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  CUCOMPLEX c;

  c.x =  (1.0 / HBAR) * time_step.y;
  c.y = -(1.0 / HBAR) * time_step.x;
  if(amp != 0.0) 
    grid_cuda_wf_propagate_potential_gpu2<<<blocks,threads>>>(grid, pot, c, amp, lx, hx, ly, hy, lz, hz, nx, ny, nz);
  else
    grid_cuda_wf_propagate_potential_gpu1<<<blocks,threads>>>(grid, pot, c, nx, ny, nz);
  cuda_error_check();
}

/********************************************************************************************************************/

/*
 * Density
 *
 */

__global__ void grid_cuda_wf_density_gpu(CUCOMPLEX *b, CUREAL *dens, INT nx, INT ny, INT nz, INT nz2) {  /* Exectutes at GPU */

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, idx2;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  idx2 = (i * ny + j) * nz2 + k;

  dens[idx2] = CUCREAL(b[idx]) * CUCREAL(b[idx]) + CUCIMAG(b[idx]) * CUCIMAG(b[idx]);
}

/*
 * Density
 *
 * wf       = Source/destination grid for operation (REAL complex *; input).
 * dens     = Density grid (CUCOMPLEX *; output).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 *
 */

extern "C" void grid_cuda_wf_densityW(CUCOMPLEX *grid, CUREAL *dens, INT nx, INT ny, INT nz) {

  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);

  grid_cuda_wf_density_gpu<<<blocks,threads>>>(grid, dens, nx, ny, nz, 2 * (nz / 2 + 1));
  cuda_error_check();
}

/********************************************************************************************************************/
