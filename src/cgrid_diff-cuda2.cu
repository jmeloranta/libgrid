#include "hip/hip_runtime.h"
/*
 * CUDA device code (REAL complex = CUCOMPLEX; cgrid) involving differentiation.
 *
 * blockDim = # of threads
 * gridDim = # of blocks
 *
 * Since the ordering of the data on multiple GPUs changes after FFT,
 * we must keep track of this. The variable space in cgrid structure (structs.h)
 * is zero if the grid is currently in real space and one if it is in reciprocal space (i.e., after FFT).
 * The real space data is distributed on GPUs according to the first index (x as in data[x][y][z])
 * where as the resiprocal space data is distributed along the second index (y as in data[x][y][z]).
 *
 * SETUP_VARIABLES and SETUP_VARIABLES_SEG are macros that follow the correct distribution over GPUs
 * based on the value of space variable. The latter also keeps track of the actual index so that
 * routines that need to access proper grid indices can do so.
 *
 */

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include "cuda-math.h"
#include "cgrid_bc-cuda.h"
#include "cuda-vars.h"
#include "hip/hip_runtime.h"

extern void *grid_gpu_mem;            /* host memory holder for the block used for reductions */
extern cudaXtDesc *grid_gpu_mem_addr; /* the corresponding GPU memory addresses */
extern "C" void cuda_error_check();

/*
 * dst = FD_X(src).
 *
 */

__global__ void cgrid_cuda_fd_gradient_x_gpu(CUCOMPLEX *src, CUCOMPLEX *dst, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = inv_delta * (cgrid_cuda_bc_x_plus(src, bc, i, j, k, nx, ny, nz) - cgrid_cuda_bc_x_minus(src, bc, i, j, k, nx, ny, nz));
}

/*
 * dst = FD_X(src)
 *
 * src       = Source for operation (gpu_mem_block *; input).
 * dst       = Destination for operation (gpu_mem_block *; input).
 * inv_delta = 1 / (2 * step) (REAL; input).
 * bc        = Boundary condition: 0 = Dirichlet, 1 = Neumann, 2 = Periodic (char; input).
 * nx        = # of points along x (INT; input).
 * ny        = # of points along y (INT; input).
 * nz        = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fd_gradient_xW(gpu_mem_block *src, gpu_mem_block *dst, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz) {

  dst->gpu_info->subFormat = src->gpu_info->subFormat;
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  cudaXtDesc *SRC = src->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;
 
  if(DST->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(DST->GPUs[0]);
  cgrid_cuda_fd_gradient_x_gpu<<<blocks,threads>>>((CUCOMPLEX *) SRC->data[0], (CUCOMPLEX *) DST->data[0], inv_delta, bc, nx, ny, nz);

  cuda_error_check();
}

/*
 * dst = FD_Y(src).
 *
 */

__global__ void cgrid_cuda_fd_gradient_y_gpu(CUCOMPLEX *src, CUCOMPLEX *dst, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = inv_delta * (cgrid_cuda_bc_y_plus(src, bc, i, j, k, nx, ny, nz) - cgrid_cuda_bc_y_minus(src, bc, i, j, k, nx, ny, nz));
}

/*
 * B = FD_Y(A)
 *
 * src       = Source for operation (gpu_mem_block *; input).
 * dst       = Destination for operation (gpu_mem_block *; input).
 * inv_delta = 1 / (2 * step) (CUREAL; input).
 * bc        = Boundary condition: 0 = Dirichlet, 1 = Neumann, 2 = Periodic (char; input).
 * nx        = # of points along x (INT; input).
 * ny        = # of points along y (INT; input).
 * nz        = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fd_gradient_yW(gpu_mem_block *src, gpu_mem_block *dst, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz) {

  dst->gpu_info->subFormat = src->gpu_info->subFormat;
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  cudaXtDesc *SRC = src->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  if(DST->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(DST->GPUs[0]);
  cgrid_cuda_fd_gradient_y_gpu<<<blocks,threads>>>((CUCOMPLEX *) SRC->data[0], (CUCOMPLEX *) DST->data[0], inv_delta, bc, nx, ny, nz);

  cuda_error_check();
}

/*
 * dst = FD_Z(src).
 *
 */

__global__ void cgrid_cuda_fd_gradient_z_gpu(CUCOMPLEX *src, CUCOMPLEX *dst, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = inv_delta * (cgrid_cuda_bc_z_plus(src, bc, i, j, k, nx, ny, nz) - cgrid_cuda_bc_z_minus(src, bc, i, j, k, nx, ny, nz));
}

/*
 * B = FD_Z(A)
 *
 * src       = Source for operation (gpu_mem_block *; input).
 * dst       = Destination for operation (gpu_mem_block *; input).
 * inv_delta = 1 / (2 * step) (CUREAL; input).
 * bc        = Boundary condition: 0 = Dirichlet, 1 = Neumann, 2 = Periodic (char; input).
 * nx        = # of points along x (INT; input).
 * ny        = # of points along y (INT; input).
 * nz        = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fd_gradient_zW(gpu_mem_block *src, gpu_mem_block *dst, CUREAL inv_delta, char bc, INT nx, INT ny, INT nz) {

  dst->gpu_info->subFormat = src->gpu_info->subFormat;
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  cudaXtDesc *SRC = src->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  if(DST->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(DST->GPUs[0]);
  cgrid_cuda_fd_gradient_z_gpu<<<blocks,threads>>>((CUCOMPLEX *) SRC->data[0], (CUCOMPLEX *) DST->data[0], inv_delta, bc, nx, ny, nz);

  cuda_error_check();
}

/*
 * dst = LAPLACE(src).
 *
 */

__global__ void cgrid_cuda_fd_laplace_gpu(CUCOMPLEX *src, CUCOMPLEX *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = inv_delta2 * (cgrid_cuda_bc_x_plus(src, bc, i, j, k, nx, ny, nz) + cgrid_cuda_bc_x_minus(src, bc, i, j, k, nx, ny, nz)
                         + cgrid_cuda_bc_y_plus(src, bc, i, j, k, nx, ny, nz) + cgrid_cuda_bc_y_minus(src, bc, i, j, k, nx, ny, nz)
                         + cgrid_cuda_bc_z_plus(src, bc, i, j, k, nx, ny, nz) + cgrid_cuda_bc_z_minus(src, bc, i, j, k, nx, ny, nz)
                         - 6.0 * src[idx]);
}

/*
 * B = LAPLACE(A)
 *
 * src        = Source for operation (gpu_mem_block *; input).
 * dst        = Destination for operation (gpu_mem_block *; output).
 * inv_delta2 = 1 / (2 * step) (CUREAL; input).
 * bc         = Boundary condition (char; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 * Returns laplacian in dst.
 *
 */

extern "C" void cgrid_cuda_fd_laplaceW(gpu_mem_block *src, gpu_mem_block *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  dst->gpu_info->subFormat = src->gpu_info->subFormat;
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  cudaXtDesc *SRC = src->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  if(DST->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(DST->GPUs[0]);
  cgrid_cuda_fd_laplace_gpu<<<blocks,threads>>>((CUCOMPLEX *) SRC->data[0], (CUCOMPLEX *) DST->data[0], inv_delta2, bc, nx, ny, nz);

  cuda_error_check();
}

/*
 * dst = LAPLACE_X(src).
 *
 */

__global__ void cgrid_cuda_fd_laplace_x_gpu(CUCOMPLEX *src, CUCOMPLEX *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = inv_delta2 * (cgrid_cuda_bc_x_plus(src, bc, i, j, k, nx, ny, nz) + cgrid_cuda_bc_x_minus(src, bc, i, j, k, nx, ny, nz) - 2.0 * src[idx]);
}

/*
 * dst = LAPLACE_X(src)
 *
 * src        = Source for operation (gpu_mem_block *; input).
 * dst        = Destination for operation (gpu_mem_block *; output).
 * inv_delta2 = 1 / (2 * step) (CUREAL; input).
 * bc         = Boundary condition (char; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fd_laplace_xW(gpu_mem_block *src, gpu_mem_block *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  dst->gpu_info->subFormat = src->gpu_info->subFormat;
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  cudaXtDesc *SRC = src->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  if(DST->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(DST->GPUs[0]);
  cgrid_cuda_fd_laplace_x_gpu<<<blocks,threads>>>((CUCOMPLEX *) SRC->data[0], (CUCOMPLEX *) DST->data[0], inv_delta2, bc, nx, ny, nz);

  cuda_error_check();
}

/*
 * dst = LAPLACE_Y(src).
 *
 */

__global__ void cgrid_cuda_fd_laplace_y_gpu(CUCOMPLEX *src, CUCOMPLEX *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = inv_delta2 * (cgrid_cuda_bc_y_plus(src, bc, i, j, k, nx, ny, nz) + cgrid_cuda_bc_y_minus(src, bc, i, j, k, nx, ny, nz) - 2.0 * src[idx]);
}

/*
 * B = LAPLACE_Y(A)
 *
 * src        = Source for operation (gpu_mem_block *; input).
 * dst        = Destination for operation (gpu_mem_block *; output).
 * inv_delta2 = 1 / (2 * step) (CUREAL; input).
 * bc         = Boundary condition (char; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fd_laplace_yW(gpu_mem_block *src, gpu_mem_block *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  dst->gpu_info->subFormat = src->gpu_info->subFormat;
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  cudaXtDesc *SRC = src->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  if(DST->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(DST->GPUs[0]);
  cgrid_cuda_fd_laplace_y_gpu<<<blocks,threads>>>((CUCOMPLEX *) SRC->data[0], (CUCOMPLEX *) DST->data[0], inv_delta2, bc, nx, ny, nz);

  cuda_error_check();
}

/*
 * dst = LAPLACE_Z(src).
 *
 */

__global__ void cgrid_cuda_fd_laplace_z_gpu(CUCOMPLEX *src, CUCOMPLEX *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  dst[idx] = inv_delta2 * (cgrid_cuda_bc_z_plus(src, bc, i, j, k, nx, ny, nz) + cgrid_cuda_bc_z_minus(src, bc, i, j, k, nx, ny, nz) - 2.0 * src[idx]);
}

/*
 * dst = LAPLACE_Z(src)
 *
 * src        = Source for operation (gpu_mem_block *; input).
 * dst        = Destination for operation (gpu_mem_block *; output).
 * inv_delta2 = 1 / (2 * step) (CUREAL; input).
 * bc         = Boundary condition (char; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fd_laplace_zW(gpu_mem_block *src, gpu_mem_block *dst, CUREAL inv_delta2, char bc, INT nx, INT ny, INT nz) {

  dst->gpu_info->subFormat = src->gpu_info->subFormat;
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  cudaXtDesc *SRC = src->gpu_info->descriptor, *DST = dst->gpu_info->descriptor;

  if(DST->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(DST->GPUs[0]);
  cgrid_cuda_fd_laplace_z_gpu<<<blocks,threads>>>((CUCOMPLEX *) SRC->data[0], (CUCOMPLEX *) DST->data[0], inv_delta2, bc, nx, ny, nz);

  cuda_error_check();
}

/*
 * FFT gradient_x
 *
 */

__global__ void cgrid_cuda_fft_gradient_x_gpu(CUCOMPLEX *b, CUREAL norm, CUREAL kx0, CUREAL step, INT nx, INT ny, INT nz, INT nx2) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  CUREAL kx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;

  if (i <= nx2)
    kx = 2.0 * M_PI * ((CUREAL) i) / (((CUREAL) nx) * step) - kx0;
  else 
    kx = 2.0 * M_PI * ((CUREAL) (i - nx)) / (((CUREAL) nx) * step) - kx0;

  b[idx] = b[idx] * CUMAKE(0.0, kx * norm);     // multiply by I * kx * norm
}

/*
 * FFT gradient_x
 *
 * dst        = Source/destination grid for operation (gpu_mem_block *; input/output).
 * norm       = FFT norm (grid->fft_norm) (CUREAL; input).
 * kx0        = Momentum shift of origin along X (CUREAL; input).
 * step       = Spatial step length (CUREAL; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fft_gradient_xW(gpu_mem_block *dst, CUREAL norm, CUREAL kx0, CUREAL step, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor;
  INT nx2 = nx / 2;

  if(dst->gpu_info->subFormat != HIPFFT_XT_FORMAT_INPLACE_SHUFFLED) {
    fprintf(stderr, "libgrid(cuda): fft_gradient_x wrong subFormat.\n");
    abort();
  }

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_gradient_x_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], norm, kx0, step, nnx1, nny1, nz, nx2);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_gradient_x_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], norm, kx0, step, nnx2, nny2, nz, nx2);
  }

  cuda_error_check();
}

/*
 * FFT gradient_y
 *
 * B = B' in Fourier space.
 *
 */

__global__ void cgrid_cuda_fft_gradient_y_gpu(CUCOMPLEX *b, CUREAL norm, CUREAL ky0, CUREAL step, INT nx, INT ny, INT nz, INT nyy, INT ny2, INT seg) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, jj = j + seg;
  CUREAL ky;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;

  if (jj <= ny2)
    ky = 2.0 * M_PI * ((CUREAL) jj) / (((CUREAL) nyy) * step) - ky0;
  else 
    ky = 2.0 * M_PI * ((CUREAL) (jj - nyy)) / (((CUREAL) nyy) * step) - ky0;

  b[idx] = b[idx] * CUMAKE(0.0, ky * norm);    // multiply by I * ky * norm
}

/*
 * FFT gradient_y
 *
 * dst        = Source/destination grid for operation (gpu_mem_block *; input/output).
 * norm       = FFT norm (grid->fft_norm) (CUREAL; input).
 * ky0        = Momentum shift of origin along Y (CUREAL; input).
 * step       = Spatial step length (CUREAL; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fft_gradient_yW(gpu_mem_block *dst, CUREAL norm, CUREAL ky0, CUREAL step, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_SEG(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor;
  INT ny2 = ny / 2, segx = 0, segy = 0;  // segx unused

  if(dst->gpu_info->subFormat != HIPFFT_XT_FORMAT_INPLACE_SHUFFLED) {
    fprintf(stderr, "libgrid(cuda): fft_gradient_y wrong subFormat.\n");
    abort();
  }

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_gradient_y_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], norm, ky0, step, nnx1, nny1, nz, ny, ny2, segy);
    segx += dsegx1;
    segy += dsegy1;
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_gradient_y_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], norm, ky0, step, nnx2, nny2, nz, ny, ny2, segy);
    segx += dsegx2;
    segy += dsegy2;
  }

  cuda_error_check();
}

/*
 * FFT gradient_z
 *
 * B = B' in Fourier space.
 *
 */

__global__ void cgrid_cuda_fft_gradient_z_gpu(CUCOMPLEX *b, CUREAL norm, CUREAL kz0, CUREAL step, INT nx, INT ny, INT nz, INT nz2) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  CUREAL kz;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;

  if (k <= nz2)
    kz = 2.0 * M_PI * ((CUREAL) k) / (((CUREAL) nz) * step) - kz0;
  else 
    kz = 2.0 * M_PI * ((CUREAL) (k - nz)) / (((CUREAL) nz) * step) - kz0;

  b[idx] = b[idx] * CUMAKE(0.0, kz * norm);   // multiply by I * kz * norm
}

/*
 * FFT gradient_z
 *
 * dst        = Source/destination grid for operation (gpu_mem_block *; input/output).
 * norm       = FFT norm (grid->fft_norm) (CUREAL; input).
 * kz0        = Momentum shift of origin along Z (CUREAL; input).
 * step       = Spatial step length (CUREAL; input).
 * nx         = # of points along x (INT; input).
 * ny         = # of points along y (INT; input).
 * nz         = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fft_gradient_zW(gpu_mem_block *dst, CUREAL norm, CUREAL kz0, CUREAL step, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor;
  INT nz2 = nz / 2;

  if(dst->gpu_info->subFormat != HIPFFT_XT_FORMAT_INPLACE_SHUFFLED) {
    fprintf(stderr, "libgrid(cuda): fft_gradient_z wrong subFormat.\n");
    abort();
  }

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_gradient_z_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], norm, kz0, step, nnx1, nny1, nz, nz2);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_gradient_z_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], norm, kz0, step, nnx2, nny2, nz, nz2);
  }

  cuda_error_check();
}

/*
 * FFT laplace.
 *
 * B = B'' in Fourier space.
 *
 */

__global__ void cgrid_cuda_fft_laplace_gpu(CUCOMPLEX *b, CUREAL norm, CUREAL kx0, CUREAL ky0, CUREAL kz0, CUREAL step, INT nx, INT ny, INT nz, INT nyy, INT nx2, INT ny2, INT nz2, INT seg) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, jj = j + seg, idx;
  CUREAL kx, ky, kz;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  
  if (i <= nx2)
    kx = 2.0 * M_PI * ((CUREAL) i) / (((CUREAL) nx) * step) - kx0;
  else 
    kx = 2.0 * M_PI * ((CUREAL) (i - nx)) / (((CUREAL) nx) * step) - kx0;
      
  if (jj <= ny2)
    ky = 2.0 * M_PI * ((CUREAL) jj) / (((CUREAL) nyy) * step) - ky0;
  else 
    ky = 2.0 * M_PI * ((CUREAL) (jj - nyy)) / (((CUREAL) nyy) * step) - ky0;
      
  if (k <= nz2)
    kz = 2.0 * M_PI * ((CUREAL) k) / (((CUREAL) nz) * step) - kz0;
  else 
    kz = 2.0 * M_PI * ((CUREAL) (k - nz)) / (((CUREAL) nz) * step) - kz0;      

  b[idx] = b[idx] * (-(kx * kx + ky * ky + kz * kz) * norm);
}

/*
 * FFT laplace
 *
 * dst      = Source/destination grid for operation (gpu_mem_block *; input/output).
 * norm     = FFT norm (grid->fft_norm) (CUREAL; input).
 * kx0      = Momentum shift of origin along X (CUREAL; input).
 * ky0      = Momentum shift of origin along Y (CUREAL; input).
 * kz0      = Momentum shift of origin along Z (CUREAL; input).
 * step     = Spatial step length (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fft_laplaceW(gpu_mem_block *dst, CUREAL norm, CUREAL kx0, CUREAL ky0, CUREAL kz0, CUREAL step, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_SEG(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor;
  INT nx2 = nx / 2, ny2 = ny / 2, nz2 = nz / 2, segx = 0, segy = 0; // segx not used

  if(dst->gpu_info->subFormat != HIPFFT_XT_FORMAT_INPLACE_SHUFFLED) {
    fprintf(stderr, "libgrid(cuda): fft_laplace wrong subFormat.\n");
    abort();
  }

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_laplace_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], norm, kx0, ky0, kz0, step, nnx1, nny1, nz, ny, nx2, ny2, nz2, segy);
    segx += dsegx1;
    segy += dsegy1;
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_laplace_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], norm, kx0, ky0, kz0, step, nnx2, nny2, nz, ny, nx2, ny2, nz2, segy);
    segx += dsegx2;
    segy += dsegy2;
  }

  cuda_error_check();
}

/*
 * FFT laplace (X).
 *
 * B = B'' in Fourier space.
 *
 */

__global__ void cgrid_cuda_fft_laplace_x_gpu(CUCOMPLEX *b, CUREAL norm, CUREAL kx0, CUREAL step, INT nx, INT ny, INT nz, INT nx2) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  CUREAL kx;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  
  if (i <= nx2)
    kx = 2.0 * M_PI * ((CUREAL) i) / (((CUREAL) nx) * step) - kx0;
  else 
    kx = 2.0 * M_PI * ((CUREAL) (i - nx)) / (((CUREAL) nx) * step) - kx0;
      
  b[idx] = b[idx] * (-kx * kx * norm);
}

/*
 * FFT laplace (X)
 *
 * dst      = Source/destination grid for operation (gpu_mem_block *; input/output).
 * norm     = FFT norm (grid->fft_norm) (CUREAL; input).
 * kx0      = Momentum shift of origin along X (CUREAL; input).
 * step     = Spatial step length (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fft_laplace_xW(gpu_mem_block *dst, CUREAL norm, CUREAL kx0, CUREAL step, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_SEG(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor;
  INT nx2 = nx / 2, segx = 0, segy = 0; // segx not used

  if(dst->gpu_info->subFormat != HIPFFT_XT_FORMAT_INPLACE_SHUFFLED) {
    fprintf(stderr, "libgrid(cuda): fft_laplace wrong subFormat.\n");
    abort();
  }

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_laplace_x_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], norm, kx0, step, nnx1, nny1, nz, nx2);
    segx += dsegx1;
    segy += dsegy1;
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_laplace_x_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], norm, kx0, step, nnx2, nny2, nz, nx2);
    segx += dsegx2;
    segy += dsegy2;
  }

  cuda_error_check();
}

/*
 * FFT laplace (Y).
 *
 * B = B'' in Fourier space.
 *
 */

__global__ void cgrid_cuda_fft_laplace_y_gpu(CUCOMPLEX *b, CUREAL norm, CUREAL ky0, CUREAL step, INT nx, INT ny, INT nz, INT nyy, INT ny2, INT seg) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, jj = j + seg, idx;
  CUREAL ky;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  
  if (jj <= ny2)
    ky = 2.0 * M_PI * ((CUREAL) jj) / (((CUREAL) nyy) * step) - ky0;
  else 
    ky = 2.0 * M_PI * ((CUREAL) (jj - nyy)) / (((CUREAL) nyy) * step) - ky0;
      
  b[idx] = b[idx] * (-ky * ky * norm);
}

/*
 * FFT laplace (Y)
 *
 * dst      = Source/destination grid for operation (gpu_mem_block *; input/output).
 * norm     = FFT norm (grid->fft_norm) (CUREAL; input).
 * ky0      = Momentum shift of origin along Y (CUREAL; input).
 * step     = Spatial step length (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fft_laplace_yW(gpu_mem_block *dst, CUREAL norm, CUREAL ky0, CUREAL step, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_SEG(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor;
  INT ny2 = ny / 2, segx = 0, segy = 0; // segx not used

  if(dst->gpu_info->subFormat != HIPFFT_XT_FORMAT_INPLACE_SHUFFLED) {
    fprintf(stderr, "libgrid(cuda): fft_laplace wrong subFormat.\n");
    abort();
  }

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_laplace_y_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], norm, ky0, step, nnx1, nny1, nz, ny, ny2, segy);
    segx += dsegx1;
    segy += dsegy1;
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_laplace_y_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], norm, ky0, step, nnx2, nny2, nz, ny, ny2, segy);
    segx += dsegx2;
    segy += dsegy2;
  }

  cuda_error_check();
}

/*
 * FFT laplace (Z).
 *
 * B = B'' in Fourier space.
 *
 */

__global__ void cgrid_cuda_fft_laplace_z_gpu(CUCOMPLEX *b, CUREAL norm, CUREAL kz0, CUREAL step, INT nx, INT ny, INT nz, INT nz2) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  CUREAL kz;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  
  if (k <= nz2)
    kz = 2.0 * M_PI * ((CUREAL) k) / (((CUREAL) nz) * step) - kz0;
  else 
    kz = 2.0 * M_PI * ((CUREAL) (k - nz)) / (((CUREAL) nz) * step) - kz0;      
      
  b[idx] = b[idx] * (-kz * kz * norm);
}

/*
 * FFT laplace (Z)
 *
 * dst      = Source/destination grid for operation (gpu_mem_block *; input/output).
 * norm     = FFT norm (grid->fft_norm) (CUREAL; input).
 * kz0      = Momentum shift of origin along Z (CUREAL; input).
 * step     = Spatial step length (CUREAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void cgrid_cuda_fft_laplace_zW(gpu_mem_block *dst, CUREAL norm, CUREAL kz0, CUREAL step, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES_SEG(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor;
  INT nz2 = nz / 2, segx = 0, segy = 0; // segx not used

  if(dst->gpu_info->subFormat != HIPFFT_XT_FORMAT_INPLACE_SHUFFLED) {
    fprintf(stderr, "libgrid(cuda): fft_laplace wrong subFormat.\n");
    abort();
  }

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_laplace_z_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], norm, kz0, step, nnx1, nny1, nz, nz2);
    segx += dsegx1;
    segy += dsegy1;
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_fft_laplace_z_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], norm, kz0, step, nnx2, nny2, nz, nz2);
    segx += dsegx2;
    segy += dsegy2;
  }

  cuda_error_check();
}

/*
 * FFT laplace expectation value.
 *
 * B = <B''> in Fourier space.
 *
 * Only periodic version implemented.
 *
 * Normalization done in cgrid-cuda.c
 *
 */

__global__ void cgrid_cuda_fft_laplace_expectation_value_gpu(CUCOMPLEX *b, CUCOMPLEX *blocks, CUREAL kx0, CUREAL ky0, CUREAL kz0, CUREAL step, INT nx, INT ny, INT nz, INT nyy, INT nx2, INT ny2, INT nz2, INT seg) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, jj = j + seg;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2, t;
  CUREAL kx, ky, kz;
  extern __shared__ CUREAL els2[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els2[t] = 0.0;
  }
  __syncthreads();

  idx = (i * ny + j) * nz + k;

  if (i <= nx2)
    kx = 2.0 * M_PI * ((REAL) i) / (((REAL) nx) * step) - kx0;
  else 
    kx = 2.0 * M_PI * ((REAL) (i - nx)) / (((REAL) nx) * step) - kx0;
      
  if (jj <= ny2)
    ky = 2.0 * M_PI * ((REAL) jj) / (((REAL) nyy) * step) - ky0;
  else 
    ky = 2.0 * M_PI * ((REAL) (jj - nyy)) / (((REAL) nyy) * step) - ky0;
      
  if (k <= nz2)
    kz = 2.0 * M_PI * ((REAL) k) / (((REAL) nz) * step) - kz0;
  else 
    kz = 2.0 * M_PI * ((REAL) (k - nz)) / (((REAL) nz) * step) - kz0;

  idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
  els2[idx2] -= (kx * kx + ky * ky + kz * kz) * (CUCREAL(b[idx]) * CUCREAL(b[idx]) + CUCIMAG(b[idx]) * CUCIMAG(b[idx]));
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2].x += els2[t];  // reduce threads
    }
  }
}

/*
 * FFT laplace expectation value
 *
 * dst      = Source/destination grid for operation (gpu_mem_block *; input/output).
 * kx0      = Momentum shift of origin along X (REAL; input).
 * ky0      = Momentum shift of origin along Y (REAL; input).
 * kz0      = Momentum shift of origin along Z (REAL; input).
 * step     = Spatial step length (REAL; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 * value    = Expectation value (REAL; output).
 *
 * Normalization done in cgrid-cuda.c
 *
 */

extern __global__ void cgrid_cuda_block_init(CUCOMPLEX *, INT);
extern __global__ void cgrid_cuda_block_reduce(CUCOMPLEX *, INT);

extern "C" void cgrid_cuda_fft_laplace_expectation_valueW(gpu_mem_block *dst, CUREAL kx0, CUREAL ky0, CUREAL kz0, CUREAL step, INT nx, INT ny, INT nz, CUCOMPLEX *value) {

  SETUP_VARIABLES_SEG(dst)
  cudaXtDesc *DST = dst->gpu_info->descriptor;
  CUCOMPLEX tmp;
  INT nx2 = nx / 2, ny2 = ny / 2, nz2 = nz / 2, segx = 0, segy = 0; // segx not used
  int s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b31 = blocks1.x * blocks1.y * blocks1.z, b32 = blocks2.x * blocks2.y * blocks2.z;
  extern int cuda_get_element(void *, int, size_t, size_t, void *);

  if(dst->gpu_info->subFormat != HIPFFT_XT_FORMAT_INPLACE_SHUFFLED) {
    fprintf(stderr, "libgrid(cuda): fft_laplace_expectation_value wrong subFormat.\n");
    abort();
  }

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_block_init<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b31);
    // Blocks, Threads, dynamic memory size
    cgrid_cuda_fft_laplace_expectation_value_gpu<<<blocks1,threads,s*sizeof(CUCOMPLEX)>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) grid_gpu_mem_addr->data[i], kx0, ky0, kz0, step, nnx1, nny1, nz, ny, nx2, ny2, nz2, segy);
    segx += dsegx1;
    segy += dsegy1;
    cgrid_cuda_block_reduce<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b31);
  }

  cuda_error_check();

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    cgrid_cuda_block_init<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b32);
    // Blocks, Threads, dynamic memory size
    cgrid_cuda_fft_laplace_expectation_value_gpu<<<blocks2,threads,s*sizeof(CUCOMPLEX)>>>((CUCOMPLEX *) DST->data[i], (CUCOMPLEX *) grid_gpu_mem_addr->data[i], kx0, ky0, kz0, step, nnx2, nny2, nz, ny, nx2, ny2, nz2, segy);
    segx += dsegx2;
    segy += dsegy2;
    cgrid_cuda_block_reduce<<<1,1>>>((CUCOMPLEX *) grid_gpu_mem_addr->data[i], b32);
  }

  // Reduce over GPUs
  *value = CUMAKE(0.0,0.0);
  for(i = 0; i < ngpu2; i++) {
    cuda_get_element(grid_gpu_mem, i, 0, sizeof(CUCOMPLEX), &tmp);
    value->x += tmp.x;  /// + overloaded to device function - work around!
    value->y += tmp.y;
  }

  cuda_error_check();
}
