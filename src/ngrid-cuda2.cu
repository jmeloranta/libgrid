#include "hip/hip_runtime.h"
/*
 * CUDA device code (mixed cgrid/rgrid).
 *
 * blockDim = # of threads
 * gridDim = # of blocks
 *
 */

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include "cuda-math.h"
#include "cgrid_bc-cuda.h"
#include "cuda-vars.h"
#include "hip/hip_runtime.h"

extern void *grid_gpu_mem;
extern cudaXtDesc *grid_gpu_mem_addr;
extern "C" void cuda_error_check();

/*
 * Real to complex_re.
 *
 * dst.re = src(real). (zeroes the imag part)
 *
 */

__global__ void grid_cuda_real_to_complex_re_gpu(CUCOMPLEX *dst, CUREAL *src, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, idx2;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;    // Index for complex grid
  idx2 = (i * ny + j) * nzz + k;  // Index for real grid

  dst[idx] = CUMAKE(src[idx2], 0.0);
}

/*
 * Real to complex_re
 *
 * dst     = Destination for operation (gpu_mem_block *; output).
 * src     = Source for operation (gpu_mem_block *; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 */

extern "C" void grid_cuda_real_to_complex_reW(gpu_mem_block *dst, gpu_mem_block *src, INT nx, INT ny, INT nz) {

  dst->gpu_info->subFormat = src->gpu_info->subFormat;
  SETUP_VARIABLES(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor, *SRC = src->gpu_info->descriptor;
  INT nzz = 2 * (nz / 2 + 1);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_cuda_real_to_complex_re_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], (CUREAL *) SRC->data[i], nnx1, nny1, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_cuda_real_to_complex_re_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], (CUREAL *) SRC->data[i], nnx2, nny2, nz, nzz);
  }

  cuda_error_check();
}

/*
 * Real to complex_im.
 *
 * dst.im = src(real). (zeroes the real part)
 *
 */

__global__ void grid_cuda_real_to_complex_im_gpu(CUCOMPLEX *dst, CUREAL *src, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, idx2;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  idx2 = (i * ny + j) * nzz + k;

  dst[idx] = CUMAKE(0.0, src[idx2]);
}

/*
 * Real to complex_im
 *
 * dst     = Destination for operation (gpu_mem_block *; output).
 * src     = Source for operation (gpu_mem_block *; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 */

extern "C" void grid_cuda_real_to_complex_imW(gpu_mem_block *dst, gpu_mem_block *src, INT nx, INT ny, INT nz) {

  dst->gpu_info->subFormat = src->gpu_info->subFormat;
  SETUP_VARIABLES(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor, *SRC = src->gpu_info->descriptor;
  INT nzz = 2 * (nz / 2 + 1);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_cuda_real_to_complex_im_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], (CUREAL *) SRC->data[i], nnx1, nny1, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_cuda_real_to_complex_im_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], (CUREAL *) SRC->data[i], nnx2, nny2, nz, nzz);
  }

  cuda_error_check();
}

/*
 * Add real to complex_re.
 *
 * dst.re = dst.re + src(real).
 *
 */

__global__ void grid_cuda_add_real_to_complex_re_gpu(CUCOMPLEX *dst, CUREAL *src, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, idx2, tmp;

  if(i >= nx || j >= ny || k >= nz) return;

  tmp = i * ny + j;
  idx = tmp * nz + k;
  idx2 = tmp * nzz + k;

  dst[idx].x = CUCREAL(dst[idx]) + src[idx2];
}

/*
 * Add real to complex.re
 *
 * grida   = Destination for operation (gpu_mem_block *; output).
 * gridb   = Source for operation (gpu_mem_block *; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 */

extern "C" void grid_cuda_add_real_to_complex_reW(gpu_mem_block *dst, gpu_mem_block *src, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor, *SRC = src->gpu_info->descriptor;
  INT nzz = 2 * (nz / 2 + 1);

  if(src->gpu_info->subFormat != dst->gpu_info->subFormat) {
    fprintf(stderr, "libgrid(cuda): add_real_to_complex_re source/destination must have the same subformat.");
    abort();
  }

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_cuda_add_real_to_complex_re_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], (CUREAL *) SRC->data[i], nnx1, nny1, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_cuda_add_real_to_complex_re_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], (CUREAL *) SRC->data[i], nnx2, nny2, nz, nzz);
  }

  cuda_error_check();
}

/*
 * Add real to complex_im.
 *
 * dst.im = dst.im + src(real).
 *
 */

__global__ void grid_cuda_add_real_to_complex_im_gpu(CUCOMPLEX *dst, CUREAL *src, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, idx2, tmp;

  if(i >= nx || j >= ny || k >= nz) return;

  tmp = i * ny + j;
  idx = tmp * nz + k;
  idx2 = tmp * nzz + k;

  dst[idx].y = CUCIMAG(dst[idx]) + src[idx2];
}

/*
 * Add real to complex_im
 *
 * grida   = Destination for operation (gpu_mem_block *; output).
 * gridb   = Source for operation (gpu_mem_block *; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 */

extern "C" void grid_cuda_add_real_to_complex_imW(gpu_mem_block *dst, gpu_mem_block *src, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor, *SRC = src->gpu_info->descriptor;
  INT nzz = 2 * (nz / 2 + 1);

  if(src->gpu_info->subFormat != dst->gpu_info->subFormat) {
    fprintf(stderr, "libgrid(cuda): add_real_to_complex_im source/destination must have the same subformat.");
    abort();
  }

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_cuda_add_real_to_complex_im_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], (CUREAL *) SRC->data[i], nnx1, nny1, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_cuda_add_real_to_complex_im_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], (CUREAL *) SRC->data[i], nnx2, nny2, nz, nzz);
  }

  cuda_error_check();
}

/*
 * Product of real grid with sqnorm of complex grid:
 *
 * dst = src1 * |src2|^2
 *
 */

__global__ void grid_cuda_product_norm_gpu(CUREAL *dst, CUREAL *src1, CUCOMPLEX *src2, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, idx2, tmp;

  if(i >= nx || j >= ny || k >= nz) return;

  tmp = i * ny + j;
  idx = tmp * nz + k;
  idx2 = tmp * nzz + k;

  dst[idx2] = src1[idx2] * CUCSQNORM(src2[idx]);
}

/*
 * Product of src1 with sqnorm of src2.
 *
 * dst     = Destination for operation (gpu_mem_block *; output).
 * src1    = Source for operation 1 (gpu_mem_block *; input).
 * src2    = Source for operation 2 (gpu_mem_block *; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 */

extern "C" void grid_cuda_product_normW(gpu_mem_block *dst, gpu_mem_block *src1, gpu_mem_block *src2, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(src2);
  cudaXtDesc *DST = dst->gpu_info->descriptor, *SRC1 = src1->gpu_info->descriptor, *SRC2 = src2->gpu_info->descriptor;
  INT nzz = 2 * (nz / 2 + 1);

  if(src1->gpu_info->subFormat != src2->gpu_info->subFormat) {
    fprintf(stderr, "libgrid(cuda): product_norm source grids must have the same subformat.");
    abort();
  }

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_cuda_product_norm_gpu<<<blocks1,threads>>>((CUREAL *) DST->data[i], (CUREAL *) SRC1->data[i], (CUCOMPLEX *) SRC2->data[i], nnx1, nny1, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_cuda_product_norm_gpu<<<blocks2,threads>>>((CUREAL *) DST->data[i], (CUREAL *) SRC1->data[i], (CUCOMPLEX *) SRC2->data[i], nnx2, nny2, nz, nzz);
  }

  cuda_error_check();
}

/*
 * Divide real grid with sqnorm of complex grid:
 *
 * dst = src1 / (|src2|^2 + eps)
 *
 */

__global__ void grid_cuda_division_norm_gpu(CUREAL *dst, CUREAL *src1, CUCOMPLEX *src2, CUREAL eps, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, idx2, tmp;

  if(i >= nx || j >= ny || k >= nz) return;

  tmp = i * ny + j;
  idx = tmp * nz + k;
  idx2 = tmp * nzz + k;

  dst[idx2] = src1[idx2] / (CUCSQNORM(src2[idx]) + eps);
}

/*
 * Division src1 with sqnorm of src2.
 *
 * dst     = Destination for operation (gpu_mem_block *; output).
 * src1    = Source for operation 1 (gpu_mem_block *; input).
 * src2    = Source for operation 2 (gpu_mem_block *; input).
 * eps     = Epsilon for division (REAL; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 */

extern "C" void grid_cuda_division_normW(gpu_mem_block *dst, gpu_mem_block *src1, gpu_mem_block *src2, REAL eps, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(src2);
  cudaXtDesc *DST = dst->gpu_info->descriptor, *SRC1 = src1->gpu_info->descriptor, *SRC2 = src2->gpu_info->descriptor;
  INT nzz = 2 * (nz / 2 + 1);

  if(src1->gpu_info->subFormat != src2->gpu_info->subFormat) {
    fprintf(stderr, "libgrid(cuda): product_norm source grids must have the same subformat.");
    abort();
  }

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_cuda_division_norm_gpu<<<blocks1,threads>>>((CUREAL *) DST->data[i], (CUREAL *) SRC1->data[i], (CUCOMPLEX *) SRC2->data[i], eps, nnx1, nny1, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_cuda_division_norm_gpu<<<blocks2,threads>>>((CUREAL *) DST->data[i], (CUREAL *) SRC1->data[i], (CUCOMPLEX *) SRC2->data[i], eps, nnx2, nny2, nz, nzz);
  }

  cuda_error_check();
}

/*
 * Product dst(complex) and src(real).
 *
 * dst = dst * src(real).
 *
 */

__global__ void grid_cuda_product_complex_with_real_gpu(CUCOMPLEX *dst, CUREAL *src, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, idx2, tmp;

  if(i >= nx || j >= ny || k >= nz) return;

  tmp = i * ny + j;
  idx = tmp * nz + k;
  idx2 = tmp * nzz + k;

  dst[idx] = dst[idx] * src[idx2];
}

/*
 * Product dst(complex) with src(real).
 *
 * dst     = Destination for operation (gpu_mem_block *; output).
 * src     = Source for operation (gpu_mem_block *; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 */

extern "C" void grid_cuda_product_complex_with_realW(gpu_mem_block *dst, gpu_mem_block *src, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor, *SRC = src->gpu_info->descriptor;
  INT nzz = 2 * (nz / 2 + 1);

  if(src->gpu_info->subFormat != dst->gpu_info->subFormat) {
    fprintf(stderr, "libgrid(cuda): product_complex_with_real source/destination must have the same subformat.");
    abort();
  }

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_cuda_product_complex_with_real_gpu<<<blocks1,threads>>>((CUCOMPLEX *) DST->data[i], (CUREAL *) SRC->data[i], nnx1, nny1, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_cuda_product_complex_with_real_gpu<<<blocks2,threads>>>((CUCOMPLEX *) DST->data[i], (CUREAL *) SRC->data[i], nnx2, nny2, nz, nzz);
  }

  cuda_error_check();
}

/*
 * Imag. part to real grid.
 *
 * dst(real) = src.im;
 *
 */

__global__ void grid_cuda_complex_im_to_real_gpu(CUREAL *dst, CUCOMPLEX *src, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, idx2, tmp;

  if(i >= nx || j >= ny || k >= nz) return;

  tmp = i * ny + j;
  idx = tmp * nz + k;
  idx2 = tmp * nzz + k;

  dst[idx2] = CUCIMAG(src[idx]);
}

/*
 * Imag. part of src to real dst.
 *
 * dst     = Destination for operation (gpu_mem_block *; output).
 * src     = Source for operation (gpu_mem_block *; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 */

extern "C" void grid_cuda_complex_im_to_realW(gpu_mem_block *dst, gpu_mem_block *src, INT nx, INT ny, INT nz) {

  dst->gpu_info->subFormat = src->gpu_info->subFormat;
  SETUP_VARIABLES(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor, *SRC = src->gpu_info->descriptor;
  INT nzz = 2 * (nz / 2 + 1);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_cuda_complex_im_to_real_gpu<<<blocks1,threads>>>((CUREAL *) DST->data[i], (CUCOMPLEX *) SRC->data[i], nnx1, nny1, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_cuda_complex_im_to_real_gpu<<<blocks2,threads>>>((CUREAL *) DST->data[i], (CUCOMPLEX *) SRC->data[i], nnx2, nny2, nz, nzz);
  }

  cuda_error_check();
}

/*
 * Real part to real grid.
 *
 * dst(real) = src.re;
 *
 */

__global__ void grid_cuda_complex_re_to_real_gpu(CUREAL *dst, CUCOMPLEX *src, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, idx2, tmp;

  if(i >= nx || j >= ny || k >= nz) return;

  tmp = i * ny + j;
  idx = tmp * nz + k;
  idx2 = tmp * nzz + k;

  dst[idx2] = CUCREAL(src[idx]);
}

/*
 * Real part of B to real A.
 *
 * dst     = Destination for operation (gpu_mem_block *; output).
 * src     = Source for operation (gpu_mem_block *; input).
 * nx      = # of points along x (INT; input).
 * ny      = # of points along y (INT; input).
 * nz      = # of points along z (INT; input).
 *
 */

extern "C" void grid_cuda_complex_re_to_realW(gpu_mem_block *dst, gpu_mem_block *src, INT nx, INT ny, INT nz) {

  dst->gpu_info->subFormat = src->gpu_info->subFormat;
  SETUP_VARIABLES(dst);
  cudaXtDesc *DST = dst->gpu_info->descriptor, *SRC = src->gpu_info->descriptor;
  INT nzz = 2 * (nz / 2 + 1);

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_cuda_complex_re_to_real_gpu<<<blocks1,threads>>>((CUREAL *) DST->data[i], (CUCOMPLEX *) SRC->data[i], nnx1, nny1, nz, nzz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DST->GPUs[i]);
    grid_cuda_complex_re_to_real_gpu<<<blocks2,threads>>>((CUREAL *) DST->data[i], (CUCOMPLEX *) SRC->data[i], nnx2, nny2, nz, nzz);
  }

  cuda_error_check();
}

/*
 * Integrate opgrid * |dgrid|^2.
 *
 */

__global__ void grid_cuda_grid_expectation_value_gpu(CUCOMPLEX *dgrid, CUREAL *opgrid, CUREAL *blocks, INT nx, INT ny, INT nz, INT nzz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z;
  INT d = blockDim.x * blockDim.y * blockDim.z, idx, idx2, idxc, t;
  extern __shared__ CUREAL els[];

  if(i >= nx || j >= ny || k >= nz) return;

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++)
      els[t] = 0.0;
  }
  __syncthreads();

  idx = (i * ny + j) * nzz + k;
  idxc = (i * ny + j) * nz + k;
  idx2 = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;

  els[idx2] += opgrid[idx] * CUCSQNORM(dgrid[idxc]);
  __syncthreads();

  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for(t = 0; t < d; t++) {
      idx2 = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
      blocks[idx2] += els[t];  // reduce threads
    }
  }
}

/*
 * Integral opgrid * |dgrid|^2
 *
 * dgrid    = Source 1 for operation (gpu_mem_block *; input).
 * opgrid   = Source 2 for operation (gpu_mem_block *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 * *value   = Return value (CUREAL *; output).
 *
 */

extern __global__ void rgrid_cuda_block_init(CUREAL *, INT);
extern __global__ void rgrid_cuda_block_reduce(CUREAL *, INT);

extern "C" void grid_cuda_grid_expectation_valueW(gpu_mem_block *dgrid, gpu_mem_block *opgrid, INT nx, INT ny, INT nz, CUREAL *value) {

  SETUP_VARIABLES_REAL(dgrid);
  cudaXtDesc *DGRID = dgrid->gpu_info->descriptor, *OPGRID = opgrid->gpu_info->descriptor;
  CUREAL tmp;
  INT s = CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK * CUDA_THREADS_PER_BLOCK, b31 = blocks1.x * blocks1.y * blocks1.z, b32 = blocks2.x * blocks2.y * blocks2.z;
  extern int cuda_get_element(void *, int, size_t, size_t, void *);

  if(dgrid->gpu_info->subFormat != HIPFFT_XT_FORMAT_INPLACE || opgrid->gpu_info->subFormat != HIPFFT_XT_FORMAT_INPLACE) {
    fprintf(stderr, "libgrid(cuda): grid_expectation_value grids must be in real space (INPLACE).");
    abort();
  }

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(DGRID->GPUs[i]);
    rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b31);
    // Blocks, Threads, dynamic memory size
    grid_cuda_grid_expectation_value_gpu<<<blocks1,threads,s*sizeof(CUREAL)>>>((CUCOMPLEX *) DGRID->data[i], (CUREAL *) OPGRID->data[i], 
                                                                                (CUREAL *) grid_gpu_mem_addr->data[i], nnx1, ny, nz, nzz);
    rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b31);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(DGRID->GPUs[i]);
    rgrid_cuda_block_init<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b32);
    // Blocks, Threads, dynamic memory size
    grid_cuda_grid_expectation_value_gpu<<<blocks2,threads,s*sizeof(CUREAL)>>>((CUCOMPLEX *) DGRID->data[i], (CUREAL *) OPGRID->data[i], 
                                                                                (CUREAL *) grid_gpu_mem_addr->data[i], nnx2, ny, nz, nzz);
    rgrid_cuda_block_reduce<<<1,1>>>((CUREAL *) grid_gpu_mem_addr->data[i], b32);
  }

  // Reduce over GPUs
  *value = 0.0;
  for(i = 0; i < ngpu2; i++) {
    cuda_get_element(grid_gpu_mem, i, 0, sizeof(CUREAL), &tmp);
    *value = *value + tmp;
  }

  cuda_error_check();
}
