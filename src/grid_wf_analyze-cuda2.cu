#include "hip/hip_runtime.h"
/*
 * CUDA device code for wf analyze.
 *
 * TODO: Many routines have periodic BCs hardcoded...
 *
 */

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>
#include "hip/hip_runtime.h"
#include "cuda-math.h"
#include "defs.h"
#include "cuda-vars.h"

extern void *grid_gpu_mem_addr;
extern "C" void cuda_error_check();

/*
 * Flux X (finite difference).
 *
 */

__global__ void grid_cuda_wf_fd_probability_flux_x_gpu(CUCOMPLEX *wf, CUREAL *flux, CUREAL inv_delta, INT nx, INT ny, INT nz, INT nz2) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, idx2;
  CUCOMPLEX wp, wm;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  idx2 = (i * ny + j) * nz2 + k;
 
  if(i == 0) wm = wf[((nx-1)*ny + j)*nz + k];
  else wm = wf[((i-1)*ny + j)*nz + k];
  if(i == nx-1) wp = wf[j*nz + k]; // i = 0
  else wp = wf[((i+1)*ny + j)*nz + k];

  flux[idx2] = inv_delta * CUCIMAG(CUCONJ(wf[idx]) * (wp - wm));
}

/*
 * Flux x (finite difference).
 *
 * wf       = Source for operation (gpu_mem_block *; input).
 * flux     = Destination grid (gpu_mem_block *; output).
 * inv_delta= hbar / (2 * mass * step) (CUREAL; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 * nz2      = # of points along z for real grid (INT).
 *
 */

extern "C" void grid_cuda_wf_fd_probability_flux_xW(gpu_mem_block *gwf, gpu_mem_block *flux, CUREAL inv_delta, INT nx, INT ny, INT nz, INT nz2) {

  flux->gpu_info->subFormat = HIPFFT_XT_FORMAT_INPLACE;
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  cudaXtDesc *GWF = gwf->gpu_info->descriptor, *FLUX = flux->gpu_info->descriptor;

  if(gwf->gpu_info->subFormat != HIPFFT_XT_FORMAT_INPLACE) {
    fprintf(stderr, "libgrid(cuda): wf_probability_flux_x wrong subformat.\n");
    abort();
  }

  if(GWF->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(GWF->GPUs[0]);
  grid_cuda_wf_fd_probability_flux_x_gpu<<<blocks,threads>>>((CUCOMPLEX *) GWF->data[0], (CUREAL *) FLUX->data[0], inv_delta, nx, ny, nz, nz2);

  cuda_error_check();
}

/*
 * Flux Y (finite difference).
 *
 */

__global__ void grid_cuda_wf_fd_probability_flux_y_gpu(CUCOMPLEX *wf, CUREAL *flux, CUREAL inv_delta, INT nx, INT ny, INT nz, INT nz2) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, idx2;
  CUCOMPLEX wp, wm;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  idx2 = (i * ny + j) * nz2 + k;

  if(j == 0) wm = wf[(i*ny + ny-1)*nz + k];
  else wm = wf[(i*ny + j - 1)*nz + k];
  if(j == ny-1) wp = wf[i*ny*nz + k];
  else wp = wf[(i*ny + j + 1)*nz + k];

  flux[idx2] = inv_delta * CUCIMAG(CUCONJ(wf[idx]) * (wp - wm));
}

/*
 * Flux y (finite difference).
 *
 * wf       = Source/destination grid for operation (gpu_mem_block *; input).
 * flux     = Flux grid (gpu_mem_block *; output).
 * inv_delta= hbar / (2 * mass * step) (CUREAL; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 * nz2      = # of points along z for real grid (INT).
 *
 */

extern "C" void grid_cuda_wf_fd_probability_flux_yW(gpu_mem_block *gwf, gpu_mem_block *flux, CUREAL inv_delta, INT nx, INT ny, INT nz, INT nz2) {

  flux->gpu_info->subFormat = HIPFFT_XT_FORMAT_INPLACE;
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  cudaXtDesc *GWF = gwf->gpu_info->descriptor, *FLUX = flux->gpu_info->descriptor;

  if(gwf->gpu_info->subFormat != HIPFFT_XT_FORMAT_INPLACE) {
    fprintf(stderr, "libgrid(cuda): wf_probability_flux_y wrong subformat.\n");
    abort();
  }

  if(GWF->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(GWF->GPUs[0]);
  grid_cuda_wf_fd_probability_flux_y_gpu<<<blocks,threads>>>((CUCOMPLEX *) GWF->data[0], (CUREAL *) FLUX->data[0], inv_delta, nx, ny, nz, nz2);

  cuda_error_check();
}

/*
 * Flux Z (finite difference).
 *
 */

__global__ void grid_cuda_wf_fd_probability_flux_z_gpu(CUCOMPLEX *wf, CUREAL *flux, CUREAL inv_delta, INT nx, INT ny, INT nz, INT nz2) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx, idx2;
  CUCOMPLEX wp, wm;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;
  idx2 = (i * ny + j) * nz2 + k;
 
  if(k == 0) wm = wf[(i*ny + j)*nz + nz-1];
  else wm = wf[(i*ny + j)*nz + k-1];
  if(k == nz-1) wp = wf[(i*ny + j)*nz];
  else wp = wf[(i*ny + j)*nz + k+1];

  flux[idx2] = inv_delta * CUCIMAG(CUCONJ(wf[idx]) * (wp - wm));
}

/*
 * Flux z (finite difference).
 *
 * wf       = Source/destination grid for operation (gpu_mem_block *; input).
 * flux     = Flux grid (gpu_mem_block *; output).
 * inv_delta= hbar / (2 * mass * step) (CUREAL; input).
 * nx       = # of points along x (INT).
 * ny       = # of points along y (INT).
 * nz       = # of points along z (INT).
 * nz2      = # of points along z for real grid (INT).
 *
 */

extern "C" void grid_cuda_wf_fd_probability_flux_zW(gpu_mem_block *gwf, gpu_mem_block *flux, CUREAL inv_delta, INT nx, INT ny, INT nz, INT nz2) {

  flux->gpu_info->subFormat = HIPFFT_XT_FORMAT_INPLACE;
  dim3 threads(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);
  dim3 blocks((nz + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (ny + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK,
              (nx + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK);
  cudaXtDesc *GWF = gwf->gpu_info->descriptor, *FLUX = flux->gpu_info->descriptor;

  if(gwf->gpu_info->subFormat != HIPFFT_XT_FORMAT_INPLACE) {
    fprintf(stderr, "libgrid(cuda): wf_probability_flux_z wrong subformat.\n");
    abort();
  }

  if(GWF->nGPUs > 1) {
    fprintf(stderr, "libgrid(cuda): Non-local grid operations disabled for multi-GPU calculations.\n");
    abort();
  }

  hipSetDevice(GWF->GPUs[0]);
  grid_cuda_wf_fd_probability_flux_z_gpu<<<blocks,threads>>>((CUCOMPLEX *) GWF->data[0], (CUREAL *) FLUX->data[0], inv_delta, nx, ny, nz, nz2);

  cuda_error_check();
}

/*
 * Entropy.
 *
 */

__global__ void grid_cuda_wf_entropy_gpu(CUCOMPLEX *grd, INT nx, INT ny, INT nz) {

  INT k = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y, i = blockIdx.z * blockDim.z + threadIdx.z, idx;
  CUCOMPLEX s;

  if(i >= nx || j >= ny || k >= nz) return;

  idx = (i * ny + j) * nz + k;

  s = grd[idx];
  grd[idx].x = s.x * LOG(GRID_EPS + s.x);
  grd[idx].y = 0.0;
}

/*
 * Entropy.
 *
 * grid     = Source/destination for operation (gpu_mem_block *; input).
 * nx       = # of points along x (INT; input).
 * ny       = # of points along y (INT; input).
 * nz       = # of points along z (INT; input).
 *
 */

extern "C" void grid_cuda_wf_entropyW(gpu_mem_block *grid, INT nx, INT ny, INT nz) {

  SETUP_VARIABLES(grid);
  cudaXtDesc *GRID = grid->gpu_info->descriptor;

  if(grid->gpu_info->subFormat != HIPFFT_XT_FORMAT_INPLACE_SHUFFLED) {
    fprintf(stderr, "libgrid(cuda): wf_entropy wrong subformat.\n");
    abort();
  }

  for(i = 0; i < ngpu1; i++) {
    hipSetDevice(GRID->GPUs[i]);
    grid_cuda_wf_entropy_gpu<<<blocks1,threads>>>((CUCOMPLEX *) GRID->data[i], nnx1, nny1, nz);
  }

  for(i = ngpu1; i < ngpu2; i++) {
    hipSetDevice(GRID->GPUs[i]);
    grid_cuda_wf_entropy_gpu<<<blocks2,threads>>>((CUCOMPLEX *) GRID->data[i], nnx2, nny2, nz);
  }

  cuda_error_check();
}
